/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * dftregistration_coderGPU_initialize.cu
 *
 * Code generation for function 'dftregistration_coderGPU_initialize'
 *
 */

/* Include files */
#include "dftregistration_coderGPU_initialize.h"
#include "_coder_dftregistration_coderGPU_mex.h"
#include "dftregistration_coderGPU.h"
#include "dftregistration_coderGPU_data.h"
#include "rt_nonfinite.h"

/* Function Declarations */
static void cublasEnsureInitialization();

/* Function Definitions */
static void cublasEnsureInitialization()
{
  if (cublasGlobalHandle == NULL) {
    hipblasCreate(&cublasGlobalHandle);
    hipblasSetPointerMode(cublasGlobalHandle, HIPBLAS_POINTER_MODE_DEVICE);
  }
}

void dftregistration_coderGPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  cublasEnsureInitialization();
  hipGetLastError();
}

/* End of code generation (dftregistration_coderGPU_initialize.cu) */
