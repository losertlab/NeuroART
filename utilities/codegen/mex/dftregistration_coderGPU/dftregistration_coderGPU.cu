#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * dftregistration_coderGPU.cu
 *
 * Code generation for function 'dftregistration_coderGPU'
 *
 */

/* Include files */
#include "dftregistration_coderGPU.h"
#include "MWCudaDimUtility.h"
#include "MWLaunchParametersUtilities.h"
#include "dftregistration_coderGPU_data.h"
#include "dftregistration_coderGPU_emxutil.h"
#include "hip/hip_math_constants.h"
#include "rt_defines.h"
#include "rt_nonfinite.h"
#include <math.h>
#include <string.h>

/* Type Definitions */
#include "hipblas.h"
#include "hipfft/hipfft.h"

/* Variable Definitions */
static hipfftHandle cufftGlobalHandle;
static hipfftHandle *cufftGlobalHandlePtr = NULL;
static int32_T cufftGlobalSize = 0U;
static int32_T cufftGlobalBatch = 0U;
static hipfftType_t cufftGlobalType = HIPFFT_R2C;
static int32_T cufftGlobalDist = 0U;

/* Function Declarations */
static __global__ void ab_dftregistration_coderGPU_ker(const int16_T yk, creal_T
  *y);
static __global__ void ac_dftregistration_coderGPU_ker(int16_T idx[512]);
static __global__ void ad_dftregistration_coderGPU_ker(const real_T *tmp_re,
  creal_T *y);
static __inline__ __device__ real_T atomicOpreal_T(real_T *address, real_T value);
static __device__ real_T b_rt_atan2d_snf(real_T u0, real_T u1);
static __device__ real_T b_rt_hypotd_snf(real_T u0, real_T u1);
static __global__ void bb_dftregistration_coderGPU_ker(const creal_T *y, const
  int16_T cloc, const emxArray_int16_T *b_y, emxArray_creal_T *z);
static __global__ void bc_dftregistration_coderGPU_ker(int16_T idx[512], creal_T
  CC[262144], creal_T y[512]);
static __global__ void bd_dftregistration_coderGPU_ker(const creal_T *y, const
  creal_T buf2ft[262144], creal_T Greg[262144]);
static __global__ void c_dftregistration_coderGPU_kern(creal_T CC[1048576]);
static __global__ void cb_dftregistration_coderGPU_ker(const int16_T x[512],
  creal_T y[512]);
static __global__ void cc_dftregistration_coderGPU_ker(const creal_T buf2ft
  [262144], const creal_T buf1ft[262144], real_T absb1[262144], real_T absb2
  [262144]);
static __global__ void cd_dftregistration_coderGPU_ker(const int16_T b[512],
  int16_T Nr[262144], int16_T Nc[262144]);
static void cufftEnsureDestruction();
static void cufftEnsureInitialization(int32_T nelem, hipfftType_t type, int32_T
  batch, int32_T idist);
static __global__ void d_dftregistration_coderGPU_kern(const creal_T buf1ft
  [262144], creal_T CC[262144]);
static __global__ void db_dftregistration_coderGPU_ker(const emxArray_creal_T *z,
  emxArray_creal_T *kernr, creal_T y[512]);
static __global__ void dc_dftregistration_coderGPU_ker(const real_T absb1[262144],
  real_T *absar);
static __global__ void dd_dftregistration_coderGPU_ker(const int16_T Nc[262144],
  const real_T col_shift, const int16_T Nr[262144], const real_T *row_shift,
  creal_T Greg[262144]);
static __global__ void e_dftregistration_coderGPU_kern(const creal_T buf2ft
  [262144], creal_T x[262144]);
static __global__ void eb_dftregistration_coderGPU_ker(const emxArray_creal_T
  *kernr, int32_T *nx);
static __global__ void ec_dftregistration_coderGPU_ker(const real_T absb1[262144],
  real_T *absar);
static __global__ void ed_dftregistration_coderGPU_ker(creal_T Greg[262144]);
static __global__ void f_dftregistration_coderGPU_kern(creal_T CC[262144],
  creal_T b_CC[1048576], creal_T x[262144]);
static __global__ void fb_dftregistration_coderGPU_ker(const int32_T *nx,
  emxArray_creal_T *kernr);
static __global__ void fc_dftregistration_coderGPU_ker(real_T absb2[262144]);
static __global__ void fd_dftregistration_coderGPU_ker(const real_T *tmp_im,
  creal_T *y);
static __global__ void g_dftregistration_coderGPU_kern(creal_T CC[1048576],
  creal_T y[1048576]);
static __global__ void gb_dftregistration_coderGPU_ker(const emxArray_creal_T
  *kernr, emxArray_creal_T *y);
static __global__ void gc_dftregistration_coderGPU_ker(const real_T absb2[262144],
  real_T *tmp_re);
static __global__ void gd_dftregistration_coderGPU_ker(const creal_T *y, real_T *
  tmp_re);
static void gpuEmxFree_creal_T(emxArray_creal_T *inter);
static void gpuEmxFree_int16_T(emxArray_int16_T *inter);
static void gpuEmxFree_real_T(emxArray_real_T *inter);
static void gpuEmxFree_uint16_T(emxArray_uint16_T *inter);
static void gpuEmxMemcpyCpuToGpu_creal_T(const emxArray_creal_T *cpu,
  emxArray_creal_T *inter, emxArray_creal_T *gpu);
static void gpuEmxMemcpyCpuToGpu_int16_T(const emxArray_int16_T *cpu,
  emxArray_int16_T *inter, emxArray_int16_T *gpu);
static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *inter, emxArray_real_T *gpu);
static void gpuEmxMemcpyCpuToGpu_uint16_T(const emxArray_uint16_T *cpu,
  emxArray_uint16_T *inter, emxArray_uint16_T *gpu);
static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu, emxArray_creal_T
  *inter);
static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *inter);
static void gpuEmxMemcpyGpuToCpu_uint16_T(emxArray_uint16_T *cpu,
  emxArray_uint16_T *inter);
static void gpuEmxReset_creal_T(emxArray_creal_T *inter);
static void gpuEmxReset_int16_T(emxArray_int16_T *inter);
static void gpuEmxReset_real_T(emxArray_real_T *inter);
static void gpuEmxReset_uint16_T(emxArray_uint16_T *inter);
static __global__ void h_dftregistration_coderGPU_kern(creal_T CC[1048576]);
static __global__ void hb_dftregistration_coderGPU_ker(const emxArray_creal_T
  *kernc, const emxArray_creal_T *y, emxArray_creal_T *out);
static __global__ void hc_dftregistration_coderGPU_ker(const real_T absb2[262144],
  real_T *tmp_re);
static __global__ void hd_dftregistration_coderGPU_ker(const real_T *tmp_im,
  const real_T *tmp_re, creal_T *y);
static __global__ void i_dftregistration_coderGPU_kern(creal_T CC[1048576],
  creal_T y[1048576]);
static __global__ void ib_dftregistration_coderGPU_ker(const real_T *absar,
  const emxArray_creal_T *out, emxArray_creal_T *CC);
static __global__ void ic_dftregistration_coderGPU_ker(const real_T col_shift,
  const real_T *row_shift, const real_T *tmp_im, const real_T Ma, const real_T
  absbr, real_T dv1[4]);
static __global__ void id_dftregistration_coderGPU_ker(const creal_T *y, real_T *
  tmp_re);
static __global__ void j_dftregistration_coderGPU_kern(creal_T CC[1048576]);
static __global__ void jb_dftregistration_coderGPU_ker(const emxArray_creal_T
  *ex, uint16_T uv[2]);
static __global__ void jc_dftregistration_coderGPU_ker(const real_T dv1[4],
  emxArray_real_T *output);
static __global__ void jd_dftregistration_coderGPU_ker(const real_T *tmp_re,
  creal_T *y);
static __global__ void k_dftregistration_coderGPU_kern(int16_T idx[1024]);
static __global__ void kb_dftregistration_coderGPU_ker(const uint16_T uv[2],
  emxArray_uint16_T *idx);
static __global__ void kc_dftregistration_coderGPU_ker(const creal_T buf2ft
  [262144], const creal_T buf1ft[262144], creal_T CC[262144], real_T absb2
  [262144], real_T absb1[262144]);
static __global__ void kd_dftregistration_coderGPU_ker(const creal_T *y, const
  creal_T buf2ft[262144], creal_T Greg[262144]);
static __global__ void l_dftregistration_coderGPU_kern(int16_T idx[1024],
  creal_T CC[1048576], creal_T ex[1024]);
static __global__ void lb_dftregistration_coderGPU_ker(const emxArray_creal_T
  *CC, const int32_T vstride, const int32_T npages, emxArray_uint16_T *idx,
  emxArray_creal_T *ex);
static __global__ void lc_dftregistration_coderGPU_ker(creal_T y[512], creal_T
  CC[262144]);
static __global__ void m_dftregistration_coderGPU_kern(const int16_T idx[1024],
  const int32_T *nx, real_T *row_shift, creal_T CC[1048576], creal_T *CCmax);
static __global__ void mb_dftregistration_coderGPU_ker(const emxArray_creal_T
  *CC, const int32_T *nx, const emxArray_uint16_T *idx, creal_T *CCmax);
static __global__ void mc_dftregistration_coderGPU_ker(creal_T *CCmax);
static __global__ void n_dftregistration_coderGPU_kern(const creal_T buf1ft
  [262144], creal_T CC[262144]);
static __global__ void nb_dftregistration_coderGPU_ker(const creal_T buf1ft
  [262144], creal_T CC[262144]);
static __global__ void nc_dftregistration_coderGPU_ker(real_T absb1[262144]);
static __global__ void o_dftregistration_coderGPU_kern(creal_T y[512], creal_T
  CC[262144]);
static __global__ void ob_dftregistration_coderGPU_ker(creal_T y[512], creal_T
  kernr[512]);
static __global__ void oc_dftregistration_coderGPU_ker(const real_T absb1[262144],
  real_T *tmp_re);
static __global__ void p_dftregistration_coderGPU_kern(creal_T *y);
static __global__ void pb_dftregistration_coderGPU_ker(creal_T *y);
static __global__ void pc_dftregistration_coderGPU_ker(const real_T absb1[262144],
  real_T *tmp_re);
static __global__ void q_dftregistration_coderGPU_kern(const creal_T buf2ft
  [262144], creal_T CC[262144]);
static __global__ void qb_dftregistration_coderGPU_ker(const creal_T buf2ft
  [262144], creal_T CC[262144]);
static __global__ void qc_dftregistration_coderGPU_ker(real_T absb2[262144]);
static __global__ void r_dftregistration_coderGPU_kern(creal_T y[512], creal_T
  CC[262144]);
static __global__ void rb_dftregistration_coderGPU_ker(creal_T y[512], creal_T
  kernr[512]);
static __global__ void rc_dftregistration_coderGPU_ker(const real_T absb2[262144],
  real_T *tmp_im);
static real_T rt_atan2d_snf(real_T u0, real_T u1);
static real_T rt_hypotd_snf(real_T u0, real_T u1);
static real_T rt_roundd_snf(real_T u);
static __global__ void s_dftregistration_coderGPU_kern(creal_T *y);
static __global__ void sb_dftregistration_coderGPU_ker(creal_T *y);
static __global__ void sc_dftregistration_coderGPU_ker(const real_T absb2[262144],
  real_T *tmp_im);
static __inline__ __device__ real_T shflDown2(real_T in1, uint32_T offset,
  uint32_T mask);
static __global__ void t_dftregistration_coderGPU_kern(const creal_T buf1ft
  [262144], const creal_T buf2ft[262144], creal_T CC[262144]);
static __global__ void tb_dftregistration_coderGPU_ker(const real_T col_shift,
  const real_T *row_shift, const real_T *tmp_im, const real_T CCmax_im, const
  real_T Ma, real_T dv1[4]);
static __global__ void tc_dftregistration_coderGPU_ker(const real_T *tmp_im,
  const real_T CCmax_im, const real_T ex_im, real_T dv[2]);
static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
  uint32_T mask);
static __global__ void u_dftregistration_coderGPU_kern(const int16_T yk, creal_T
  *y);
static __global__ void ub_dftregistration_coderGPU_ker(const real_T dv1[4],
  emxArray_real_T *output);
static __global__ void uc_dftregistration_coderGPU_ker(const real_T dv[2],
  emxArray_real_T *output);
static __global__ void v_dftregistration_coderGPU_kern(const int16_T coff, const
  emxArray_int16_T *y, emxArray_creal_T *b_y);
static __global__ void vb_dftregistration_coderGPU_ker(const creal_T buf1ft
  [262144], const creal_T buf2ft[262144], creal_T CC[262144]);
static __global__ void vc_dftregistration_coderGPU_ker(const real_T *tmp_im,
  creal_T *y);
static __global__ void w_dftregistration_coderGPU_kern(const emxArray_creal_T *y,
  const creal_T *b_y, const int16_T b[512], int32_T *nx, emxArray_creal_T *kernc);
static __global__ void wb_dftregistration_coderGPU_ker(creal_T b_y1[262144]);
static __global__ void wc_dftregistration_coderGPU_ker(const creal_T *y, real_T *
  tmp_re);
static __device__ real_T workGroupReduction(real_T val, uint32_T mask, uint32_T
  numActiveWarps);
static __global__ void x_dftregistration_coderGPU_kern(const emxArray_creal_T
  *kernc, int32_T *nx);
static __global__ void xb_dftregistration_coderGPU_ker(creal_T b_y1[262144],
  creal_T x[262144]);
static __global__ void xc_dftregistration_coderGPU_ker(const real_T *tmp_im,
  const real_T *tmp_re, creal_T *y);
static __global__ void y_dftregistration_coderGPU_kern(const int32_T *nx,
  emxArray_creal_T *kernc);
static __global__ void yb_dftregistration_coderGPU_ker(creal_T CC[262144]);
static __global__ void yc_dftregistration_coderGPU_ker(const creal_T *y, real_T *
  tmp_re);

/* Function Definitions */
static __global__ __launch_bounds__(32, 1) void ab_dftregistration_coderGPU_ker(
  const int16_T yk, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->im = -6.2831853071795862 / static_cast<real_T>(yk);
  }
}

static __global__ __launch_bounds__(512, 1) void ac_dftregistration_coderGPU_ker
  (int16_T idx[512])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    idx[i] = static_cast<int16_T>(1);
  }
}

static __global__ __launch_bounds__(32, 1) void ad_dftregistration_coderGPU_ker(
  const real_T *tmp_re, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = exp(*tmp_re);
    y->im = 0.0;
  }
}

static __inline__ __device__ real_T atomicOpreal_T(real_T *address, real_T value)
{
  unsigned long long int *address_as_up;
  unsigned long long int old;
  unsigned long long int assumed;
  address_as_up = (unsigned long long int *)address;
  old = *address_as_up;
  do {
    assumed = old;
    old = atomicCAS(address_as_up, old, __double_as_longlong(value +
      __longlong_as_double(old)));
  } while (assumed != old);

  return __longlong_as_double(old);
}

static __device__ real_T b_rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  int32_T b_u0;
  int32_T b_u1;
  if ((static_cast<int32_T>(isnan(u0))) || (static_cast<int32_T>(isnan(u1)))) {
    y = HIP_NAN;
  } else if ((static_cast<int32_T>(isinf(u0))) && (static_cast<int32_T>(isinf(u1))))
  {
    if (u0 > 0.0) {
      b_u0 = 1;
    } else {
      b_u0 = -1;
    }

    if (u1 > 0.0) {
      b_u1 = 1;
    } else {
      b_u1 = -1;
    }

    y = atan2(static_cast<real_T>(b_u0), static_cast<real_T>(b_u1));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = atan2(u0, u1);
  }

  return y;
}

static __device__ real_T b_rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T y;
  real_T a;
  real_T b;
  a = fabs(u0);
  b = fabs(u1);
  if (a < b) {
    a /= b;
    y = b * sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * sqrt(b * b + 1.0);
  } else if (isnan(b)) {
    y = b;
  } else {
    y = a * 1.4142135623730951;
  }

  return y;
}

static __global__ __launch_bounds__(1024, 1) void
  bb_dftregistration_coderGPU_ker(const creal_T *y, const int16_T cloc, const
  emxArray_int16_T *b_y, emxArray_creal_T *z)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(b_y->size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    iv0 = static_cast<int32_T>(b_y->data[i]) - static_cast<int32_T>(cloc);
    if (iv0 > 32767) {
      iv0 = 32767;
    } else {
      if (iv0 < -32768) {
        iv0 = -32768;
      }
    }

    z->data[i].re = static_cast<real_T>(iv0) * 0.0;
    iv0 = static_cast<int32_T>(b_y->data[i]) - static_cast<int32_T>(cloc);
    if (iv0 > 32767) {
      iv0 = 32767;
    } else {
      if (iv0 < -32768) {
        iv0 = -32768;
      }
    }

    z->data[i].im = static_cast<real_T>(iv0) * y->im;
  }
}

static __global__ __launch_bounds__(512, 1) void bc_dftregistration_coderGPU_ker
  (int16_T idx[512], creal_T CC[262144], creal_T y[512])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T v;
  int32_T i;
  int32_T xpageoffset;
  boolean_T p;
  boolean_T SCALEA;
  real_T x;
  real_T br;
  real_T absar;
  real_T r;
  real_T absbr;
  real_T bi;
  real_T ai;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  v = static_cast<int32_T>(threadId);
  if (v < 512) {
    iv0 = (v * 511 + v) + 1;
    y[v] = CC[iv0 - 1];
    for (i = 0; i < 511; i++) {
      xpageoffset = (iv0 + i) + 1;
      if ((static_cast<int32_T>(isnan(CC[xpageoffset - 1].re))) ||
          (static_cast<int32_T>(isnan(CC[xpageoffset - 1].im)))) {
        p = false;
      } else if ((static_cast<int32_T>(isnan(y[v].re))) || (static_cast<int32_T>
                  (isnan(y[v].im)))) {
        p = true;
      } else {
        if ((static_cast<int32_T>(fabs(y[v].re) > 8.9884656743115785E+307)) || (
             static_cast<int32_T>(fabs(y[v].im) > 8.9884656743115785E+307))) {
          SCALEA = true;
        } else {
          SCALEA = false;
        }

        if ((static_cast<int32_T>(fabs(CC[xpageoffset - 1].re) >
              8.9884656743115785E+307)) || (static_cast<int32_T>(fabs
              (CC[xpageoffset - 1].im) > 8.9884656743115785E+307))) {
          p = true;
        } else {
          p = false;
        }

        if ((static_cast<int32_T>(SCALEA)) || (static_cast<int32_T>(p))) {
          x = b_rt_hypotd_snf(y[v].re / 2.0, y[v].im / 2.0);
          br = b_rt_hypotd_snf(CC[xpageoffset - 1].re / 2.0, CC[xpageoffset - 1]
                               .im / 2.0);
        } else {
          x = b_rt_hypotd_snf(y[v].re, y[v].im);
          br = b_rt_hypotd_snf(CC[xpageoffset - 1].re, CC[xpageoffset - 1].im);
        }

        if (x == br) {
          absar = fabs(y[v].re);
          r = fabs(y[v].im);
          absbr = fabs(CC[xpageoffset - 1].re);
          bi = fabs(CC[xpageoffset - 1].im);
          if (absar > r) {
            ai = absar;
            absar = r;
          } else {
            ai = r;
          }

          if (absbr > bi) {
            r = absbr;
            absbr = bi;
          } else {
            r = bi;
          }

          if (ai > r) {
            if (absar < absbr) {
              x = ai - r;
              br = (absar / 2.0 + absbr / 2.0) / (ai / 2.0 + r / 2.0) * (absbr -
                absar);
            } else {
              x = ai;
              br = r;
            }
          } else if (ai < r) {
            if (absar > absbr) {
              br = r - ai;
              x = (absar / 2.0 + absbr / 2.0) / (ai / 2.0 + r / 2.0) * (absar -
                absbr);
            } else {
              x = ai;
              br = r;
            }
          } else {
            x = absar;
            br = absbr;
          }

          if (x == br) {
            x = b_rt_atan2d_snf(y[v].im, y[v].re);
            br = b_rt_atan2d_snf(CC[xpageoffset - 1].im, CC[xpageoffset - 1].re);
            if (x == br) {
              br = CC[xpageoffset - 1].re;
              bi = CC[xpageoffset - 1].im;
              if (x > 0.78539816339744828) {
                if (x > 2.3561944901923448) {
                  x = -y[v].im;
                  br = -bi;
                } else {
                  x = -y[v].re;
                  br = -br;
                }
              } else if (x > -0.78539816339744828) {
                x = y[v].im;
                br = bi;
              } else if (x > -2.3561944901923448) {
                x = y[v].re;
              } else {
                x = -y[v].im;
                br = -bi;
              }

              if (x == br) {
                x = 0.0;
                br = 0.0;
              }
            }
          }
        }

        p = (x < br);
      }

      if (p) {
        y[v] = CC[xpageoffset - 1];
        idx[v] = static_cast<int16_T>(i + 2);
      }
    }
  }
}

static __global__ __launch_bounds__(512, 1) void bd_dftregistration_coderGPU_ker
  (const creal_T *y, const creal_T buf2ft[262144], creal_T Greg[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    Greg[i].re = buf2ft[i].re * y->re - buf2ft[i].im * y->im;
    Greg[i].im = buf2ft[i].re * y->im + buf2ft[i].im * y->re;
  }
}

static __global__ __launch_bounds__(512, 1) void c_dftregistration_coderGPU_kern
  (creal_T CC[1048576])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 1048576) {
    /*  First upsample by a factor of 2 to obtain initial estimate */
    /*  Embed Fourier data in a 2x larger array */
    CC[i].re = 0.0;
    CC[i].im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void cb_dftregistration_coderGPU_ker
  (const int16_T x[512], creal_T y[512])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    y[i].re = static_cast<real_T>(x[i]);
    y[i].im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void cc_dftregistration_coderGPU_ker
  (const creal_T buf2ft[262144], const creal_T buf1ft[262144], real_T absb1
   [262144], real_T absb2[262144])
{
  uint32_T threadId;
  real_T r;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 262144) {
    r = b_rt_hypotd_snf(buf1ft[iv0].re, buf1ft[iv0].im);
    absb2[iv0] = b_rt_hypotd_snf(buf2ft[iv0].re, buf2ft[iv0].im);
    r *= r;
    absb1[iv0] = r;
  }
}

static __global__ __launch_bounds__(512, 1) void cd_dftregistration_coderGPU_ker
  (const int16_T b[512], int16_T Nr[262144], int16_T Nc[262144])
{
  uint32_T threadId;
  int32_T i;
  int32_T xpageoffset;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId % 512U);
  xpageoffset = static_cast<int32_T>((threadId - static_cast<uint32_T>(i)) /
    512U);
  if (xpageoffset < 512) {
    Nc[i + (xpageoffset << 9)] = b[xpageoffset];
    Nr[i + (xpageoffset << 9)] = b[i];
  }
}

static void cufftEnsureDestruction()
{
  if (cufftGlobalHandlePtr != NULL) {
    hipfftDestroy(*cufftGlobalHandlePtr);
    cufftGlobalHandlePtr = NULL;
    cufftGlobalSize = 0;
    cufftGlobalBatch = 0;
    cufftGlobalType = HIPFFT_R2C;
    cufftGlobalDist = 0;
  }
}

static void cufftEnsureInitialization(int32_T nelem, hipfftType_t type, int32_T
  batch, int32_T idist)
{
  if ((cufftGlobalDist != idist) || (cufftGlobalType != type) ||
      (cufftGlobalSize != nelem) || (cufftGlobalBatch != batch)) {
    if (cufftGlobalHandlePtr != NULL) {
      cufftEnsureDestruction();
    }

    cufftGlobalHandlePtr = &cufftGlobalHandle;
    hipfftPlanMany(cufftGlobalHandlePtr, 1U, &nelem, &idist, 1, idist, &nelem, 1,
                  nelem, type, batch);
    cufftGlobalSize = nelem;
    cufftGlobalBatch = batch;
    cufftGlobalType = type;
    cufftGlobalDist = idist;
  }
}

static __global__ __launch_bounds__(512, 1) void d_dftregistration_coderGPU_kern
  (const creal_T buf1ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    CC[i] = buf1ft[i];
  }
}

static __global__ __launch_bounds__(512, 1) void db_dftregistration_coderGPU_ker
  (const emxArray_creal_T *z, emxArray_creal_T *kernr, creal_T y[512])
{
  uint32_T threadId;
  int32_T xpageoffset;
  int32_T iv0;
  real_T bi;
  int32_T i;
  real_T r;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 32767U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 32767U);
  if (i < 512) {
    xpageoffset = z->size[0] - 1;
    if (iv0 <= xpageoffset) {
      bi = z->data[iv0].re * y[i].re - z->data[iv0].im * y[i].im;
      r = z->data[iv0].re * y[i].im + z->data[iv0].im * y[i].re;
      kernr->data[iv0 + kernr->size[0] * i].re = bi;
      kernr->data[iv0 + kernr->size[0] * i].im = r;
    }
  }
}

static __global__ __launch_bounds__(32, 1) void dc_dftregistration_coderGPU_ker(
  const real_T absb1[262144], real_T *absar)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *absar = absb1[0];
  }
}

static __global__ __launch_bounds__(512, 1) void dd_dftregistration_coderGPU_ker
  (const int16_T Nc[262144], const real_T col_shift, const int16_T Nr[262144],
   const real_T *row_shift, creal_T Greg[262144])
{
  uint32_T threadId;
  real_T r;
  int32_T i;
  real_T bi;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    r = *row_shift * static_cast<real_T>(Nr[i]) / 512.0;
    bi = col_shift * static_cast<real_T>(Nc[i]) / 512.0;
    Greg[i].re = (r - bi) * 0.0;
    Greg[i].im = (r - bi) * 6.2831853071795862;
  }
}

static __global__ __launch_bounds__(512, 1) void e_dftregistration_coderGPU_kern
  (const creal_T buf2ft[262144], creal_T x[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    x[i] = buf2ft[i];
  }
}

static __global__ __launch_bounds__(32, 1) void eb_dftregistration_coderGPU_ker(
  const emxArray_creal_T *kernr, int32_T *nx)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *nx = kernr->size[0] << 9;
  }
}

static __global__ __launch_bounds__(512, 1) void ec_dftregistration_coderGPU_ker
  (const real_T absb1[262144], real_T *absar)
{
  uint32_T idx;
  real_T tmpRed0;
  uint32_T threadStride;
  uint32_T threadId;
  uint32_T thBlkId;
  uint32_T mask;
  uint32_T numActiveThreads;
  uint32_T numActiveWarps;
  uint32_T blockStride;
  uint32_T m;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  numActiveThreads = blockStride;
  if (mwIsLastBlock()) {
    m = 262143U % blockStride;
    if (m > 0U) {
      numActiveThreads = m;
    }
  }

  numActiveWarps = ((numActiveThreads + warpSize) - 1U) / warpSize;
  if (threadId <= 262142U) {
    tmpRed0 = absb1[static_cast<int32_T>(threadId) + 1];
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 262142U);
  for (idx = threadId + threadStride; idx <= 262142U; idx += threadStride) {
    tmpRed0 += absb1[static_cast<int32_T>(idx) + 1];
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, numActiveWarps);
  if (thBlkId == 0U) {
    atomicOpreal_T(&absar[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(512, 1) void ed_dftregistration_coderGPU_ker
  (creal_T Greg[262144])
{
  uint32_T threadId;
  int32_T iv0;
  creal_T x;
  real_T r;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 262144) {
    if (Greg[iv0].im == 0.0) {
      x.re = exp(Greg[iv0].re);
      x.im = 0.0;
    } else if ((static_cast<int32_T>((static_cast<int32_T>(isinf(Greg[iv0].im)))
      && (static_cast<int32_T>(isinf(Greg[iv0].re))))) && (static_cast<int32_T>
                (Greg[iv0].re < 0.0))) {
      x.re = 0.0;
      x.im = 0.0;
    } else {
      r = exp(Greg[iv0].re / 2.0);
      x.re = r * (r * cos(Greg[iv0].im));
      x.im = r * (r * sin(Greg[iv0].im));
    }

    Greg[iv0] = x;
  }
}

static __global__ __launch_bounds__(512, 1) void f_dftregistration_coderGPU_kern
  (creal_T CC[262144], creal_T b_CC[1048576], creal_T x[262144])
{
  uint32_T threadId;
  real_T r;
  int32_T iv0;
  int32_T i;
  real_T bi;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 512U);
  if (i < 512) {
    r = x[iv0 + (i << 9)].re;
    bi = -x[iv0 + (i << 9)].im;
    b_CC[(iv0 + ((i + 256) << 10)) + 256].re = CC[iv0 + (i << 9)].re * r -
      CC[iv0 + (i << 9)].im * bi;
    b_CC[(iv0 + ((i + 256) << 10)) + 256].im = CC[iv0 + (i << 9)].re * bi +
      CC[iv0 + (i << 9)].im * r;
  }
}

static __global__ __launch_bounds__(1024, 1) void
  fb_dftregistration_coderGPU_ker(const int32_T *nx, emxArray_creal_T *kernr)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  creal_T x;
  real_T r;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(*nx - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx);
    if (kernr->data[iv0].im == 0.0) {
      x.re = exp(kernr->data[iv0].re);
      x.im = 0.0;
    } else if ((static_cast<int32_T>((static_cast<int32_T>(isinf(kernr->data[iv0]
        .im))) && (static_cast<int32_T>(isinf(kernr->data[iv0].re))))) && (
                static_cast<int32_T>(kernr->data[iv0].re < 0.0))) {
      x.re = 0.0;
      x.im = 0.0;
    } else {
      r = exp(kernr->data[iv0].re / 2.0);
      x.re = r * (r * cos(kernr->data[iv0].im));
      x.im = r * (r * sin(kernr->data[iv0].im));
    }

    kernr->data[iv0] = x;
  }
}

static __global__ __launch_bounds__(512, 1) void fc_dftregistration_coderGPU_ker
  (real_T absb2[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    absb2[i] *= absb2[i];
  }
}

static __global__ __launch_bounds__(32, 1) void fd_dftregistration_coderGPU_ker(
  const real_T *tmp_im, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = *tmp_im * 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void g_dftregistration_coderGPU_kern
  (creal_T CC[1048576], creal_T y[1048576])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 1024U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 1024U);
  if (i < 1024) {
    y[iv0 + (i << 10)] = CC[i + (iv0 << 10)];
  }
}

static __global__ __launch_bounds__(1024, 1) void
  gb_dftregistration_coderGPU_ker(const emxArray_creal_T *kernr,
  emxArray_creal_T *y)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(kernr->size[0] * 512 - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    y->data[i].re = 0.0;
    y->data[i].im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void gc_dftregistration_coderGPU_ker(
  const real_T absb2[262144], real_T *tmp_re)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_re = absb2[0];
  }
}

static __global__ __launch_bounds__(32, 1) void gd_dftregistration_coderGPU_ker(
  const creal_T *y, real_T *tmp_re)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_re = exp(y->re / 2.0);
  }
}

static void gpuEmxFree_creal_T(emxArray_creal_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

static void gpuEmxFree_int16_T(emxArray_int16_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

static void gpuEmxFree_real_T(emxArray_real_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

static void gpuEmxFree_uint16_T(emxArray_uint16_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

static void gpuEmxMemcpyCpuToGpu_creal_T(const emxArray_creal_T *cpu,
  emxArray_creal_T *inter, emxArray_creal_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  int32_T allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int32_T));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(creal_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(creal_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32ULL, hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_int16_T(const emxArray_int16_T *cpu,
  emxArray_int16_T *inter, emxArray_int16_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  int32_T allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int32_T));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(int16_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(int16_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32ULL, hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *inter, emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  int32_T allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int32_T));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(real_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32ULL, hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_uint16_T(const emxArray_uint16_T *cpu,
  emxArray_uint16_T *inter, emxArray_uint16_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  int32_T allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int32_T));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(uint16_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(uint16_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int32_T),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32ULL, hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu, emxArray_creal_T
  *inter)
{
  int32_T actualSize;
  int32_T i;
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  hipMemcpy(cpu->data, inter->data, actualSize * sizeof(creal_T),
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu->size, inter->size, inter->numDimensions * sizeof(int32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *inter)
{
  int32_T actualSize;
  int32_T i;
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  hipMemcpy(cpu->data, inter->data, actualSize * sizeof(real_T),
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu->size, inter->size, inter->numDimensions * sizeof(int32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxMemcpyGpuToCpu_uint16_T(emxArray_uint16_T *cpu,
  emxArray_uint16_T *inter)
{
  int32_T actualSize;
  int32_T i;
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  hipMemcpy(cpu->data, inter->data, actualSize * sizeof(uint16_T),
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu->size, inter->size, inter->numDimensions * sizeof(int32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_creal_T(emxArray_creal_T *inter)
{
  memset(inter, 0, sizeof(emxArray_creal_T));
}

static void gpuEmxReset_int16_T(emxArray_int16_T *inter)
{
  memset(inter, 0, sizeof(emxArray_int16_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *inter)
{
  memset(inter, 0, sizeof(emxArray_real_T));
}

static void gpuEmxReset_uint16_T(emxArray_uint16_T *inter)
{
  memset(inter, 0, sizeof(emxArray_uint16_T));
}

static __global__ __launch_bounds__(512, 1) void h_dftregistration_coderGPU_kern
  (creal_T CC[1048576])
{
  uint32_T threadId;
  int32_T i;
  real_T bi;
  real_T r;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 1048576) {
    if (CC[i].im == 0.0) {
      bi = CC[i].re / 1024.0;
      r = 0.0;
    } else if (CC[i].re == 0.0) {
      bi = 0.0;
      r = CC[i].im / 1024.0;
    } else {
      bi = CC[i].re / 1024.0;
      r = CC[i].im / 1024.0;
    }

    CC[i].re = bi;
    CC[i].im = r;
  }
}

static __global__ __launch_bounds__(1024, 1) void
  hb_dftregistration_coderGPU_ker(const emxArray_creal_T *kernc, const
  emxArray_creal_T *y, emxArray_creal_T *out)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(y->size[0] * kernc->size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    out->data[i].re = 0.0;
    out->data[i].im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void hc_dftregistration_coderGPU_ker
  (const real_T absb2[262144], real_T *tmp_re)
{
  uint32_T idx;
  real_T tmpRed0;
  uint32_T threadStride;
  uint32_T threadId;
  uint32_T thBlkId;
  uint32_T mask;
  uint32_T numActiveThreads;
  uint32_T numActiveWarps;
  uint32_T blockStride;
  uint32_T m;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  numActiveThreads = blockStride;
  if (mwIsLastBlock()) {
    m = 262143U % blockStride;
    if (m > 0U) {
      numActiveThreads = m;
    }
  }

  numActiveWarps = ((numActiveThreads + warpSize) - 1U) / warpSize;
  if (threadId <= 262142U) {
    tmpRed0 = absb2[static_cast<int32_T>(threadId) + 1];
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 262142U);
  for (idx = threadId + threadStride; idx <= 262142U; idx += threadStride) {
    tmpRed0 += absb2[static_cast<int32_T>(idx) + 1];
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, numActiveWarps);
  if (thBlkId == 0U) {
    atomicOpreal_T(&tmp_re[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(32, 1) void hd_dftregistration_coderGPU_ker(
  const real_T *tmp_im, const real_T *tmp_re, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = *tmp_re * (*tmp_re * cos(*tmp_im));
    y->im = *tmp_re * (*tmp_re * sin(*tmp_im));
  }
}

static __global__ __launch_bounds__(512, 1) void i_dftregistration_coderGPU_kern
  (creal_T CC[1048576], creal_T y[1048576])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 1024U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 1024U);
  if (i < 1024) {
    y[iv0 + (i << 10)] = CC[i + (iv0 << 10)];
  }
}

static __global__ __launch_bounds__(1024, 1) void
  ib_dftregistration_coderGPU_ker(const real_T *absar, const emxArray_creal_T
  *out, emxArray_creal_T *CC)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  real_T r;
  int32_T i;
  real_T ai;
  real_T bi;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(out->size[0] * out->size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    r = out->data[i].re;
    ai = -out->data[i].im;
    if (ai == 0.0) {
      bi = r / *absar;
      r = 0.0;
    } else if (r == 0.0) {
      bi = 0.0;
      r = ai / *absar;
    } else {
      bi = r / *absar;
      r = ai / *absar;
    }

    CC->data[i].re = bi;
    CC->data[i].im = r;
  }
}

static __global__ __launch_bounds__(32, 1) void ic_dftregistration_coderGPU_ker(
  const real_T col_shift, const real_T *row_shift, const real_T *tmp_im, const
  real_T Ma, const real_T absbr, real_T dv1[4])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    dv1[0] = sqrt(b_rt_hypotd_snf(1.0 - absbr, 0.0 - Ma));
    dv1[1] = *tmp_im;
    dv1[2] = *row_shift;
    dv1[3] = col_shift;
  }
}

static __global__ __launch_bounds__(32, 1) void id_dftregistration_coderGPU_ker(
  const creal_T *y, real_T *tmp_re)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_re = y->re;
  }
}

static __global__ __launch_bounds__(512, 1) void j_dftregistration_coderGPU_kern
  (creal_T CC[1048576])
{
  uint32_T threadId;
  int32_T i;
  real_T bi;
  real_T r;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 1048576) {
    if (CC[i].im == 0.0) {
      bi = CC[i].re / 1024.0;
      r = 0.0;
    } else if (CC[i].re == 0.0) {
      bi = 0.0;
      r = CC[i].im / 1024.0;
    } else {
      bi = CC[i].re / 1024.0;
      r = CC[i].im / 1024.0;
    }

    CC[i].re = bi;
    CC[i].im = r;
  }
}

static __global__ __launch_bounds__(32, 1) void jb_dftregistration_coderGPU_ker(
  const emxArray_creal_T *ex, uint16_T uv[2])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 2) {
    uv[i] = static_cast<uint16_T>(ex->size[i]);
  }
}

static __global__ __launch_bounds__(32, 1) void jc_dftregistration_coderGPU_ker(
  const real_T dv1[4], emxArray_real_T *output)
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 4) {
    output->data[i] = dv1[i];
  }
}

static __global__ __launch_bounds__(32, 1) void jd_dftregistration_coderGPU_ker(
  const real_T *tmp_re, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = exp(*tmp_re);
    y->im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void k_dftregistration_coderGPU_kern
  (int16_T idx[1024])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 1024) {
    /*  Calculate cross-correlation */
    idx[i] = static_cast<int16_T>(1);
  }
}

static __global__ __launch_bounds__(1024, 1) void
  kb_dftregistration_coderGPU_ker(const uint16_T uv[2], emxArray_uint16_T *idx)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T b_idx;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(static_cast<int32_T>(uv[1]) - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint32_T>(loopEnd); b_idx +=
       threadStride) {
    i = static_cast<int32_T>(b_idx);
    idx->data[i] = static_cast<uint16_T>(1U);
  }
}

static __global__ __launch_bounds__(512, 1) void kc_dftregistration_coderGPU_ker
  (const creal_T buf2ft[262144], const creal_T buf1ft[262144], creal_T CC[262144],
   real_T absb2[262144], real_T absb1[262144])
{
  uint32_T threadId;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 262144) {
    absb1[iv0] = b_rt_hypotd_snf(buf1ft[iv0].re, buf1ft[iv0].im);
    absb2[iv0] = b_rt_hypotd_snf(buf2ft[iv0].re, buf2ft[iv0].im);
    CC[iv0].re = buf1ft[iv0].re * buf2ft[iv0].re - buf1ft[iv0].im * -buf2ft[iv0]
      .im;
    CC[iv0].im = buf1ft[iv0].re * -buf2ft[iv0].im + buf1ft[iv0].im * buf2ft[iv0]
      .re;
  }
}

static __global__ __launch_bounds__(512, 1) void kd_dftregistration_coderGPU_ker
  (const creal_T *y, const creal_T buf2ft[262144], creal_T Greg[262144])
{
  uint32_T threadId;
  real_T r;
  int32_T i;
  real_T bi;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    r = buf2ft[i].re * Greg[i].re - buf2ft[i].im * Greg[i].im;
    bi = buf2ft[i].re * Greg[i].im + buf2ft[i].im * Greg[i].re;
    Greg[i].re = r * y->re - bi * y->im;
    Greg[i].im = r * y->im + bi * y->re;
  }
}

static __global__ __launch_bounds__(512, 1) void l_dftregistration_coderGPU_kern
  (int16_T idx[1024], creal_T CC[1048576], creal_T ex[1024])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T v;
  int32_T i;
  int32_T xpageoffset;
  boolean_T p;
  boolean_T SCALEA;
  real_T x;
  real_T br;
  real_T absar;
  real_T r;
  real_T absbr;
  real_T bi;
  real_T ai;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  v = static_cast<int32_T>(threadId);
  if (v < 1024) {
    iv0 = (v * 1023 + v) + 1;
    ex[v] = CC[iv0 - 1];
    for (i = 0; i < 1023; i++) {
      xpageoffset = (iv0 + i) + 1;
      if ((static_cast<int32_T>(isnan(CC[xpageoffset - 1].re))) ||
          (static_cast<int32_T>(isnan(CC[xpageoffset - 1].im)))) {
        p = false;
      } else if ((static_cast<int32_T>(isnan(ex[v].re))) || (static_cast<int32_T>
                  (isnan(ex[v].im)))) {
        p = true;
      } else {
        if ((static_cast<int32_T>(fabs(ex[v].re) > 8.9884656743115785E+307)) ||
            (static_cast<int32_T>(fabs(ex[v].im) > 8.9884656743115785E+307))) {
          SCALEA = true;
        } else {
          SCALEA = false;
        }

        if ((static_cast<int32_T>(fabs(CC[xpageoffset - 1].re) >
              8.9884656743115785E+307)) || (static_cast<int32_T>(fabs
              (CC[xpageoffset - 1].im) > 8.9884656743115785E+307))) {
          p = true;
        } else {
          p = false;
        }

        if ((static_cast<int32_T>(SCALEA)) || (static_cast<int32_T>(p))) {
          x = b_rt_hypotd_snf(ex[v].re / 2.0, ex[v].im / 2.0);
          br = b_rt_hypotd_snf(CC[xpageoffset - 1].re / 2.0, CC[xpageoffset - 1]
                               .im / 2.0);
        } else {
          x = b_rt_hypotd_snf(ex[v].re, ex[v].im);
          br = b_rt_hypotd_snf(CC[xpageoffset - 1].re, CC[xpageoffset - 1].im);
        }

        if (x == br) {
          absar = fabs(ex[v].re);
          r = fabs(ex[v].im);
          absbr = fabs(CC[xpageoffset - 1].re);
          bi = fabs(CC[xpageoffset - 1].im);
          if (absar > r) {
            ai = absar;
            absar = r;
          } else {
            ai = r;
          }

          if (absbr > bi) {
            r = absbr;
            absbr = bi;
          } else {
            r = bi;
          }

          if (ai > r) {
            if (absar < absbr) {
              x = ai - r;
              br = (absar / 2.0 + absbr / 2.0) / (ai / 2.0 + r / 2.0) * (absbr -
                absar);
            } else {
              x = ai;
              br = r;
            }
          } else if (ai < r) {
            if (absar > absbr) {
              br = r - ai;
              x = (absar / 2.0 + absbr / 2.0) / (ai / 2.0 + r / 2.0) * (absar -
                absbr);
            } else {
              x = ai;
              br = r;
            }
          } else {
            x = absar;
            br = absbr;
          }

          if (x == br) {
            x = b_rt_atan2d_snf(ex[v].im, ex[v].re);
            br = b_rt_atan2d_snf(CC[xpageoffset - 1].im, CC[xpageoffset - 1].re);
            if (x == br) {
              br = CC[xpageoffset - 1].re;
              bi = CC[xpageoffset - 1].im;
              if (x > 0.78539816339744828) {
                if (x > 2.3561944901923448) {
                  x = -ex[v].im;
                  br = -bi;
                } else {
                  x = -ex[v].re;
                  br = -br;
                }
              } else if (x > -0.78539816339744828) {
                x = ex[v].im;
                br = bi;
              } else if (x > -2.3561944901923448) {
                x = ex[v].re;
              } else {
                x = -ex[v].im;
                br = -bi;
              }

              if (x == br) {
                x = 0.0;
                br = 0.0;
              }
            }
          }
        }

        p = (x < br);
      }

      if (p) {
        ex[v] = CC[xpageoffset - 1];
        idx[v] = static_cast<int16_T>(i + 2);
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void
  lb_dftregistration_coderGPU_ker(const emxArray_creal_T *CC, const int32_T
  vstride, const int32_T npages, emxArray_uint16_T *idx, emxArray_creal_T *ex)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T b_idx;
  int32_T iv0;
  int32_T v;
  int32_T i;
  int32_T xpageoffset;
  boolean_T p;
  boolean_T SCALEA;
  real_T x;
  real_T br;
  real_T absar;
  real_T r;
  real_T absbr;
  real_T bi;
  real_T ai;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(npages - 1);
  for (b_idx = threadId; b_idx <= static_cast<uint32_T>(loopEnd); b_idx +=
       threadStride) {
    v = static_cast<int32_T>(b_idx);
    iv0 = (v * (vstride - 1) + v) + 1;
    ex->data[v] = CC->data[iv0 - 1];
    for (i = 0; i <= vstride - 2; i++) {
      xpageoffset = (iv0 + i) + 1;
      if ((static_cast<int32_T>(isnan(CC->data[xpageoffset - 1].re))) || (
           static_cast<int32_T>(isnan(CC->data[xpageoffset - 1].im)))) {
        p = false;
      } else if ((static_cast<int32_T>(isnan(ex->data[v].re))) ||
                 (static_cast<int32_T>(isnan(ex->data[v].im)))) {
        p = true;
      } else {
        if ((static_cast<int32_T>(fabs(ex->data[v].re) > 8.9884656743115785E+307))
            || (static_cast<int32_T>(fabs(ex->data[v].im) >
              8.9884656743115785E+307))) {
          SCALEA = true;
        } else {
          SCALEA = false;
        }

        if ((static_cast<int32_T>(fabs(CC->data[xpageoffset - 1].re) >
              8.9884656743115785E+307)) || (static_cast<int32_T>(fabs(CC->
               data[xpageoffset - 1].im) > 8.9884656743115785E+307))) {
          p = true;
        } else {
          p = false;
        }

        if ((static_cast<int32_T>(SCALEA)) || (static_cast<int32_T>(p))) {
          x = b_rt_hypotd_snf(ex->data[v].re / 2.0, ex->data[v].im / 2.0);
          br = b_rt_hypotd_snf(CC->data[xpageoffset - 1].re / 2.0, CC->
                               data[xpageoffset - 1].im / 2.0);
        } else {
          x = b_rt_hypotd_snf(ex->data[v].re, ex->data[v].im);
          br = b_rt_hypotd_snf(CC->data[xpageoffset - 1].re, CC->
                               data[xpageoffset - 1].im);
        }

        if (x == br) {
          absar = fabs(ex->data[v].re);
          r = fabs(ex->data[v].im);
          absbr = fabs(CC->data[xpageoffset - 1].re);
          bi = fabs(CC->data[xpageoffset - 1].im);
          if (absar > r) {
            ai = absar;
            absar = r;
          } else {
            ai = r;
          }

          if (absbr > bi) {
            r = absbr;
            absbr = bi;
          } else {
            r = bi;
          }

          if (ai > r) {
            if (absar < absbr) {
              x = ai - r;
              br = (absar / 2.0 + absbr / 2.0) / (ai / 2.0 + r / 2.0) * (absbr -
                absar);
            } else {
              x = ai;
              br = r;
            }
          } else if (ai < r) {
            if (absar > absbr) {
              br = r - ai;
              x = (absar / 2.0 + absbr / 2.0) / (ai / 2.0 + r / 2.0) * (absar -
                absbr);
            } else {
              x = ai;
              br = r;
            }
          } else {
            x = absar;
            br = absbr;
          }

          if (x == br) {
            x = b_rt_atan2d_snf(ex->data[v].im, ex->data[v].re);
            br = b_rt_atan2d_snf(CC->data[xpageoffset - 1].im, CC->
                                 data[xpageoffset - 1].re);
            if (x == br) {
              r = ex->data[v].re;
              ai = ex->data[v].im;
              br = CC->data[xpageoffset - 1].re;
              bi = CC->data[xpageoffset - 1].im;
              if (x > 0.78539816339744828) {
                if (x > 2.3561944901923448) {
                  r = -ai;
                  br = -bi;
                } else {
                  r = -r;
                  br = -br;
                }
              } else if (x > -0.78539816339744828) {
                r = ai;
                br = bi;
              } else {
                if (!static_cast<int32_T>(x > -2.3561944901923448)) {
                  r = -ai;
                  br = -bi;
                }
              }

              x = r;
              if (r == br) {
                x = 0.0;
                br = 0.0;
              }
            }
          }
        }

        p = (x < br);
      }

      if (p) {
        ex->data[v] = CC->data[xpageoffset - 1];
        idx->data[v] = static_cast<uint16_T>(i + 2);
      }
    }
  }
}

static __global__ __launch_bounds__(512, 1) void lc_dftregistration_coderGPU_ker
  (creal_T y[512], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T xpageoffset;
  int32_T i;
  real_T bi;
  real_T r;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    xpageoffset = i << 9;
    bi = CC[xpageoffset].re;
    r = CC[xpageoffset].im;
    for (iv0 = 0; iv0 < 511; iv0++) {
      bi += CC[(xpageoffset + iv0) + 1].re;
      r += CC[(xpageoffset + iv0) + 1].im;
    }

    y[i].re = bi;
    y[i].im = r;
  }
}

static __global__ __launch_bounds__(32, 1) void m_dftregistration_coderGPU_kern(
  const int16_T idx[1024], const int32_T *nx, real_T *row_shift, creal_T CC
  [1048576], creal_T *CCmax)
{
  uint32_T threadId;
  int32_T tmpIdx;
  int32_T b_idx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    /*  Obtain shift in original pixel grid from the position of the */
    /*  crosscorrelation peak  */
    *CCmax = CC[(static_cast<int32_T>(idx[*nx]) + (*nx << 10)) - 1];
    if (static_cast<int32_T>(idx[*nx]) > 512) {
      b_idx = static_cast<int32_T>(idx[*nx]) - 1025;
    } else {
      b_idx = static_cast<int32_T>(idx[*nx]) - 1;
    }

    *row_shift = static_cast<real_T>(b_idx) / 2.0;
  }
}

static __global__ __launch_bounds__(32, 1) void mb_dftregistration_coderGPU_ker(
  const emxArray_creal_T *CC, const int32_T *nx, const emxArray_uint16_T *idx,
  creal_T *CCmax)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *CCmax = CC->data[(static_cast<int32_T>(idx->data[*nx]) + CC->size[0] * *nx)
      - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void mc_dftregistration_coderGPU_ker
  (creal_T *CCmax)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    CCmax->re = 0.0;
    CCmax->im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void n_dftregistration_coderGPU_kern
  (const creal_T buf1ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    CC[i].re = buf1ft[i].re * buf1ft[i].re - buf1ft[i].im * -buf1ft[i].im;
    CC[i].im = buf1ft[i].re * -buf1ft[i].im + buf1ft[i].im * buf1ft[i].re;
  }
}

static __global__ __launch_bounds__(512, 1) void nb_dftregistration_coderGPU_ker
  (const creal_T buf1ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    CC[i].re = buf1ft[i].re * buf1ft[i].re - buf1ft[i].im * -buf1ft[i].im;
    CC[i].im = buf1ft[i].re * -buf1ft[i].im + buf1ft[i].im * buf1ft[i].re;
  }
}

static __global__ __launch_bounds__(512, 1) void nc_dftregistration_coderGPU_ker
  (real_T absb1[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    absb1[i] *= absb1[i];
  }
}

static __global__ __launch_bounds__(512, 1) void o_dftregistration_coderGPU_kern
  (creal_T y[512], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T xpageoffset;
  int32_T i;
  real_T bi;
  real_T r;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    xpageoffset = i << 9;
    bi = CC[xpageoffset].re;
    r = CC[xpageoffset].im;
    for (iv0 = 0; iv0 < 511; iv0++) {
      bi += CC[(xpageoffset + iv0) + 1].re;
      r += CC[(xpageoffset + iv0) + 1].im;
    }

    y[i].re = bi;
    y[i].im = r;
  }
}

static __global__ __launch_bounds__(512, 1) void ob_dftregistration_coderGPU_ker
  (creal_T y[512], creal_T kernr[512])
{
  uint32_T threadId;
  creal_T x;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 512) {
    x.re = 1.0;
    x.im = 0.0;
    kernr[iv0] = x;
    y[iv0].re = 0.0;
    y[iv0].im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void oc_dftregistration_coderGPU_ker(
  const real_T absb1[262144], real_T *tmp_re)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_re = absb1[0];
  }
}

static __global__ __launch_bounds__(32, 1) void p_dftregistration_coderGPU_kern
  (creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = 0.0;
    y->im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void pb_dftregistration_coderGPU_ker
  (creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = 0.0;
    y->im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void pc_dftregistration_coderGPU_ker
  (const real_T absb1[262144], real_T *tmp_re)
{
  uint32_T idx;
  real_T tmpRed0;
  uint32_T threadStride;
  uint32_T threadId;
  uint32_T thBlkId;
  uint32_T mask;
  uint32_T numActiveThreads;
  uint32_T numActiveWarps;
  uint32_T blockStride;
  uint32_T m;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  numActiveThreads = blockStride;
  if (mwIsLastBlock()) {
    m = 262143U % blockStride;
    if (m > 0U) {
      numActiveThreads = m;
    }
  }

  numActiveWarps = ((numActiveThreads + warpSize) - 1U) / warpSize;
  if (threadId <= 262142U) {
    tmpRed0 = absb1[static_cast<int32_T>(threadId) + 1];
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 262142U);
  for (idx = threadId + threadStride; idx <= 262142U; idx += threadStride) {
    tmpRed0 += absb1[static_cast<int32_T>(idx) + 1];
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, numActiveWarps);
  if (thBlkId == 0U) {
    atomicOpreal_T(&tmp_re[0], tmpRed0);
  }
}

static __global__ __launch_bounds__(512, 1) void q_dftregistration_coderGPU_kern
  (const creal_T buf2ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    CC[i].re = buf2ft[i].re * buf2ft[i].re - buf2ft[i].im * -buf2ft[i].im;
    CC[i].im = buf2ft[i].re * -buf2ft[i].im + buf2ft[i].im * buf2ft[i].re;
  }
}

static __global__ __launch_bounds__(512, 1) void qb_dftregistration_coderGPU_ker
  (const creal_T buf2ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    CC[i].re = buf2ft[i].re * buf2ft[i].re - buf2ft[i].im * -buf2ft[i].im;
    CC[i].im = buf2ft[i].re * -buf2ft[i].im + buf2ft[i].im * buf2ft[i].re;
  }
}

static __global__ __launch_bounds__(512, 1) void qc_dftregistration_coderGPU_ker
  (real_T absb2[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    absb2[i] *= absb2[i];
  }
}

static __global__ __launch_bounds__(512, 1) void r_dftregistration_coderGPU_kern
  (creal_T y[512], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T xpageoffset;
  int32_T i;
  real_T bi;
  real_T r;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    xpageoffset = i << 9;
    bi = CC[xpageoffset].re;
    r = CC[xpageoffset].im;
    for (iv0 = 0; iv0 < 511; iv0++) {
      bi += CC[(xpageoffset + iv0) + 1].re;
      r += CC[(xpageoffset + iv0) + 1].im;
    }

    y[i].re = bi;
    y[i].im = r;
  }
}

static __global__ __launch_bounds__(512, 1) void rb_dftregistration_coderGPU_ker
  (creal_T y[512], creal_T kernr[512])
{
  uint32_T threadId;
  creal_T x;
  int32_T iv0;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 512) {
    x.re = 1.0;
    x.im = 0.0;
    kernr[iv0] = x;
    y[iv0].re = 0.0;
    y[iv0].im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void rc_dftregistration_coderGPU_ker(
  const real_T absb2[262144], real_T *tmp_im)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_im = absb2[0];
  }
}

static real_T rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  int32_T b_u0;
  int32_T b_u1;
  if (rtIsNaN(u0) || rtIsNaN(u1)) {
    y = rtNaN;
  } else if (rtIsInf(u0) && rtIsInf(u1)) {
    if (u0 > 0.0) {
      b_u0 = 1;
    } else {
      b_u0 = -1;
    }

    if (u1 > 0.0) {
      b_u1 = 1;
    } else {
      b_u1 = -1;
    }

    y = atan2(static_cast<real_T>(b_u0), static_cast<real_T>(b_u1));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = atan2(u0, u1);
  }

  return y;
}

static real_T rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T y;
  real_T a;
  real_T b;
  a = fabs(u0);
  b = fabs(u1);
  if (a < b) {
    a /= b;
    y = b * sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * sqrt(b * b + 1.0);
  } else if (rtIsNaN(b)) {
    y = b;
  } else {
    y = a * 1.4142135623730951;
  }

  return y;
}

static real_T rt_roundd_snf(real_T u)
{
  real_T y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

static __global__ __launch_bounds__(32, 1) void s_dftregistration_coderGPU_kern
  (creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = 0.0;
    y->im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void sb_dftregistration_coderGPU_ker
  (creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = 0.0;
    y->im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void sc_dftregistration_coderGPU_ker
  (const real_T absb2[262144], real_T *tmp_im)
{
  uint32_T idx;
  real_T tmpRed0;
  uint32_T threadStride;
  uint32_T threadId;
  uint32_T thBlkId;
  uint32_T mask;
  uint32_T numActiveThreads;
  uint32_T numActiveWarps;
  uint32_T blockStride;
  uint32_T m;
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  blockStride = static_cast<uint32_T>(mwGetThreadsPerBlock());
  tmpRed0 = 0.0;
  numActiveThreads = blockStride;
  if (mwIsLastBlock()) {
    m = 262143U % blockStride;
    if (m > 0U) {
      numActiveThreads = m;
    }
  }

  numActiveWarps = ((numActiveThreads + warpSize) - 1U) / warpSize;
  if (threadId <= 262142U) {
    tmpRed0 = absb2[static_cast<int32_T>(threadId) + 1];
  }

  mask = __ballot_sync(MAX_uint32_T, threadId <= 262142U);
  for (idx = threadId + threadStride; idx <= 262142U; idx += threadStride) {
    tmpRed0 += absb2[static_cast<int32_T>(idx) + 1];
  }

  tmpRed0 = workGroupReduction(tmpRed0, mask, numActiveWarps);
  if (thBlkId == 0U) {
    atomicOpreal_T(&tmp_im[0], tmpRed0);
  }
}

static __inline__ __device__ real_T shflDown2(real_T in1, uint32_T offset,
  uint32_T mask)
{
  int2 tmp;
  tmp = *(int2 *)&in1;
  tmp.x = __shfl_down_sync(mask, tmp.x, offset);
  tmp.y = __shfl_down_sync(mask, tmp.y, offset);
  return *(real_T *)&tmp;
}

static __global__ __launch_bounds__(512, 1) void t_dftregistration_coderGPU_kern
  (const creal_T buf1ft[262144], const creal_T buf2ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    CC[i].re = buf2ft[i].re * buf1ft[i].re - buf2ft[i].im * -buf1ft[i].im;
    CC[i].im = buf2ft[i].re * -buf1ft[i].im + buf2ft[i].im * buf1ft[i].re;
  }
}

static __global__ __launch_bounds__(32, 1) void tb_dftregistration_coderGPU_ker(
  const real_T col_shift, const real_T *row_shift, const real_T *tmp_im, const
  real_T CCmax_im, const real_T Ma, real_T dv1[4])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    /*  If its only one row or column the shift along that dimension has no */
    /*  effect. We set to zero. */
    dv1[0] = sqrt(b_rt_hypotd_snf(1.0 - Ma, 0.0 - CCmax_im));
    dv1[1] = *tmp_im;
    dv1[2] = *row_shift;
    dv1[3] = col_shift;
  }
}

static __global__ __launch_bounds__(32, 1) void tc_dftregistration_coderGPU_ker(
  const real_T *tmp_im, const real_T CCmax_im, const real_T ex_im, real_T dv[2])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    dv[0] = sqrt(b_rt_hypotd_snf(1.0 - ex_im, 0.0 - CCmax_im));
    dv[1] = *tmp_im;
  }
}

static __device__ real_T threadGroupReduction(real_T val, uint32_T lane,
  uint32_T mask)
{
  real_T other;
  uint32_T offset;
  uint32_T activeSize;
  activeSize = __popc(mask);
  offset = (activeSize + 1U) / 2U;
  while (activeSize > 1U) {
    other = shflDown2(val, offset, mask);
    if (lane + offset < activeSize) {
      val += other;
    }

    activeSize = offset;
    offset = (offset + 1U) / 2U;
  }

  return val;
}

static __global__ __launch_bounds__(32, 1) void u_dftregistration_coderGPU_kern(
  const int16_T yk, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->im = -6.2831853071795862 / static_cast<real_T>(yk);
  }
}

static __global__ __launch_bounds__(32, 1) void ub_dftregistration_coderGPU_ker(
  const real_T dv1[4], emxArray_real_T *output)
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 4) {
    output->data[i] = dv1[i];
  }
}

static __global__ __launch_bounds__(32, 1) void uc_dftregistration_coderGPU_ker(
  const real_T dv[2], emxArray_real_T *output)
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 2) {
    output->data[i] = dv[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void
  v_dftregistration_coderGPU_kern(const int16_T coff, const emxArray_int16_T *y,
  emxArray_creal_T *b_y)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(y->size[0] * y->size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    iv0 = static_cast<int32_T>(y->data[i]) - static_cast<int32_T>(coff);
    if (iv0 > 32767) {
      iv0 = 32767;
    } else {
      if (iv0 < -32768) {
        iv0 = -32768;
      }
    }

    b_y->data[i].re = static_cast<real_T>(iv0);
    b_y->data[i].im = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void vb_dftregistration_coderGPU_ker
  (const creal_T buf1ft[262144], const creal_T buf2ft[262144], creal_T CC[262144])
{
  uint32_T threadId;
  real_T r;
  int32_T i;
  int32_T iv0;
  real_T bi;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 512U);
  if (i < 512) {
    r = buf2ft[i + (iv0 << 9)].re;
    bi = -buf2ft[i + (iv0 << 9)].im;
    CC[iv0 + (i << 9)].re = buf1ft[i + (iv0 << 9)].re * r - buf1ft[i + (iv0 << 9)]
      .im * bi;
    CC[iv0 + (i << 9)].im = buf1ft[i + (iv0 << 9)].re * bi + buf1ft[i + (iv0 <<
      9)].im * r;
  }
}

static __global__ __launch_bounds__(32, 1) void vc_dftregistration_coderGPU_ker(
  const real_T *tmp_im, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = *tmp_im * 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void
  w_dftregistration_coderGPU_kern(const emxArray_creal_T *y, const creal_T *b_y,
  const int16_T b[512], int32_T *nx, emxArray_creal_T *kernc)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  real_T r;
  int32_T iv0;
  real_T bi;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = 512LL * (static_cast<int64_T>(*nx) + 1LL) - 1LL;
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx % 512U);
    i = static_cast<int32_T>((idx - static_cast<uint32_T>(iv0)) / 512U);
    r = static_cast<real_T>(b[iv0]) * b_y->im;
    bi = 0.0 * y->data[i].re - r * y->data[i].im;
    r = 0.0 * y->data[i].im + r * y->data[i].re;
    kernc->data[iv0 + (i << 9)].re = bi;
    kernc->data[iv0 + (i << 9)].im = r;
  }
}

static __global__ __launch_bounds__(512, 1) void wb_dftregistration_coderGPU_ker
  (creal_T b_y1[262144])
{
  uint32_T threadId;
  int32_T i;
  real_T bi;
  real_T r;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    if (b_y1[i].im == 0.0) {
      bi = b_y1[i].re / 512.0;
      r = 0.0;
    } else if (b_y1[i].re == 0.0) {
      bi = 0.0;
      r = b_y1[i].im / 512.0;
    } else {
      bi = b_y1[i].re / 512.0;
      r = b_y1[i].im / 512.0;
    }

    b_y1[i].re = bi;
    b_y1[i].im = r;
  }
}

static __global__ __launch_bounds__(32, 1) void wc_dftregistration_coderGPU_ker(
  const creal_T *y, real_T *tmp_re)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_re = exp(y->re / 2.0);
  }
}

static __device__ real_T workGroupReduction(real_T val, uint32_T mask, uint32_T
  numActiveWarps)
{
  __shared__ real_T shared[32];
  uint32_T lane;
  uint32_T widx;
  uint32_T thBlkId;
  thBlkId = static_cast<uint32_T>(mwGetThreadIndexWithinBlock());
  lane = thBlkId % warpSize;
  widx = thBlkId / warpSize;
  val = threadGroupReduction(val, lane, mask);
  if (lane == 0U) {
    shared[widx] = val;
  }

  __syncthreads();
  mask = __ballot_sync(MAX_uint32_T, lane < numActiveWarps);
  val = shared[lane];
  if (widx == 0U) {
    val = threadGroupReduction(val, lane, mask);
  }

  return val;
}

static __global__ __launch_bounds__(32, 1) void x_dftregistration_coderGPU_kern(
  const emxArray_creal_T *kernc, int32_T *nx)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *nx = kernc->size[1] << 9;
  }
}

static __global__ __launch_bounds__(512, 1) void xb_dftregistration_coderGPU_ker
  (creal_T b_y1[262144], creal_T x[262144])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 512U);
  if (i < 512) {
    x[iv0 + (i << 9)] = b_y1[i + (iv0 << 9)];
  }
}

static __global__ __launch_bounds__(32, 1) void xc_dftregistration_coderGPU_ker(
  const real_T *tmp_im, const real_T *tmp_re, creal_T *y)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    y->re = *tmp_re * (*tmp_re * cos(*tmp_im));
    y->im = *tmp_re * (*tmp_re * sin(*tmp_im));
  }
}

static __global__ __launch_bounds__(1024, 1) void
  y_dftregistration_coderGPU_kern(const int32_T *nx, emxArray_creal_T *kernc)
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  creal_T x;
  real_T r;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(*nx - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx);
    if (kernc->data[iv0].im == 0.0) {
      x.re = exp(kernc->data[iv0].re);
      x.im = 0.0;
    } else if ((static_cast<int32_T>((static_cast<int32_T>(isinf(kernc->data[iv0]
        .im))) && (static_cast<int32_T>(isinf(kernc->data[iv0].re))))) && (
                static_cast<int32_T>(kernc->data[iv0].re < 0.0))) {
      x.re = 0.0;
      x.im = 0.0;
    } else {
      r = exp(kernc->data[iv0].re / 2.0);
      x.re = r * (r * cos(kernc->data[iv0].im));
      x.im = r * (r * sin(kernc->data[iv0].im));
    }

    kernc->data[iv0] = x;
  }
}

static __global__ __launch_bounds__(512, 1) void yb_dftregistration_coderGPU_ker
  (creal_T CC[262144])
{
  uint32_T threadId;
  int32_T i;
  real_T bi;
  real_T r;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    if (CC[i].im == 0.0) {
      bi = CC[i].re / 512.0;
      r = 0.0;
    } else if (CC[i].re == 0.0) {
      bi = 0.0;
      r = CC[i].im / 512.0;
    } else {
      bi = CC[i].re / 512.0;
      r = CC[i].im / 512.0;
    }

    CC[i].re = bi;
    CC[i].im = r;
  }
}

static __global__ __launch_bounds__(32, 1) void yc_dftregistration_coderGPU_ker(
  const creal_T *y, real_T *tmp_re)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *tmp_re = y->re;
  }
}

void dftregistration_coderGPU(c_dftregistration_coderGPUStack *SD, const creal_T
  buf1ft[262144], const creal_T buf2ft[262144], int16_T usfac, emxArray_real_T
  *output, creal_T Greg[262144])
{
  real_T row_shift;
  real_T col_shift;
  int32_T iv0;
  creal_T y[512];
  int32_T nx;
  creal_T CCmax;
  int32_T vstride;
  int32_T k;
  int32_T npages;
  int32_T vspread;
  int32_T midoffset;
  int32_T i2;
  real_T tmp_re;
  int32_T i;
  int16_T idx[512];
  int32_T i1;
  int32_T j;
  real_T tmp_im;
  real_T rg00_im;
  real_T ex_im;
  int32_T ia;
  real_T CCmax_im;
  int32_T ib;
  real_T br;
  creal_T ex[1024];
  boolean_T SCALEA;
  real_T absar;
  boolean_T SCALEB;
  real_T absbr;
  creal_T b_y;
  real_T Ma;
  int16_T yk;
  static const int16_T b[512] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13,
    14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31, 32,
    33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47, 48, 49, 50, 51,
    52, 53, 54, 55, 56, 57, 58, 59, 60, 61, 62, 63, 64, 65, 66, 67, 68, 69, 70,
    71, 72, 73, 74, 75, 76, 77, 78, 79, 80, 81, 82, 83, 84, 85, 86, 87, 88, 89,
    90, 91, 92, 93, 94, 95, 96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106,
    107, 108, 109, 110, 111, 112, 113, 114, 115, 116, 117, 118, 119, 120, 121,
    122, 123, 124, 125, 126, 127, 128, 129, 130, 131, 132, 133, 134, 135, 136,
    137, 138, 139, 140, 141, 142, 143, 144, 145, 146, 147, 148, 149, 150, 151,
    152, 153, 154, 155, 156, 157, 158, 159, 160, 161, 162, 163, 164, 165, 166,
    167, 168, 169, 170, 171, 172, 173, 174, 175, 176, 177, 178, 179, 180, 181,
    182, 183, 184, 185, 186, 187, 188, 189, 190, 191, 192, 193, 194, 195, 196,
    197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207, 208, 209, 210, 211,
    212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223, 224, 225, 226,
    227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239, 240, 241,
    242, 243, 244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255, -256,
    -255, -254, -253, -252, -251, -250, -249, -248, -247, -246, -245, -244, -243,
    -242, -241, -240, -239, -238, -237, -236, -235, -234, -233, -232, -231, -230,
    -229, -228, -227, -226, -225, -224, -223, -222, -221, -220, -219, -218, -217,
    -216, -215, -214, -213, -212, -211, -210, -209, -208, -207, -206, -205, -204,
    -203, -202, -201, -200, -199, -198, -197, -196, -195, -194, -193, -192, -191,
    -190, -189, -188, -187, -186, -185, -184, -183, -182, -181, -180, -179, -178,
    -177, -176, -175, -174, -173, -172, -171, -170, -169, -168, -167, -166, -165,
    -164, -163, -162, -161, -160, -159, -158, -157, -156, -155, -154, -153, -152,
    -151, -150, -149, -148, -147, -146, -145, -144, -143, -142, -141, -140, -139,
    -138, -137, -136, -135, -134, -133, -132, -131, -130, -129, -128, -127, -126,
    -125, -124, -123, -122, -121, -120, -119, -118, -117, -116, -115, -114, -113,
    -112, -111, -110, -109, -108, -107, -106, -105, -104, -103, -102, -101, -100,
    -99, -98, -97, -96, -95, -94, -93, -92, -91, -90, -89, -88, -87, -86, -85,
    -84, -83, -82, -81, -80, -79, -78, -77, -76, -75, -74, -73, -72, -71, -70,
    -69, -68, -67, -66, -65, -64, -63, -62, -61, -60, -59, -58, -57, -56, -55,
    -54, -53, -52, -51, -50, -49, -48, -47, -46, -45, -44, -43, -42, -41, -40,
    -39, -38, -37, -36, -35, -34, -33, -32, -31, -30, -29, -28, -27, -26, -25,
    -24, -23, -22, -21, -20, -19, -18, -17, -16, -15, -14, -13, -12, -11, -10,
    -9, -8, -7, -6, -5, -4, -3, -2, -1 };

  int16_T z;
  uint16_T c_y;
  uint16_T q;
  int16_T b_z;
  emxArray_creal_T *out;
  int16_T c_z;
  emxArray_int16_T *d_y;
  int16_T cloc;
  int16_T coff;
  emxArray_creal_T *e_y;
  emxArray_creal_T *kernc;
  emxArray_int16_T *f_y;
  emxArray_creal_T *d_z;
  emxArray_creal_T *kernr;
  static const int16_T x[512] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13,
    14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30, 31, 32,
    33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47, 48, 49, 50, 51,
    52, 53, 54, 55, 56, 57, 58, 59, 60, 61, 62, 63, 64, 65, 66, 67, 68, 69, 70,
    71, 72, 73, 74, 75, 76, 77, 78, 79, 80, 81, 82, 83, 84, 85, 86, 87, 88, 89,
    90, 91, 92, 93, 94, 95, 96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106,
    107, 108, 109, 110, 111, 112, 113, 114, 115, 116, 117, 118, 119, 120, 121,
    122, 123, 124, 125, 126, 127, 128, 129, 130, 131, 132, 133, 134, 135, 136,
    137, 138, 139, 140, 141, 142, 143, 144, 145, 146, 147, 148, 149, 150, 151,
    152, 153, 154, 155, 156, 157, 158, 159, 160, 161, 162, 163, 164, 165, 166,
    167, 168, 169, 170, 171, 172, 173, 174, 175, 176, 177, 178, 179, 180, 181,
    182, 183, 184, 185, 186, 187, 188, 189, 190, 191, 192, 193, 194, 195, 196,
    197, 198, 199, 200, 201, 202, 203, 204, 205, 206, 207, 208, 209, 210, 211,
    212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223, 224, 225, 226,
    227, 228, 229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239, 240, 241,
    242, 243, 244, 245, 246, 247, 248, 249, 250, 251, 252, 253, 254, 255, -256,
    -255, -254, -253, -252, -251, -250, -249, -248, -247, -246, -245, -244, -243,
    -242, -241, -240, -239, -238, -237, -236, -235, -234, -233, -232, -231, -230,
    -229, -228, -227, -226, -225, -224, -223, -222, -221, -220, -219, -218, -217,
    -216, -215, -214, -213, -212, -211, -210, -209, -208, -207, -206, -205, -204,
    -203, -202, -201, -200, -199, -198, -197, -196, -195, -194, -193, -192, -191,
    -190, -189, -188, -187, -186, -185, -184, -183, -182, -181, -180, -179, -178,
    -177, -176, -175, -174, -173, -172, -171, -170, -169, -168, -167, -166, -165,
    -164, -163, -162, -161, -160, -159, -158, -157, -156, -155, -154, -153, -152,
    -151, -150, -149, -148, -147, -146, -145, -144, -143, -142, -141, -140, -139,
    -138, -137, -136, -135, -134, -133, -132, -131, -130, -129, -128, -127, -126,
    -125, -124, -123, -122, -121, -120, -119, -118, -117, -116, -115, -114, -113,
    -112, -111, -110, -109, -108, -107, -106, -105, -104, -103, -102, -101, -100,
    -99, -98, -97, -96, -95, -94, -93, -92, -91, -90, -89, -88, -87, -86, -85,
    -84, -83, -82, -81, -80, -79, -78, -77, -76, -75, -74, -73, -72, -71, -70,
    -69, -68, -67, -66, -65, -64, -63, -62, -61, -60, -59, -58, -57, -56, -55,
    -54, -53, -52, -51, -50, -49, -48, -47, -46, -45, -44, -43, -42, -41, -40,
    -39, -38, -37, -36, -35, -34, -33, -32, -31, -30, -29, -28, -27, -26, -25,
    -24, -23, -22, -21, -20, -19, -18, -17, -16, -15, -14, -13, -12, -11, -10,
    -9, -8, -7, -6, -5, -4, -3, -2, -1 };

  emxArray_creal_T *g_y;
  creal_T alpha1;
  creal_T beta1;
  emxArray_creal_T *CC;
  emxArray_creal_T *b_ex;
  uint16_T uv[2];
  emxArray_uint16_T *b_idx;
  creal_T *gpu_alpha1;
  emxArray_creal_T *gpu_kernr;
  creal_T (*gpu_CC)[262144];
  creal_T *gpu_beta1;
  emxArray_creal_T *gpu_y;
  emxArray_creal_T *gpu_kernc;
  emxArray_creal_T *gpu_out;
  creal_T (*b_gpu_kernr)[512];
  creal_T (*b_gpu_y)[512];
  creal_T (*gpu_x)[262144];
  creal_T (*c_gpu_y)[1048576];
  creal_T (*b_gpu_CC)[1048576];
  creal_T (*gpu_y1)[262144];
  int16_T (*gpu_idx)[1024];
  creal_T (*gpu_ex)[1024];
  int32_T *gpu_nx;
  real_T *gpu_row_shift;
  creal_T *gpu_CCmax;
  creal_T *d_gpu_y;
  emxArray_int16_T *e_gpu_y;
  dim3 grid;
  dim3 block;
  boolean_T validLaunchParams;
  emxArray_creal_T *f_gpu_y;
  int16_T (*gpu_b)[512];
  dim3 b_grid;
  dim3 b_block;
  boolean_T b_validLaunchParams;
  dim3 c_grid;
  dim3 c_block;
  boolean_T c_validLaunchParams;
  emxArray_int16_T *g_gpu_y;
  dim3 d_grid;
  dim3 d_block;
  boolean_T d_validLaunchParams;
  emxArray_creal_T *gpu_z;
  int16_T (*b_gpu_x)[512];
  dim3 e_grid;
  dim3 e_block;
  boolean_T e_validLaunchParams;
  dim3 f_grid;
  dim3 f_block;
  boolean_T f_validLaunchParams;
  dim3 g_grid;
  dim3 g_block;
  boolean_T g_validLaunchParams;
  real_T *gpu_absar;
  dim3 h_grid;
  dim3 h_block;
  boolean_T h_validLaunchParams;
  emxArray_creal_T *c_gpu_CC;
  emxArray_creal_T *b_gpu_ex;
  uint16_T (*gpu_uv)[2];
  dim3 i_grid;
  dim3 i_block;
  boolean_T i_validLaunchParams;
  emxArray_uint16_T *b_gpu_idx;
  dim3 j_grid;
  dim3 j_block;
  boolean_T j_validLaunchParams;
  real_T *gpu_tmp_im;
  real_T (*gpu_dv1)[4];
  emxArray_real_T *gpu_output;
  int16_T (*c_gpu_idx)[512];
  real_T (*gpu_absb1)[262144];
  real_T (*gpu_absb2)[262144];
  real_T *gpu_tmp_re;
  real_T (*gpu_dv)[2];
  int16_T (*gpu_Nr)[262144];
  int16_T (*gpu_Nc)[262144];
  boolean_T CC_dirtyOnGpu;
  boolean_T kernc_dirtyOnGpu;
  boolean_T y_dirtyOnGpu;
  boolean_T x_dirtyOnGpu;
  boolean_T b_CC_dirtyOnGpu;
  boolean_T row_shift_dirtyOnGpu;
  boolean_T CCmax_dirtyOnGpu;
  boolean_T b_y_dirtyOnGpu;
  boolean_T c_y_dirtyOnGpu;
  boolean_T z_dirtyOnGpu;
  boolean_T c_CC_dirtyOnGpu;
  boolean_T ex_dirtyOnGpu;
  boolean_T idx_dirtyOnGpu;
  boolean_T CC_dirtyOnCpu;
  boolean_T y_dirtyOnCpu;
  boolean_T kernc_dirtyOnCpu;
  boolean_T out_dirtyOnCpu;
  boolean_T x_dirtyOnCpu;
  boolean_T b_CC_dirtyOnCpu;
  boolean_T nx_dirtyOnCpu;
  boolean_T row_shift_dirtyOnCpu;
  boolean_T b_y_dirtyOnCpu;
  boolean_T c_y_dirtyOnCpu;
  boolean_T b_dirtyOnCpu;
  boolean_T z_dirtyOnCpu;
  boolean_T absar_dirtyOnCpu;
  boolean_T c_CC_dirtyOnCpu;
  boolean_T idx_dirtyOnCpu;
  boolean_T tmp_im_dirtyOnCpu;
  boolean_T tmp_re_dirtyOnCpu;
  emxArray_int16_T inter_y;
  emxArray_creal_T b_inter_y;
  emxArray_creal_T inter_kernc;
  emxArray_int16_T c_inter_y;
  emxArray_creal_T inter_z;
  emxArray_creal_T inter_kernr;
  emxArray_creal_T d_inter_y;
  emxArray_creal_T inter_out;
  emxArray_creal_T inter_CC;
  emxArray_creal_T inter_ex;
  emxArray_uint16_T inter_idx;
  emxArray_real_T inter_output;
  int32_T b_nx;
  hipMalloc(&gpu_Nc, 524288ULL);
  hipMalloc(&gpu_Nr, 524288ULL);
  hipMalloc(&gpu_dv, 16ULL);
  hipMalloc(&gpu_absb2, 2097152ULL);
  hipMalloc(&gpu_absb1, 2097152ULL);
  hipMalloc(&c_gpu_idx, 1024ULL);
  hipMalloc(&gpu_y1, 4194304ULL);
  hipMalloc(&gpu_dv1, 32ULL);
  hipMalloc(&gpu_output, 32ULL);
  gpuEmxReset_real_T(&inter_output);
  hipMalloc(&b_gpu_kernr, 8192ULL);
  hipMalloc(&gpu_CCmax, 16ULL);
  hipMalloc(&gpu_uv, 4ULL);
  hipMalloc(&b_gpu_idx, 32ULL);
  gpuEmxReset_uint16_T(&inter_idx);
  hipMalloc(&b_gpu_ex, 32ULL);
  gpuEmxReset_creal_T(&inter_ex);
  hipMalloc(&c_gpu_CC, 32ULL);
  gpuEmxReset_creal_T(&inter_CC);
  hipMalloc(&gpu_out, 32ULL);
  gpuEmxReset_creal_T(&inter_out);
  hipMalloc(&gpu_beta1, 16ULL);
  hipMalloc(&gpu_alpha1, 16ULL);
  hipMalloc(&gpu_y, 32ULL);
  gpuEmxReset_creal_T(&d_inter_y);
  hipMalloc(&gpu_kernr, 32ULL);
  gpuEmxReset_creal_T(&inter_kernr);
  hipMalloc(&b_gpu_x, 1024ULL);
  hipMalloc(&gpu_z, 32ULL);
  gpuEmxReset_creal_T(&inter_z);
  hipMalloc(&g_gpu_y, 32ULL);
  gpuEmxReset_int16_T(&c_inter_y);
  hipMalloc(&gpu_b, 1024ULL);
  hipMalloc(&gpu_kernc, 32ULL);
  gpuEmxReset_creal_T(&inter_kernc);
  hipMalloc(&f_gpu_y, 32ULL);
  gpuEmxReset_creal_T(&b_inter_y);
  hipMalloc(&e_gpu_y, 32ULL);
  gpuEmxReset_int16_T(&inter_y);
  hipMalloc(&d_gpu_y, 16ULL);
  hipMalloc(&b_gpu_y, 8192ULL);
  hipMalloc(&gpu_row_shift, 8ULL);
  hipMalloc(&gpu_absar, 8ULL);
  hipMalloc(&gpu_ex, 16384ULL);
  hipMalloc(&gpu_idx, 2048ULL);
  hipMalloc(&c_gpu_y, 16777216ULL);
  hipMalloc(&b_gpu_CC, 16777216ULL);
  hipMalloc(&gpu_x, 4194304ULL);
  hipMalloc(&gpu_tmp_im, 8ULL);
  hipMalloc(&gpu_tmp_re, 8ULL);
  hipMalloc(&gpu_CC, 4194304ULL);
  hipMalloc(&gpu_nx, 4ULL);
  tmp_re_dirtyOnCpu = false;
  absar_dirtyOnCpu = false;
  b_y_dirtyOnCpu = false;
  b_dirtyOnCpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);

  /*   this _coder version has calls to exist() removed to enable mex compilation */
  /*  Add kernelfun pragma to trigger GPU kernel creation */
  /*  function [output Greg] = dftregistration(buf1ft,buf2ft,usfac); */
  /*  Efficient subpixel image registration by crosscorrelation. This code */
  /*  gives the same precision as the FFT upsampled cross correlation in a */
  /*  small fraction of the computation time and with reduced memory  */
  /*  requirements. It obtains an initial estimate of the crosscorrelation peak */
  /*  by an FFT and then refines the shift estimation by upsampling the DFT */
  /*  only in a small neighborhood of that estimate by means of a  */
  /*  matrix-multiply DFT. With this procedure all the image points are used to */
  /*  compute the upsampled crosscorrelation. */
  /*  Manuel Guizar - Dec 13, 2007 */
  /*  Portions of this code were taken from code written by Ann M. Kowalczyk  */
  /*  and James R. Fienup.  */
  /*  J.R. Fienup and A.M. Kowalczyk, "Phase retrieval for a complex-valued  */
  /*  object by using a low-resolution image," J. Opt. Soc. Am. A 7, 450-458  */
  /*  (1990). */
  /*  Citation for this algorithm: */
  /*  Manuel Guizar-Sicairos, Samuel T. Thurman, and James R. Fienup,  */
  /*  "Efficient subpixel image registration algorithms," Opt. Lett. 33,  */
  /*  156-158 (2008). */
  /*  Inputs */
  /*  buf1ft    Fourier transform of reference image,  */
  /*            DC in (1,1)   [DO NOT FFTSHIFT] */
  /*  buf2ft    Fourier transform of image to register,  */
  /*            DC in (1,1) [DO NOT FFTSHIFT] */
  /*  usfac     Upsampling factor (integer). Images will be registered to  */
  /*            within 1/usfac of a pixel. For example usfac = 20 means the */
  /*            images will be registered within 1/20 of a pixel. (default = 1) */
  /*  Outputs */
  /*  output =  [error,diffphase,net_row_shift,net_col_shift] */
  /*  error     Translation invariant normalized RMS error between f and g */
  /*  diffphase     Global phase difference between the two images (should be */
  /*                zero if images are non-negative). */
  /*  net_row_shift net_col_shift   Pixel shifts between images */
  /*  Greg      (Optional) Fourier transform of registered version of buf2ft, */
  /*            the global phase difference is compensated for. */
  /*  Default usfac to 1 */
  /* %%if exist('usfac', 'var')~=1, usfac=1; end */
  row_shift = 0.0;
  row_shift_dirtyOnGpu = false;
  col_shift = 0.0;

  /*  Compute error for no pixel shift */
  if (usfac == 0) {
    kc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (buf2ft, buf1ft, *gpu_CC, *gpu_absb2, *gpu_absb1);
    lc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*b_gpu_y, *gpu_CC);
    y_dirtyOnGpu = true;
    mc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (gpu_CCmax);
    CCmax_dirtyOnGpu = true;
    for (iv0 = 0; iv0 < 512; iv0++) {
      if (y_dirtyOnGpu) {
        hipMemcpy(&y[0], b_gpu_y, 8192ULL, hipMemcpyDeviceToHost);
        y_dirtyOnGpu = false;
      }

      if (CCmax_dirtyOnGpu) {
        hipMemcpy(&CCmax, gpu_CCmax, 16ULL, hipMemcpyDeviceToHost);
        CCmax_dirtyOnGpu = false;
      }

      CCmax.re += y[iv0].re;
      CCmax.im += y[iv0].im;
    }

    nc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb1);
    oc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_absb1, gpu_tmp_re);
    pc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb1, gpu_tmp_re);
    qc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb2);
    rc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_absb2, gpu_tmp_im);
    sc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb2, gpu_tmp_im);
    if (CCmax_dirtyOnGpu) {
      hipMemcpy(&CCmax, gpu_CCmax, 16ULL, hipMemcpyDeviceToHost);
    }

    ex_im = CCmax.re * CCmax.re - CCmax.im * -CCmax.im;
    CCmax_im = CCmax.re * -CCmax.im + CCmax.im * CCmax.re;
    hipMemcpy(&tmp_im, gpu_tmp_im, 8ULL, hipMemcpyDeviceToHost);
    hipMemcpy(&tmp_re, gpu_tmp_re, 8ULL, hipMemcpyDeviceToHost);
    br = tmp_im * tmp_re;
    if (CCmax_im == 0.0) {
      ex_im /= br;
      CCmax_im = 0.0;
    } else if (ex_im == 0.0) {
      ex_im = 0.0;
      CCmax_im /= br;
    } else {
      ex_im /= br;
      CCmax_im /= br;
    }

    tmp_im = rt_atan2d_snf(CCmax.im, CCmax.re);
    hipMemcpy(gpu_tmp_im, &tmp_im, 8ULL, hipMemcpyHostToDevice);
    tmp_im_dirtyOnCpu = false;
    tc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (gpu_tmp_im, CCmax_im, ex_im, *gpu_dv);
    nx = output->size[0] * output->size[1];
    output->size[0] = 1;
    output->size[1] = 2;
    emxEnsureCapacity_real_T(output, nx);
    gpuEmxMemcpyCpuToGpu_real_T(output, &inter_output, gpu_output);
    uc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_dv, gpu_output);

    /*  Whole-pixel shift - Compute crosscorrelation by an IFFT and locate the */
    /*  peak */
  } else if (usfac == 1) {
    vb_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (buf1ft, buf2ft, *gpu_CC);
    cufftEnsureInitialization(512, HIPFFT_Z2Z, 512, 512);
    hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*gpu_CC)[0],
                 (hipfftDoubleComplex *)&(*gpu_y1)[0], 1);
    wb_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_y1);
    xb_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_y1, *gpu_x);
    cufftEnsureInitialization(512, HIPFFT_Z2Z, 512, 512);
    hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*gpu_x)[0],
                 (hipfftDoubleComplex *)&(*gpu_CC)[0], 1);
    yb_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_CC);
    ac_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*c_gpu_idx);
    bc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*c_gpu_idx, *gpu_CC, *b_gpu_y);
    nx = 0;
    hipMemcpy(&y[0], b_gpu_y, 8192ULL, hipMemcpyDeviceToHost);
    rg00_im = y[0].re;
    ex_im = y[0].im;
    for (i = 0; i < 511; i++) {
      if (rtIsNaN(y[i + 1].re) || rtIsNaN(y[i + 1].im)) {
        SCALEA = false;
      } else if (rtIsNaN(rg00_im) || rtIsNaN(ex_im)) {
        SCALEA = true;
      } else {
        if ((fabs(rg00_im) > 8.9884656743115785E+307) || (fabs(ex_im) >
             8.9884656743115785E+307)) {
          SCALEA = true;
        } else {
          SCALEA = false;
        }

        if ((fabs(y[i + 1].re) > 8.9884656743115785E+307) || (fabs(y[i + 1].im) >
             8.9884656743115785E+307)) {
          SCALEB = true;
        } else {
          SCALEB = false;
        }

        if (SCALEA || SCALEB) {
          tmp_im = rt_hypotd_snf(rg00_im / 2.0, ex_im / 2.0);
          br = rt_hypotd_snf(y[i + 1].re / 2.0, y[i + 1].im / 2.0);
        } else {
          tmp_im = rt_hypotd_snf(rg00_im, ex_im);
          br = rt_hypotd_snf(y[i + 1].re, y[i + 1].im);
        }

        if (tmp_im == br) {
          absar = fabs(rg00_im);
          absar_dirtyOnCpu = true;
          tmp_re = fabs(ex_im);
          absbr = fabs(y[i + 1].re);
          tmp_im = fabs(y[i + 1].im);
          if (absar > tmp_re) {
            Ma = absar;
            absar = tmp_re;
          } else {
            Ma = tmp_re;
          }

          if (absbr > tmp_im) {
            tmp_re = absbr;
            tmp_re_dirtyOnCpu = true;
            absbr = tmp_im;
          } else {
            tmp_re = tmp_im;
            tmp_re_dirtyOnCpu = true;
          }

          if (Ma > tmp_re) {
            if (absar < absbr) {
              tmp_im = Ma - tmp_re;
              br = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + tmp_re / 2.0) *
                (absbr - absar);
            } else {
              tmp_im = Ma;
              br = tmp_re;
            }
          } else if (Ma < tmp_re) {
            if (absar > absbr) {
              br = tmp_re - Ma;
              tmp_im = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + tmp_re / 2.0) *
                (absar - absbr);
            } else {
              tmp_im = Ma;
              br = tmp_re;
            }
          } else {
            tmp_im = absar;
            br = absbr;
          }

          if (tmp_im == br) {
            tmp_im = rt_atan2d_snf(ex_im, rg00_im);
            br = rt_atan2d_snf(y[i + 1].im, y[i + 1].re);
            if (tmp_im == br) {
              br = y[i + 1].re;
              tmp_re = y[i + 1].im;
              if (tmp_im > 0.78539816339744828) {
                if (tmp_im > 2.3561944901923448) {
                  tmp_im = -ex_im;
                  br = -tmp_re;
                } else {
                  tmp_im = -rg00_im;
                  br = -br;
                }
              } else if (tmp_im > -0.78539816339744828) {
                tmp_im = ex_im;
                br = tmp_re;
              } else if (tmp_im > -2.3561944901923448) {
                tmp_im = rg00_im;
              } else {
                tmp_im = -ex_im;
                br = -tmp_re;
              }

              if (tmp_im == br) {
                tmp_im = 0.0;
                br = 0.0;
              }
            }
          }
        }

        SCALEA = (tmp_im < br);
      }

      if (SCALEA) {
        rg00_im = y[i + 1].re;
        ex_im = y[i + 1].im;
        nx = i + 1;
      }
    }

    cc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (buf2ft, buf1ft, *gpu_absb1, *gpu_absb2);
    if (absar_dirtyOnCpu) {
      hipMemcpy(gpu_absar, &absar, 8ULL, hipMemcpyHostToDevice);
    }

    dc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_absb1, gpu_absar);
    ec_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb1, gpu_absar);
    fc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb2);
    if (tmp_re_dirtyOnCpu) {
      hipMemcpy(gpu_tmp_re, &tmp_re, 8ULL, hipMemcpyHostToDevice);
      tmp_re_dirtyOnCpu = false;
    }

    gc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_absb2, gpu_tmp_re);
    hc_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_absb2, gpu_tmp_re);

    /*  slow code */
    /* rgzero = sum(abs(buf2ft(:)).^2)/(m*n);  */
    hipMemcpy(&SD->f0.b_CC[0], gpu_CC, 4194304ULL, hipMemcpyDeviceToHost);
    hipMemcpy(&idx[0], c_gpu_idx, 1024ULL, hipMemcpyDeviceToHost);
    absbr = SD->f0.b_CC[(idx[nx] + (nx << 9)) - 1].re;
    Ma = -SD->f0.b_CC[(idx[nx] + (nx << 9)) - 1].im;
    tmp_im = SD->f0.b_CC[(idx[nx] + (nx << 9)) - 1].re * absbr - SD->f0.b_CC
      [(idx[nx] + (nx << 9)) - 1].im * Ma;
    Ma = SD->f0.b_CC[(idx[nx] + (nx << 9)) - 1].re * Ma + SD->f0.b_CC[(idx[nx] +
      (nx << 9)) - 1].im * absbr;
    hipMemcpy(&absar, gpu_absar, 8ULL, hipMemcpyDeviceToHost);
    hipMemcpy(&tmp_re, gpu_tmp_re, 8ULL, hipMemcpyDeviceToHost);
    br = tmp_re / 262144.0 * (absar / 262144.0);
    if (Ma == 0.0) {
      absbr = tmp_im / br;
      Ma = 0.0;
    } else if (tmp_im == 0.0) {
      absbr = 0.0;
      Ma /= br;
    } else {
      absbr = tmp_im / br;
      Ma /= br;
    }

    tmp_im = rt_atan2d_snf(SD->f0.b_CC[(idx[nx] + (nx << 9)) - 1].im,
      SD->f0.b_CC[(idx[nx] + (nx << 9)) - 1].re);
    if (idx[nx] > 256) {
      row_shift = idx[nx] - 513;
    } else {
      row_shift = idx[nx] - 1;
    }

    if (nx + 1 > 256) {
      col_shift = nx - 512;
    } else {
      col_shift = nx;
    }

    hipMemcpy(gpu_row_shift, &row_shift, 8ULL, hipMemcpyHostToDevice);
    hipMemcpy(gpu_tmp_im, &tmp_im, 8ULL, hipMemcpyHostToDevice);
    tmp_im_dirtyOnCpu = false;
    ic_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (col_shift, gpu_row_shift, gpu_tmp_im, Ma, absbr, *gpu_dv1);
    nx = output->size[0] * output->size[1];
    output->size[0] = 1;
    output->size[1] = 4;
    emxEnsureCapacity_real_T(output, nx);
    gpuEmxMemcpyCpuToGpu_real_T(output, &inter_output, gpu_output);
    jc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_dv1, gpu_output);

    /*  Partial-pixel shift */
  } else {
    /*  First upsample by a factor of 2 to obtain initial estimate */
    /*  Embed Fourier data in a 2x larger array */
    c_dftregistration_coderGPU_kern<<<dim3(2048U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*b_gpu_CC);
    d_dftregistration_coderGPU_kern<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (buf1ft, *gpu_CC);
    CC_dirtyOnCpu = false;
    CC_dirtyOnGpu = true;
    for (nx = 0; nx < 2; nx++) {
      vstride = 1;
      for (k = 0; k < nx; k++) {
        vstride <<= 9;
      }

      npages = 1;
      for (k = 0; k <= -nx; k++) {
        npages <<= 9;
      }

      vspread = 511 * vstride;
      midoffset = (vstride << 8) - 1;
      i2 = 0;
      for (i = 0; i < npages; i++) {
        i1 = i2;
        i2 += vspread;
        for (j = 0; j < vstride; j++) {
          i1++;
          i2++;
          ia = i1 - 1;
          ib = i1 + midoffset;
          for (k = 0; k < 256; k++) {
            if (CC_dirtyOnGpu) {
              hipMemcpy(&SD->f0.b_CC[0], gpu_CC, 4194304ULL,
                         hipMemcpyDeviceToHost);
              CC_dirtyOnGpu = false;
            }

            tmp_re = SD->f0.b_CC[ia].re;
            tmp_im = SD->f0.b_CC[ia].im;
            SD->f0.b_CC[ia] = SD->f0.b_CC[ib];
            SD->f0.b_CC[ib].re = tmp_re;
            SD->f0.b_CC[ib].im = tmp_im;
            CC_dirtyOnCpu = true;
            ia += vstride;
            ib += vstride;
          }
        }
      }
    }

    e_dftregistration_coderGPU_kern<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (buf2ft, *gpu_x);
    x_dirtyOnCpu = false;
    x_dirtyOnGpu = true;
    for (nx = 0; nx < 2; nx++) {
      vstride = 1;
      for (k = 0; k < nx; k++) {
        vstride <<= 9;
      }

      npages = 1;
      for (k = 0; k <= -nx; k++) {
        npages <<= 9;
      }

      vspread = 511 * vstride;
      midoffset = (vstride << 8) - 1;
      i2 = 0;
      for (i = 0; i < npages; i++) {
        i1 = i2;
        i2 += vspread;
        for (j = 0; j < vstride; j++) {
          i1++;
          i2++;
          ia = i1 - 1;
          ib = i1 + midoffset;
          for (k = 0; k < 256; k++) {
            if (x_dirtyOnGpu) {
              hipMemcpy(&SD->f0.x[0], gpu_x, 4194304ULL, hipMemcpyDeviceToHost);
              x_dirtyOnGpu = false;
            }

            tmp_re = SD->f0.x[ia].re;
            tmp_im = SD->f0.x[ia].im;
            SD->f0.x[ia] = SD->f0.x[ib];
            SD->f0.x[ib].re = tmp_re;
            SD->f0.x[ib].im = tmp_im;
            x_dirtyOnCpu = true;
            ia += vstride;
            ib += vstride;
          }
        }
      }
    }

    if (CC_dirtyOnCpu) {
      hipMemcpy(gpu_CC, &SD->f0.b_CC[0], 4194304ULL, hipMemcpyHostToDevice);
    }

    if (x_dirtyOnCpu) {
      hipMemcpy(gpu_x, &SD->f0.x[0], 4194304ULL, hipMemcpyHostToDevice);
    }

    f_dftregistration_coderGPU_kern<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_CC, *b_gpu_CC, *gpu_x);
    b_CC_dirtyOnCpu = false;
    b_CC_dirtyOnGpu = true;

    /*  Compute crosscorrelation and locate the peak  */
    for (nx = 0; nx < 2; nx++) {
      vstride = 1;
      for (k = 0; k < nx; k++) {
        vstride <<= 10;
      }

      npages = 1;
      for (k = 0; k <= -nx; k++) {
        npages <<= 10;
      }

      vspread = 1023 * vstride;
      midoffset = (vstride << 9) - 1;
      i2 = 0;
      for (i = 0; i < npages; i++) {
        i1 = i2;
        i2 += vspread;
        for (j = 0; j < vstride; j++) {
          i1++;
          i2++;
          ia = i1 - 1;
          ib = i1 + midoffset;
          for (k = 0; k < 512; k++) {
            if (b_CC_dirtyOnGpu) {
              hipMemcpy(&SD->f0.CC[0], b_gpu_CC, 16777216ULL,
                         hipMemcpyDeviceToHost);
              b_CC_dirtyOnGpu = false;
            }

            tmp_re = SD->f0.CC[ia].re;
            tmp_im = SD->f0.CC[ia].im;
            SD->f0.CC[ia] = SD->f0.CC[ib];
            SD->f0.CC[ib].re = tmp_re;
            SD->f0.CC[ib].im = tmp_im;
            b_CC_dirtyOnCpu = true;
            ia += vstride;
            ib += vstride;
          }
        }
      }
    }

    if (b_CC_dirtyOnCpu) {
      hipMemcpy(b_gpu_CC, &SD->f0.CC[0], 16777216ULL, hipMemcpyHostToDevice);
    }

    g_dftregistration_coderGPU_kern<<<dim3(2048U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*b_gpu_CC, *c_gpu_y);
    cufftEnsureInitialization(1024, HIPFFT_Z2Z, 1024, 1024);
    hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*c_gpu_y)[0],
                 (hipfftDoubleComplex *)&(*b_gpu_CC)[0], 1);
    h_dftregistration_coderGPU_kern<<<dim3(2048U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*b_gpu_CC);
    i_dftregistration_coderGPU_kern<<<dim3(2048U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*b_gpu_CC, *c_gpu_y);
    cufftEnsureInitialization(1024, HIPFFT_Z2Z, 1024, 1024);
    hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*c_gpu_y)[0],
                 (hipfftDoubleComplex *)&(*b_gpu_CC)[0], 1);
    j_dftregistration_coderGPU_kern<<<dim3(2048U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*b_gpu_CC);

    /*  Calculate cross-correlation */
    k_dftregistration_coderGPU_kern<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_idx);
    l_dftregistration_coderGPU_kern<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_idx, *b_gpu_CC, *gpu_ex);
    nx = 0;
    hipMemcpy(&ex[0], gpu_ex, 16384ULL, hipMemcpyDeviceToHost);
    rg00_im = ex[0].re;
    ex_im = ex[0].im;
    for (i = 0; i < 1023; i++) {
      if (rtIsNaN(ex[i + 1].re) || rtIsNaN(ex[i + 1].im)) {
        SCALEA = false;
      } else if (rtIsNaN(rg00_im) || rtIsNaN(ex_im)) {
        SCALEA = true;
      } else {
        if ((fabs(rg00_im) > 8.9884656743115785E+307) || (fabs(ex_im) >
             8.9884656743115785E+307)) {
          SCALEA = true;
        } else {
          SCALEA = false;
        }

        if ((fabs(ex[i + 1].re) > 8.9884656743115785E+307) || (fabs(ex[i + 1].im)
             > 8.9884656743115785E+307)) {
          SCALEB = true;
        } else {
          SCALEB = false;
        }

        if (SCALEA || SCALEB) {
          tmp_im = rt_hypotd_snf(rg00_im / 2.0, ex_im / 2.0);
          br = rt_hypotd_snf(ex[i + 1].re / 2.0, ex[i + 1].im / 2.0);
        } else {
          tmp_im = rt_hypotd_snf(rg00_im, ex_im);
          br = rt_hypotd_snf(ex[i + 1].re, ex[i + 1].im);
        }

        if (tmp_im == br) {
          absar = fabs(rg00_im);
          tmp_re = fabs(ex_im);
          absbr = fabs(ex[i + 1].re);
          tmp_im = fabs(ex[i + 1].im);
          if (absar > tmp_re) {
            Ma = absar;
            absar = tmp_re;
          } else {
            Ma = tmp_re;
          }

          if (absbr > tmp_im) {
            tmp_re = absbr;
            absbr = tmp_im;
          } else {
            tmp_re = tmp_im;
          }

          if (Ma > tmp_re) {
            if (absar < absbr) {
              tmp_im = Ma - tmp_re;
              br = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + tmp_re / 2.0) *
                (absbr - absar);
            } else {
              tmp_im = Ma;
              br = tmp_re;
            }
          } else if (Ma < tmp_re) {
            if (absar > absbr) {
              br = tmp_re - Ma;
              tmp_im = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + tmp_re / 2.0) *
                (absar - absbr);
            } else {
              tmp_im = Ma;
              br = tmp_re;
            }
          } else {
            tmp_im = absar;
            br = absbr;
          }

          if (tmp_im == br) {
            tmp_im = rt_atan2d_snf(ex_im, rg00_im);
            br = rt_atan2d_snf(ex[i + 1].im, ex[i + 1].re);
            if (tmp_im == br) {
              br = ex[i + 1].re;
              tmp_re = ex[i + 1].im;
              if (tmp_im > 0.78539816339744828) {
                if (tmp_im > 2.3561944901923448) {
                  tmp_im = -ex_im;
                  br = -tmp_re;
                } else {
                  tmp_im = -rg00_im;
                  br = -br;
                }
              } else if (tmp_im > -0.78539816339744828) {
                tmp_im = ex_im;
                br = tmp_re;
              } else if (tmp_im > -2.3561944901923448) {
                tmp_im = rg00_im;
              } else {
                tmp_im = -ex_im;
                br = -tmp_re;
              }

              if (tmp_im == br) {
                tmp_im = 0.0;
                br = 0.0;
              }
            }
          }
        }

        SCALEA = (tmp_im < br);
      }

      if (SCALEA) {
        rg00_im = ex[i + 1].re;
        ex_im = ex[i + 1].im;
        nx = i + 1;
      }
    }

    /*  Obtain shift in original pixel grid from the position of the */
    /*  crosscorrelation peak  */
    hipMemcpy(gpu_nx, &nx, 4ULL, hipMemcpyHostToDevice);
    hipMemcpy(gpu_row_shift, &row_shift, 8ULL, hipMemcpyHostToDevice);
    m_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_idx, gpu_nx, gpu_row_shift, *b_gpu_CC, gpu_CCmax);
    row_shift_dirtyOnCpu = false;
    row_shift_dirtyOnGpu = true;
    if (nx + 1 > 512) {
      b_nx = nx - 1024;
    } else {
      b_nx = nx;
    }

    col_shift = static_cast<real_T>(b_nx) / 2.0;

    /*  If upsampling > 2, then refine estimate with matrix multiply DFT */
    if (usfac > 2) {
      /* %% DFT computation %%% */
      /*  Initial shift estimate in upsampled grid */
      hipMemcpy(&row_shift, gpu_row_shift, 8ULL, hipMemcpyDeviceToHost);
      nx = static_cast<int32_T>(rt_roundd_snf(row_shift * static_cast<real_T>
        (usfac)));
      if (nx < 32768) {
        if (nx >= -32768) {
          yk = static_cast<int16_T>(nx);
        } else {
          yk = MIN_int16_T;
        }
      } else {
        yk = MAX_int16_T;
      }

      if (yk >= 0) {
        c_y = static_cast<uint16_T>(yk);
      } else if (yk == -32768) {
        c_y = 32768U;
      } else {
        c_y = static_cast<uint16_T>(-yk);
      }

      q = static_cast<uint16_T>(static_cast<uint32_T>(c_y) / usfac);
      c_y = static_cast<uint16_T>(static_cast<uint32_T>(c_y) -
        static_cast<uint16_T>(static_cast<uint32_T>(q) * usfac));
      if ((c_y > 0) && (c_y >= static_cast<int32_T>(static_cast<uint32_T>(usfac)
            >> 1) + (usfac & 1))) {
        q = static_cast<uint16_T>(q + 1);
      }

      z = static_cast<int16_T>(q);
      if (yk < 0) {
        z = static_cast<int16_T>(-q);
      }

      nx = static_cast<int32_T>(rt_roundd_snf(col_shift * static_cast<real_T>
        (usfac)));
      if (nx < 32768) {
        if (nx >= -32768) {
          yk = static_cast<int16_T>(nx);
        } else {
          yk = MIN_int16_T;
        }
      } else {
        yk = MAX_int16_T;
      }

      if (yk >= 0) {
        c_y = static_cast<uint16_T>(yk);
      } else if (yk == -32768) {
        c_y = 32768U;
      } else {
        c_y = static_cast<uint16_T>(-yk);
      }

      q = static_cast<uint16_T>(static_cast<uint32_T>(c_y) / usfac);
      c_y = static_cast<uint16_T>(static_cast<uint32_T>(c_y) -
        static_cast<uint16_T>(static_cast<uint32_T>(q) * usfac));
      if ((c_y > 0) && (c_y >= static_cast<int32_T>(static_cast<uint32_T>(usfac)
            >> 1) + (usfac & 1))) {
        q = static_cast<uint16_T>(q + 1);
      }

      b_z = static_cast<int16_T>(q);
      if (yk < 0) {
        b_z = static_cast<int16_T>(-q);
      }

      emxInit_creal_T(&out, 2, true);
      nx = static_cast<int32_T>(rt_roundd_snf(static_cast<real_T>(usfac) * 1.5));
      if (nx < 32768) {
        yk = static_cast<int16_T>(nx);
      } else {
        yk = MAX_int16_T;
      }

      c_z = static_cast<int16_T>(rt_roundd_snf(static_cast<real_T>(yk) / 2.0));

      /*  Center of output array at dftshift+1 */
      /*  Matrix multiply DFT around the current shift estimate */
      vspread = usfac * usfac;
      if (vspread > 32767) {
        vspread = 32767;
      }

      t_dftregistration_coderGPU_kern<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
        (buf1ft, buf2ft, *gpu_CC);
      emxInit_int16_T(&d_y, 2, true);
      nx = z * usfac;
      if (nx > 32767) {
        nx = 32767;
      } else {
        if (nx < -32768) {
          nx = -32768;
        }
      }

      nx = c_z - nx;
      if (nx > 32767) {
        nx = 32767;
      }

      cloc = static_cast<int16_T>(nx);
      nx = b_z * usfac;
      if (nx > 32767) {
        nx = 32767;
      } else {
        if (nx < -32768) {
          nx = -32768;
        }
      }

      nx = c_z - nx;
      if (nx > 32767) {
        nx = 32767;
      }

      coff = static_cast<int16_T>(nx);

      /*   this _coder version has calls to exist() removed to enable mex compilation */
      /*  function out=dftups(in,nor,noc,usfac,roff,coff); */
      /*  Upsampled DFT by matrix multiplies, can compute an upsampled DFT in just */
      /*  a small region. */
      /*  usfac         Upsampling factor (default usfac = 1) */
      /*  [nor,noc]     Number of pixels in the output upsampled DFT, in */
      /*                units of upsampled pixels (default = size(in)) */
      /*  roff, coff    Row and column offsets, allow to shift the output array to */
      /*                a region of interest on the DFT (default = 0) */
      /*  Recieves DC in upper left corner, image center must be in (1,1)  */
      /*  Manuel Guizar - Dec 13, 2007 */
      /*  Modified from dftus, by J.R. Fienup 7/31/06 */
      /*  This code is intended to provide the same result as if the following */
      /*  operations were performed */
      /*    - Embed the array "in" in an array that is usfac times larger in each */
      /*      dimension. ifftshift to bring the center of the image to (1,1). */
      /*    - Take the FFT of the larger array */
      /*    - Extract an [nor, noc] region of the result. Starting with the  */
      /*      [roff+1 coff+1] element. */
      /*  It achieves this result by computing the DFT in the output array without */
      /*  the need to zeropad. Much faster and memory efficient than the */
      /*  zero-padded FFT approach if [nor noc] are much smaller than [nr*usfac nc*usfac] */
      /*  Set defaults */
      /*  % % if exist('roff', 'var')~=1, roff=0; end */
      /*  % % if exist('coff', 'var')~=1, coff=0; end */
      /*  % % if exist('usfac', 'var')~=1, usfac=1; end */
      /*  % % if exist('noc', 'var')~=1, noc=nc; end */
      /*  % % if exist('nor', 'var')~=1, nor=nr; end */
      /*  Compute kernels and obtain DFT by matrix products */
      if (usfac > 63) {
        yk = MAX_int16_T;
      } else {
        yk = static_cast<int16_T>(usfac << 9);
      }

      u_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (yk, d_gpu_y);
      nx = static_cast<int32_T>(rt_roundd_snf(static_cast<real_T>(usfac) * 1.5));
      if (nx < 32768) {
        yk = static_cast<int16_T>(nx);
      } else {
        yk = MAX_int16_T;
      }

      vstride = yk;
      nx = d_y->size[0] * d_y->size[1];
      d_y->size[0] = 1;
      d_y->size[1] = yk;
      emxEnsureCapacity_int16_T(d_y, nx);
      d_y->data[0] = 0;
      yk = 0;
      for (k = 0; k <= vstride - 2; k++) {
        yk = static_cast<int16_T>(yk + 1);
        d_y->data[k + 1] = yk;
      }

      emxInit_creal_T(&e_y, 2, true);
      c_y_dirtyOnGpu = false;
      nx = e_y->size[0] * e_y->size[1];
      e_y->size[0] = 1;
      e_y->size[1] = d_y->size[1];
      emxEnsureCapacity_creal_T(e_y, nx);
      c_y_dirtyOnCpu = true;
      validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((d_y->size[0]
        * d_y->size[1] - 1) + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        gpuEmxMemcpyCpuToGpu_int16_T(d_y, &inter_y, e_gpu_y);
        gpuEmxMemcpyCpuToGpu_creal_T(e_y, &b_inter_y, f_gpu_y);
        v_dftregistration_coderGPU_kern<<<grid, block>>>(coff, e_gpu_y, f_gpu_y);
        c_y_dirtyOnCpu = false;
        c_y_dirtyOnGpu = true;
      }

      emxFree_int16_T(&d_y);
      gpuEmxFree_int16_T(&inter_y);
      emxInit_creal_T(&kernc, 2, true);
      kernc_dirtyOnGpu = false;
      nx = kernc->size[0] * kernc->size[1];
      kernc->size[0] = 512;
      if (c_y_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal_T(e_y, &b_inter_y);
      }

      kernc->size[1] = e_y->size[1];
      emxEnsureCapacity_creal_T(kernc, nx);
      kernc_dirtyOnCpu = true;
      nx = e_y->size[1] - 1;
      nx_dirtyOnCpu = true;
      b_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>(512LL *
        (nx + 1LL)), &b_grid, &b_block, 1024U, 65535U);
      if (b_validLaunchParams) {
        gpuEmxMemcpyCpuToGpu_creal_T(kernc, &inter_kernc, gpu_kernc);
        hipMemcpy(gpu_nx, &nx, 4ULL, hipMemcpyHostToDevice);
        nx_dirtyOnCpu = false;
        if (c_y_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_creal_T(e_y, &b_inter_y, f_gpu_y);
        }

        hipMemcpy(gpu_b, (void *)&b[0], 1024ULL, hipMemcpyHostToDevice);
        b_dirtyOnCpu = false;
        w_dftregistration_coderGPU_kern<<<b_grid, b_block>>>(f_gpu_y, d_gpu_y,
          *gpu_b, gpu_nx, gpu_kernc);
        kernc_dirtyOnCpu = false;
        kernc_dirtyOnGpu = true;
      }

      emxFree_creal_T(&e_y);
      gpuEmxFree_creal_T(&b_inter_y);
      if (kernc_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(kernc, &inter_kernc, gpu_kernc);
      }

      if (nx_dirtyOnCpu) {
        hipMemcpy(gpu_nx, &nx, 4ULL, hipMemcpyHostToDevice);
      }

      x_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (gpu_kernc, gpu_nx);
      hipMemcpy(&nx, gpu_nx, 4ULL, hipMemcpyDeviceToHost);
      c_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((nx - 1) +
        1LL), &c_grid, &c_block, 1024U, 65535U);
      if (c_validLaunchParams) {
        y_dftregistration_coderGPU_kern<<<c_grid, c_block>>>(gpu_nx, gpu_kernc);
        kernc_dirtyOnGpu = true;
      }

      emxInit_int16_T(&f_y, 2, true);
      if (usfac > 63) {
        yk = MAX_int16_T;
      } else {
        yk = static_cast<int16_T>(usfac << 9);
      }

      ab_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (yk, d_gpu_y);
      nx = static_cast<int32_T>(rt_roundd_snf(static_cast<real_T>(usfac) * 1.5));
      if (nx < 32768) {
        yk = static_cast<int16_T>(nx);
      } else {
        yk = MAX_int16_T;
      }

      vstride = yk;
      nx = f_y->size[0] * f_y->size[1];
      f_y->size[0] = 1;
      f_y->size[1] = yk;
      emxEnsureCapacity_int16_T(f_y, nx);
      f_y->data[0] = 0;
      yk = 0;
      for (k = 0; k <= vstride - 2; k++) {
        yk = static_cast<int16_T>(yk + 1);
        f_y->data[k + 1] = yk;
      }

      emxInit_creal_T(&d_z, 1, true);
      z_dirtyOnGpu = false;
      nx = d_z->size[0];
      d_z->size[0] = f_y->size[1];
      emxEnsureCapacity_creal_T(d_z, nx);
      z_dirtyOnCpu = true;
      d_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((f_y->
        size[1] - 1) + 1LL), &d_grid, &d_block, 1024U, 65535U);
      if (d_validLaunchParams) {
        gpuEmxMemcpyCpuToGpu_int16_T(f_y, &c_inter_y, g_gpu_y);
        gpuEmxMemcpyCpuToGpu_creal_T(d_z, &inter_z, gpu_z);
        bb_dftregistration_coderGPU_ker<<<d_grid, d_block>>>(d_gpu_y, cloc,
          g_gpu_y, gpu_z);
        z_dirtyOnCpu = false;
        z_dirtyOnGpu = true;
      }

      emxFree_int16_T(&f_y);
      gpuEmxFree_int16_T(&c_inter_y);
      hipMemcpy(b_gpu_x, (void *)&x[0], 1024ULL, hipMemcpyHostToDevice);
      cb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
        b_gpu_x, *b_gpu_y);
      emxInit_creal_T(&kernr, 2, true);
      nx = kernr->size[0] * kernr->size[1];
      if (z_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal_T(d_z, &inter_z);
      }

      kernr->size[0] = d_z->size[0];
      kernr->size[1] = 512;
      emxEnsureCapacity_creal_T(kernr, nx);
      gpuEmxMemcpyCpuToGpu_creal_T(kernr, &inter_kernr, gpu_kernr);
      if (z_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(d_z, &inter_z, gpu_z);
      }

      db_dftregistration_coderGPU_ker<<<dim3(32767U, 1U, 1U), dim3(512U, 1U, 1U)>>>
        (gpu_z, gpu_kernr, *b_gpu_y);
      emxFree_creal_T(&d_z);
      gpuEmxFree_creal_T(&inter_z);
      hipMemcpy(gpu_nx, &nx, 4ULL, hipMemcpyHostToDevice);
      eb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (gpu_kernr, gpu_nx);
      hipMemcpy(&nx, gpu_nx, 4ULL, hipMemcpyDeviceToHost);
      e_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((nx - 1) +
        1LL), &e_grid, &e_block, 1024U, 65535U);
      if (e_validLaunchParams) {
        fb_dftregistration_coderGPU_ker<<<e_grid, e_block>>>(gpu_nx, gpu_kernr);
      }

      emxInit_creal_T(&g_y, 2, true);
      nx = g_y->size[0] * g_y->size[1];
      gpuEmxMemcpyGpuToCpu_creal_T(kernr, &inter_kernr);
      g_y->size[0] = kernr->size[0];
      g_y->size[1] = 512;
      emxEnsureCapacity_creal_T(g_y, nx);
      y_dirtyOnCpu = true;
      f_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>
        ((kernr->size[0] * 512 - 1) + 1LL), &f_grid, &f_block, 1024U, 65535U);
      if (f_validLaunchParams) {
        gpuEmxMemcpyCpuToGpu_creal_T(g_y, &d_inter_y, gpu_y);
        gb_dftregistration_coderGPU_ker<<<f_grid, f_block>>>(gpu_kernr, gpu_y);
        y_dirtyOnCpu = false;
      }

      alpha1.re = 1.0;
      alpha1.im = 0.0;
      beta1.re = 0.0;
      beta1.im = 0.0;
      hipMemcpy(gpu_alpha1, &alpha1, 16ULL, hipMemcpyHostToDevice);
      hipMemcpy(gpu_beta1, &beta1, 16ULL, hipMemcpyHostToDevice);
      if (y_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(g_y, &d_inter_y, gpu_y);
      }

      hipblasZgemm(cublasGlobalHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, kernr->size[0],
                  512, 512, (hipDoubleComplex *)gpu_alpha1, (hipDoubleComplex *)
                  &inter_kernr.data[0], kernr->size[0], (hipDoubleComplex *)
                  &(*gpu_CC)[0], 512, (hipDoubleComplex *)gpu_beta1,
                  (hipDoubleComplex *)&d_inter_y.data[0], kernr->size[0]);
      nx = out->size[0] * out->size[1];
      gpuEmxMemcpyGpuToCpu_creal_T(g_y, &d_inter_y);
      out->size[0] = g_y->size[0];
      if (kernc_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal_T(kernc, &inter_kernc);
      }

      out->size[1] = kernc->size[1];
      emxEnsureCapacity_creal_T(out, nx);
      out_dirtyOnCpu = true;
      emxFree_creal_T(&kernr);
      gpuEmxFree_creal_T(&inter_kernr);
      g_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((g_y->
        size[0] * kernc->size[1] - 1) + 1LL), &g_grid, &g_block, 1024U, 65535U);
      if (g_validLaunchParams) {
        gpuEmxMemcpyCpuToGpu_creal_T(out, &inter_out, gpu_out);
        hb_dftregistration_coderGPU_ker<<<g_grid, g_block>>>(gpu_kernc, gpu_y,
          gpu_out);
        out_dirtyOnCpu = false;
      }

      emxInit_creal_T(&CC, 2, true);
      c_CC_dirtyOnGpu = false;
      alpha1.re = 1.0;
      alpha1.im = 0.0;
      beta1.re = 0.0;
      beta1.im = 0.0;
      hipMemcpy(gpu_alpha1, &alpha1, 16ULL, hipMemcpyHostToDevice);
      hipMemcpy(gpu_beta1, &beta1, 16ULL, hipMemcpyHostToDevice);
      if (out_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(out, &inter_out, gpu_out);
      }

      hipblasZgemm(cublasGlobalHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, g_y->size[0],
                  kernc->size[1], 512, (hipDoubleComplex *)gpu_alpha1,
                  (hipDoubleComplex *)&d_inter_y.data[0], g_y->size[0],
                  (hipDoubleComplex *)&inter_kernc.data[0], 512, (hipDoubleComplex
        *)gpu_beta1, (hipDoubleComplex *)&inter_out.data[0], g_y->size[0]);
      absar = 262144.0 * static_cast<real_T>(vspread);
      nx = CC->size[0] * CC->size[1];
      gpuEmxMemcpyGpuToCpu_creal_T(out, &inter_out);
      CC->size[0] = out->size[0];
      CC->size[1] = out->size[1];
      emxEnsureCapacity_creal_T(CC, nx);
      c_CC_dirtyOnCpu = true;
      emxFree_creal_T(&g_y);
      gpuEmxFree_creal_T(&d_inter_y);
      emxFree_creal_T(&kernc);
      gpuEmxFree_creal_T(&inter_kernc);
      h_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((out->
        size[0] * out->size[1] - 1) + 1LL), &h_grid, &h_block, 1024U, 65535U);
      if (h_validLaunchParams) {
        hipMemcpy(gpu_absar, &absar, 8ULL, hipMemcpyHostToDevice);
        gpuEmxMemcpyCpuToGpu_creal_T(CC, &inter_CC, c_gpu_CC);
        ib_dftregistration_coderGPU_ker<<<h_grid, h_block>>>(gpu_absar, gpu_out,
          c_gpu_CC);
        c_CC_dirtyOnCpu = false;
        c_CC_dirtyOnGpu = true;
      }

      emxFree_creal_T(&out);
      gpuEmxFree_creal_T(&inter_out);
      emxInit_creal_T(&b_ex, 2, true);
      ex_dirtyOnGpu = false;

      /*  Locate maximum and map back to original pixel grid  */
      nx = b_ex->size[0] * b_ex->size[1];
      b_ex->size[0] = 1;
      if (c_CC_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal_T(CC, &inter_CC);
      }

      b_ex->size[1] = static_cast<uint16_T>(CC->size[1]);
      emxEnsureCapacity_creal_T(b_ex, nx);
      gpuEmxMemcpyCpuToGpu_creal_T(b_ex, &inter_ex, b_gpu_ex);
      jb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (b_gpu_ex, *gpu_uv);
      emxInit_uint16_T(&b_idx, 2, true);
      idx_dirtyOnGpu = false;
      nx = b_idx->size[0] * b_idx->size[1];
      b_idx->size[0] = 1;
      hipMemcpy(&uv[0], gpu_uv, 4ULL, hipMemcpyDeviceToHost);
      b_idx->size[1] = uv[1];
      emxEnsureCapacity_uint16_T(b_idx, nx);
      idx_dirtyOnCpu = true;
      i_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((uv[1] - 1)
        + 1LL), &i_grid, &i_block, 1024U, 65535U);
      if (i_validLaunchParams) {
        gpuEmxMemcpyCpuToGpu_uint16_T(b_idx, &inter_idx, b_gpu_idx);
        kb_dftregistration_coderGPU_ker<<<i_grid, i_block>>>(*gpu_uv, b_gpu_idx);
        idx_dirtyOnCpu = false;
        idx_dirtyOnGpu = true;
      }

      vstride = CC->size[0];
      npages = CC->size[1];
      j_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((npages -
        1) + 1LL), &j_grid, &j_block, 1024U, 65535U);
      if (j_validLaunchParams) {
        if (c_CC_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_creal_T(CC, &inter_CC, c_gpu_CC);
          c_CC_dirtyOnCpu = false;
        }

        if (idx_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_uint16_T(b_idx, &inter_idx, b_gpu_idx);
          idx_dirtyOnCpu = false;
        }

        lb_dftregistration_coderGPU_ker<<<j_grid, j_block>>>(c_gpu_CC, vstride,
          npages, b_gpu_idx, b_gpu_ex);
        idx_dirtyOnGpu = true;
        ex_dirtyOnGpu = true;
      }

      nx = 0;
      if (ex_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal_T(b_ex, &inter_ex);
      }

      vstride = b_ex->size[1];
      rg00_im = b_ex->data[0].re;
      ex_im = b_ex->data[0].im;
      for (i = 0; i <= vstride - 2; i++) {
        if (rtIsNaN(b_ex->data[i + 1].re) || rtIsNaN(b_ex->data[i + 1].im)) {
          SCALEA = false;
        } else if (rtIsNaN(rg00_im) || rtIsNaN(ex_im)) {
          SCALEA = true;
        } else {
          if ((fabs(rg00_im) > 8.9884656743115785E+307) || (fabs(ex_im) >
               8.9884656743115785E+307)) {
            SCALEA = true;
          } else {
            SCALEA = false;
          }

          if ((fabs(b_ex->data[i + 1].re) > 8.9884656743115785E+307) || (fabs
               (b_ex->data[i + 1].im) > 8.9884656743115785E+307)) {
            SCALEB = true;
          } else {
            SCALEB = false;
          }

          if (SCALEA || SCALEB) {
            tmp_im = rt_hypotd_snf(rg00_im / 2.0, ex_im / 2.0);
            br = rt_hypotd_snf(b_ex->data[i + 1].re / 2.0, b_ex->data[i + 1].im /
                               2.0);
          } else {
            tmp_im = rt_hypotd_snf(rg00_im, ex_im);
            br = rt_hypotd_snf(b_ex->data[i + 1].re, b_ex->data[i + 1].im);
          }

          if (tmp_im == br) {
            absar = fabs(rg00_im);
            tmp_re = fabs(ex_im);
            absbr = fabs(b_ex->data[i + 1].re);
            tmp_im = fabs(b_ex->data[i + 1].im);
            if (absar > tmp_re) {
              Ma = absar;
              absar = tmp_re;
            } else {
              Ma = tmp_re;
            }

            if (absbr > tmp_im) {
              tmp_re = absbr;
              absbr = tmp_im;
            } else {
              tmp_re = tmp_im;
            }

            if (Ma > tmp_re) {
              if (absar < absbr) {
                tmp_im = Ma - tmp_re;
                br = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + tmp_re / 2.0) *
                  (absbr - absar);
              } else {
                tmp_im = Ma;
                br = tmp_re;
              }
            } else if (Ma < tmp_re) {
              if (absar > absbr) {
                br = tmp_re - Ma;
                tmp_im = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + tmp_re / 2.0)
                  * (absar - absbr);
              } else {
                tmp_im = Ma;
                br = tmp_re;
              }
            } else {
              tmp_im = absar;
              br = absbr;
            }

            if (tmp_im == br) {
              tmp_im = rt_atan2d_snf(ex_im, rg00_im);
              br = rt_atan2d_snf(b_ex->data[i + 1].im, b_ex->data[i + 1].re);
              if (tmp_im == br) {
                br = b_ex->data[i + 1].re;
                tmp_re = b_ex->data[i + 1].im;
                if (tmp_im > 0.78539816339744828) {
                  if (tmp_im > 2.3561944901923448) {
                    tmp_im = -ex_im;
                    br = -tmp_re;
                  } else {
                    tmp_im = -rg00_im;
                    br = -br;
                  }
                } else if (tmp_im > -0.78539816339744828) {
                  tmp_im = ex_im;
                  br = tmp_re;
                } else if (tmp_im > -2.3561944901923448) {
                  tmp_im = rg00_im;
                } else {
                  tmp_im = -ex_im;
                  br = -tmp_re;
                }

                if (tmp_im == br) {
                  tmp_im = 0.0;
                  br = 0.0;
                }
              }
            }
          }

          SCALEA = (tmp_im < br);
        }

        if (SCALEA) {
          rg00_im = b_ex->data[i + 1].re;
          ex_im = b_ex->data[i + 1].im;
          nx = i + 1;
        }
      }

      emxFree_creal_T(&b_ex);
      gpuEmxFree_creal_T(&inter_ex);
      hipMemcpy(gpu_nx, &nx, 4ULL, hipMemcpyHostToDevice);
      if (c_CC_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(CC, &inter_CC, c_gpu_CC);
      }

      if (idx_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_uint16_T(b_idx, &inter_idx, b_gpu_idx);
      }

      mb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (c_gpu_CC, gpu_nx, b_gpu_idx, gpu_CCmax);
      emxFree_creal_T(&CC);
      gpuEmxFree_creal_T(&inter_CC);
      nb_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
        (buf1ft, *gpu_CC);

      /*   this _coder version has calls to exist() removed to enable mex compilation */
      /*  function out=dftups(in,nor,noc,usfac,roff,coff); */
      /*  Upsampled DFT by matrix multiplies, can compute an upsampled DFT in just */
      /*  a small region. */
      /*  usfac         Upsampling factor (default usfac = 1) */
      /*  [nor,noc]     Number of pixels in the output upsampled DFT, in */
      /*                units of upsampled pixels (default = size(in)) */
      /*  roff, coff    Row and column offsets, allow to shift the output array to */
      /*                a region of interest on the DFT (default = 0) */
      /*  Recieves DC in upper left corner, image center must be in (1,1)  */
      /*  Manuel Guizar - Dec 13, 2007 */
      /*  Modified from dftus, by J.R. Fienup 7/31/06 */
      /*  This code is intended to provide the same result as if the following */
      /*  operations were performed */
      /*    - Embed the array "in" in an array that is usfac times larger in each */
      /*      dimension. ifftshift to bring the center of the image to (1,1). */
      /*    - Take the FFT of the larger array */
      /*    - Extract an [nor, noc] region of the result. Starting with the  */
      /*      [roff+1 coff+1] element. */
      /*  It achieves this result by computing the DFT in the output array without */
      /*  the need to zeropad. Much faster and memory efficient than the */
      /*  zero-padded FFT approach if [nor noc] are much smaller than [nr*usfac nc*usfac] */
      /*  Set defaults */
      /*  % % if exist('roff', 'var')~=1, roff=0; end */
      /*  % % if exist('coff', 'var')~=1, coff=0; end */
      /*  % % if exist('usfac', 'var')~=1, usfac=1; end */
      /*  % % if exist('noc', 'var')~=1, noc=nc; end */
      /*  % % if exist('nor', 'var')~=1, nor=nr; end */
      /*  Compute kernels and obtain DFT by matrix products */
      alpha1.re = 1.0;
      alpha1.im = 0.0;
      beta1.re = 0.0;
      beta1.im = 0.0;
      ob_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
        b_gpu_y, *b_gpu_kernr);
      hipMemcpy(gpu_alpha1, &alpha1, 16ULL, hipMemcpyHostToDevice);
      hipMemcpy(gpu_beta1, &beta1, 16ULL, hipMemcpyHostToDevice);
      hipblasZgemm(cublasGlobalHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 512, 512,
                  (hipDoubleComplex *)gpu_alpha1, (hipDoubleComplex *)
                  &(*b_gpu_kernr)[0], 1, (hipDoubleComplex *)&(*gpu_CC)[0], 512,
                  (hipDoubleComplex *)gpu_beta1, (hipDoubleComplex *)&(*b_gpu_y)[0],
                  1);
      y_dirtyOnGpu = true;
      pb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (d_gpu_y);
      b_y_dirtyOnGpu = true;
      for (iv0 = 0; iv0 < 512; iv0++) {
        if (y_dirtyOnGpu) {
          hipMemcpy(&y[0], b_gpu_y, 8192ULL, hipMemcpyDeviceToHost);
          y_dirtyOnGpu = false;
        }

        if (b_y_dirtyOnGpu) {
          hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
          b_y_dirtyOnGpu = false;
        }

        b_y.re += y[iv0].re - y[iv0].im * 0.0;
        b_y.im += y[iv0].re * 0.0 + y[iv0].im;
        b_y_dirtyOnCpu = true;
      }

      br = 262144.0 * static_cast<real_T>(vspread);
      if (b_y_dirtyOnGpu) {
        hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
      }

      if (b_y.im == 0.0) {
        Ma = b_y.re / br;
        rg00_im = 0.0;
      } else if (b_y.re == 0.0) {
        Ma = 0.0;
        rg00_im = b_y.im / br;
      } else {
        Ma = b_y.re / br;
        rg00_im = b_y.im / br;
      }

      qb_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
        (buf2ft, *gpu_CC);

      /*   this _coder version has calls to exist() removed to enable mex compilation */
      /*  function out=dftups(in,nor,noc,usfac,roff,coff); */
      /*  Upsampled DFT by matrix multiplies, can compute an upsampled DFT in just */
      /*  a small region. */
      /*  usfac         Upsampling factor (default usfac = 1) */
      /*  [nor,noc]     Number of pixels in the output upsampled DFT, in */
      /*                units of upsampled pixels (default = size(in)) */
      /*  roff, coff    Row and column offsets, allow to shift the output array to */
      /*                a region of interest on the DFT (default = 0) */
      /*  Recieves DC in upper left corner, image center must be in (1,1)  */
      /*  Manuel Guizar - Dec 13, 2007 */
      /*  Modified from dftus, by J.R. Fienup 7/31/06 */
      /*  This code is intended to provide the same result as if the following */
      /*  operations were performed */
      /*    - Embed the array "in" in an array that is usfac times larger in each */
      /*      dimension. ifftshift to bring the center of the image to (1,1). */
      /*    - Take the FFT of the larger array */
      /*    - Extract an [nor, noc] region of the result. Starting with the  */
      /*      [roff+1 coff+1] element. */
      /*  It achieves this result by computing the DFT in the output array without */
      /*  the need to zeropad. Much faster and memory efficient than the */
      /*  zero-padded FFT approach if [nor noc] are much smaller than [nr*usfac nc*usfac] */
      /*  Set defaults */
      /*  % % if exist('roff', 'var')~=1, roff=0; end */
      /*  % % if exist('coff', 'var')~=1, coff=0; end */
      /*  % % if exist('usfac', 'var')~=1, usfac=1; end */
      /*  % % if exist('noc', 'var')~=1, noc=nc; end */
      /*  % % if exist('nor', 'var')~=1, nor=nr; end */
      /*  Compute kernels and obtain DFT by matrix products */
      alpha1.re = 1.0;
      alpha1.im = 0.0;
      beta1.re = 0.0;
      beta1.im = 0.0;
      rb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
        b_gpu_y, *b_gpu_kernr);
      hipMemcpy(gpu_alpha1, &alpha1, 16ULL, hipMemcpyHostToDevice);
      hipMemcpy(gpu_beta1, &beta1, 16ULL, hipMemcpyHostToDevice);
      hipblasZgemm(cublasGlobalHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, 512, 512,
                  (hipDoubleComplex *)gpu_alpha1, (hipDoubleComplex *)
                  &(*b_gpu_kernr)[0], 1, (hipDoubleComplex *)&(*gpu_CC)[0], 512,
                  (hipDoubleComplex *)gpu_beta1, (hipDoubleComplex *)&(*b_gpu_y)[0],
                  1);
      y_dirtyOnGpu = true;
      if (b_y_dirtyOnCpu) {
        hipMemcpy(d_gpu_y, &b_y, 16ULL, hipMemcpyHostToDevice);
        b_y_dirtyOnCpu = false;
      }

      sb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (d_gpu_y);
      b_y_dirtyOnGpu = true;
      for (iv0 = 0; iv0 < 512; iv0++) {
        if (y_dirtyOnGpu) {
          hipMemcpy(&y[0], b_gpu_y, 8192ULL, hipMemcpyDeviceToHost);
          y_dirtyOnGpu = false;
        }

        if (b_y_dirtyOnGpu) {
          hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
          b_y_dirtyOnGpu = false;
        }

        b_y.re += y[iv0].re - y[iv0].im * 0.0;
        b_y.im += y[iv0].re * 0.0 + y[iv0].im;
        b_y_dirtyOnCpu = true;
      }

      br = 262144.0 * static_cast<real_T>(vspread);
      if (b_y_dirtyOnGpu) {
        hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
      }

      if (b_y.im == 0.0) {
        absar = b_y.re / br;
        tmp_re = 0.0;
        tmp_re_dirtyOnCpu = true;
      } else if (b_y.re == 0.0) {
        absar = 0.0;
        tmp_re = b_y.im / br;
        tmp_re_dirtyOnCpu = true;
      } else {
        absar = b_y.re / br;
        tmp_re = b_y.im / br;
        tmp_re_dirtyOnCpu = true;
      }

      if (idx_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_uint16_T(b_idx, &inter_idx);
      }

      yk = static_cast<int16_T>((b_idx->data[nx] - c_z) - 1);
      cloc = static_cast<int16_T>(nx - c_z);
      emxFree_uint16_T(&b_idx);
      gpuEmxFree_uint16_T(&inter_idx);
      if (yk >= 0) {
        c_y = static_cast<uint16_T>(yk);
      } else {
        c_y = static_cast<uint16_T>(-yk);
      }

      q = static_cast<uint16_T>(static_cast<uint32_T>(c_y) / usfac);
      c_y = static_cast<uint16_T>(static_cast<uint32_T>(c_y) -
        static_cast<uint16_T>(static_cast<uint32_T>(q) * usfac));
      if ((c_y > 0) && (c_y >= static_cast<int32_T>(static_cast<uint32_T>(usfac)
            >> 1) + (usfac & 1))) {
        q = static_cast<uint16_T>(q + 1);
      }

      c_z = static_cast<int16_T>(q);
      if (yk < 0) {
        c_z = static_cast<int16_T>(-q);
      }

      row_shift = z + c_z;
      row_shift_dirtyOnGpu = false;
      row_shift_dirtyOnCpu = true;
      if (cloc >= 0) {
        c_y = static_cast<uint16_T>(cloc);
      } else {
        c_y = static_cast<uint16_T>(-cloc);
      }

      q = static_cast<uint16_T>(static_cast<uint32_T>(c_y) / usfac);
      c_y = static_cast<uint16_T>(static_cast<uint32_T>(c_y) -
        static_cast<uint16_T>(static_cast<uint32_T>(q) * usfac));
      if ((c_y > 0) && (c_y >= static_cast<int32_T>(static_cast<uint32_T>(usfac)
            >> 1) + (usfac & 1))) {
        q = static_cast<uint16_T>(q + 1);
      }

      z = static_cast<int16_T>(q);
      if (cloc < 0) {
        z = static_cast<int16_T>(-q);
      }

      col_shift = b_z + z;

      /*  If upsampling = 2, no additional pixel shift refinement */
    } else {
      n_dftregistration_coderGPU_kern<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
        (buf1ft, *gpu_CC);
      o_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
        b_gpu_y, *gpu_CC);
      y_dirtyOnGpu = true;
      p_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (d_gpu_y);
      b_y_dirtyOnGpu = true;
      for (iv0 = 0; iv0 < 512; iv0++) {
        if (y_dirtyOnGpu) {
          hipMemcpy(&y[0], b_gpu_y, 8192ULL, hipMemcpyDeviceToHost);
          y_dirtyOnGpu = false;
        }

        if (b_y_dirtyOnGpu) {
          hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
          b_y_dirtyOnGpu = false;
        }

        b_y.re += y[iv0].re;
        b_y.im += y[iv0].im;
        b_y_dirtyOnCpu = true;
      }

      if (b_y_dirtyOnGpu) {
        hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
      }

      if (b_y.im == 0.0) {
        tmp_re = b_y.re / 1024.0;
        tmp_im = 0.0;
      } else if (b_y.re == 0.0) {
        tmp_re = 0.0;
        tmp_im = b_y.im / 1024.0;
      } else {
        tmp_re = b_y.re / 1024.0;
        tmp_im = b_y.im / 1024.0;
      }

      if (tmp_im == 0.0) {
        Ma = tmp_re / 1024.0;
        rg00_im = 0.0;
      } else if (tmp_re == 0.0) {
        Ma = 0.0;
        rg00_im = tmp_im / 1024.0;
      } else {
        Ma = tmp_re / 1024.0;
        rg00_im = tmp_im / 1024.0;
      }

      q_dftregistration_coderGPU_kern<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
        (buf2ft, *gpu_CC);
      r_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
        b_gpu_y, *gpu_CC);
      y_dirtyOnGpu = true;
      if (b_y_dirtyOnCpu) {
        hipMemcpy(d_gpu_y, &b_y, 16ULL, hipMemcpyHostToDevice);
        b_y_dirtyOnCpu = false;
      }

      s_dftregistration_coderGPU_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (d_gpu_y);
      b_y_dirtyOnGpu = true;
      for (iv0 = 0; iv0 < 512; iv0++) {
        if (y_dirtyOnGpu) {
          hipMemcpy(&y[0], b_gpu_y, 8192ULL, hipMemcpyDeviceToHost);
          y_dirtyOnGpu = false;
        }

        if (b_y_dirtyOnGpu) {
          hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
          b_y_dirtyOnGpu = false;
        }

        b_y.re += y[iv0].re;
        b_y.im += y[iv0].im;
        b_y_dirtyOnCpu = true;
      }

      if (b_y_dirtyOnGpu) {
        hipMemcpy(&b_y, d_gpu_y, 16ULL, hipMemcpyDeviceToHost);
      }

      if (b_y.im == 0.0) {
        tmp_re = b_y.re / 1024.0;
        tmp_im = 0.0;
      } else if (b_y.re == 0.0) {
        tmp_re = 0.0;
        tmp_im = b_y.im / 1024.0;
      } else {
        tmp_re = b_y.re / 1024.0;
        tmp_im = b_y.im / 1024.0;
      }

      if (tmp_im == 0.0) {
        absar = tmp_re / 1024.0;
        tmp_re = 0.0;
        tmp_re_dirtyOnCpu = true;
      } else if (tmp_re == 0.0) {
        absar = 0.0;
        tmp_re = tmp_im / 1024.0;
        tmp_re_dirtyOnCpu = true;
      } else {
        absar = tmp_re / 1024.0;
        tmp_re = tmp_im / 1024.0;
        tmp_re_dirtyOnCpu = true;
      }
    }

    hipMemcpy(&CCmax, gpu_CCmax, 16ULL, hipMemcpyDeviceToHost);
    ex_im = CCmax.re * CCmax.re - CCmax.im * -CCmax.im;
    CCmax_im = CCmax.re * -CCmax.im + CCmax.im * CCmax.re;
    absbr = Ma * absar - rg00_im * tmp_re;
    rg00_im = Ma * tmp_re + rg00_im * absar;
    if (rg00_im == 0.0) {
      if (CCmax_im == 0.0) {
        Ma = ex_im / absbr;
        CCmax_im = 0.0;
      } else if (ex_im == 0.0) {
        Ma = 0.0;
        CCmax_im /= absbr;
      } else {
        Ma = ex_im / absbr;
        CCmax_im /= absbr;
      }
    } else if (absbr == 0.0) {
      if (ex_im == 0.0) {
        Ma = CCmax_im / rg00_im;
        CCmax_im = 0.0;
      } else if (CCmax_im == 0.0) {
        Ma = 0.0;
        CCmax_im = -(ex_im / rg00_im);
      } else {
        Ma = CCmax_im / rg00_im;
        CCmax_im = -(ex_im / rg00_im);
      }
    } else {
      absar = fabs(absbr);
      tmp_re = fabs(rg00_im);
      if (absar > tmp_re) {
        tmp_im = rg00_im / absbr;
        tmp_re = absbr + tmp_im * rg00_im;
        Ma = (ex_im + tmp_im * CCmax_im) / tmp_re;
        CCmax_im = (CCmax_im - tmp_im * ex_im) / tmp_re;
      } else if (tmp_re == absar) {
        if (absbr > 0.0) {
          tmp_im = 0.5;
        } else {
          tmp_im = -0.5;
        }

        if (rg00_im > 0.0) {
          tmp_re = 0.5;
        } else {
          tmp_re = -0.5;
        }

        Ma = (ex_im * tmp_im + CCmax_im * tmp_re) / absar;
        CCmax_im = (CCmax_im * tmp_im - ex_im * tmp_re) / absar;
      } else {
        tmp_im = absbr / rg00_im;
        tmp_re = rg00_im + tmp_im * absbr;
        Ma = (tmp_im * ex_im + CCmax_im) / tmp_re;
        CCmax_im = (tmp_im * CCmax_im - ex_im) / tmp_re;
      }
    }

    tmp_im = rt_atan2d_snf(CCmax.im, CCmax.re);

    /*  If its only one row or column the shift along that dimension has no */
    /*  effect. We set to zero. */
    if (row_shift_dirtyOnCpu) {
      hipMemcpy(gpu_row_shift, &row_shift, 8ULL, hipMemcpyHostToDevice);
    }

    hipMemcpy(gpu_tmp_im, &tmp_im, 8ULL, hipMemcpyHostToDevice);
    tmp_im_dirtyOnCpu = false;
    tb_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (col_shift, gpu_row_shift, gpu_tmp_im, CCmax_im, Ma, *gpu_dv1);
    nx = output->size[0] * output->size[1];
    output->size[0] = 1;
    output->size[1] = 4;
    emxEnsureCapacity_real_T(output, nx);
    gpuEmxMemcpyCpuToGpu_real_T(output, &inter_output, gpu_output);
    ub_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_dv1, gpu_output);
  }

  if (b_y_dirtyOnCpu) {
    hipMemcpy(d_gpu_y, &b_y, 16ULL, hipMemcpyHostToDevice);
  }

  if (tmp_im_dirtyOnCpu) {
    hipMemcpy(gpu_tmp_im, &tmp_im, 8ULL, hipMemcpyHostToDevice);
  }

  vc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (gpu_tmp_im, d_gpu_y);
  if (tmp_im == 0.0) {
    if (tmp_re_dirtyOnCpu) {
      hipMemcpy(gpu_tmp_re, &tmp_re, 8ULL, hipMemcpyHostToDevice);
    }

    yc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (d_gpu_y, gpu_tmp_re);
    ad_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (gpu_tmp_re, d_gpu_y);
  } else {
    if (tmp_re_dirtyOnCpu) {
      hipMemcpy(gpu_tmp_re, &tmp_re, 8ULL, hipMemcpyHostToDevice);
    }

    wc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (d_gpu_y, gpu_tmp_re);
    xc_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (gpu_tmp_im, gpu_tmp_re, d_gpu_y);
  }

  bd_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (d_gpu_y, buf2ft, Greg);

  /*  Compute registered version of buf2ft */
  if (usfac > 0) {
    if (b_dirtyOnCpu) {
      hipMemcpy(gpu_b, (void *)&b[0], 1024ULL, hipMemcpyHostToDevice);
    }

    cd_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_b, *gpu_Nr, *gpu_Nc);
    if (row_shift_dirtyOnGpu) {
      hipMemcpy(&row_shift, gpu_row_shift, 8ULL, hipMemcpyDeviceToHost);
    }

    row_shift = -row_shift;
    hipMemcpy(gpu_row_shift, &row_shift, 8ULL, hipMemcpyHostToDevice);
    dd_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*
      gpu_Nc, col_shift, *gpu_Nr, gpu_row_shift, Greg);
    ed_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (Greg);
    fd_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (gpu_tmp_im, d_gpu_y);
    if (tmp_im == 0.0) {
      id_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (d_gpu_y, gpu_tmp_re);
      jd_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (gpu_tmp_re, d_gpu_y);
    } else {
      gd_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (d_gpu_y, gpu_tmp_re);
      hd_dftregistration_coderGPU_ker<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (gpu_tmp_im, gpu_tmp_re, d_gpu_y);
    }

    kd_dftregistration_coderGPU_ker<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (d_gpu_y, buf2ft, Greg);
  }

  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  cufftEnsureDestruction();
  gpuEmxMemcpyGpuToCpu_real_T(output, &inter_output);
  hipFree(gpu_nx);
  hipFree(*gpu_CC);
  hipFree(gpu_tmp_re);
  hipFree(gpu_tmp_im);
  hipFree(*gpu_x);
  hipFree(*b_gpu_CC);
  hipFree(*c_gpu_y);
  hipFree(*gpu_idx);
  hipFree(*gpu_ex);
  hipFree(gpu_absar);
  hipFree(gpu_row_shift);
  hipFree(*b_gpu_y);
  hipFree(d_gpu_y);
  hipFree(e_gpu_y);
  hipFree(f_gpu_y);
  hipFree(gpu_kernc);
  hipFree(*gpu_b);
  hipFree(g_gpu_y);
  hipFree(gpu_z);
  hipFree(*b_gpu_x);
  hipFree(gpu_kernr);
  hipFree(gpu_y);
  hipFree(gpu_alpha1);
  hipFree(gpu_beta1);
  hipFree(gpu_out);
  hipFree(c_gpu_CC);
  hipFree(b_gpu_ex);
  hipFree(b_gpu_idx);
  hipFree(*gpu_uv);
  hipFree(gpu_CCmax);
  hipFree(*b_gpu_kernr);
  gpuEmxFree_real_T(&inter_output);
  hipFree(gpu_output);
  hipFree(*gpu_dv1);
  hipFree(*gpu_y1);
  hipFree(*c_gpu_idx);
  hipFree(*gpu_absb1);
  hipFree(*gpu_absb2);
  hipFree(*gpu_dv);
  hipFree(*gpu_Nr);
  hipFree(*gpu_Nc);
}

/* End of code generation (dftregistration_coderGPU.cu) */
