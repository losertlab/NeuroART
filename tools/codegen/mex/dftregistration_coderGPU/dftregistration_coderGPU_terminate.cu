/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * dftregistration_coderGPU_terminate.cu
 *
 * Code generation for function 'dftregistration_coderGPU_terminate'
 *
 */

/* Include files */
#include "dftregistration_coderGPU_terminate.h"
#include "_coder_dftregistration_coderGPU_mex.h"
#include "dftregistration_coderGPU.h"
#include "dftregistration_coderGPU_data.h"
#include "rt_nonfinite.h"

/* Function Declarations */
static void cublasEnsureDestruction();

/* Function Definitions */
static void cublasEnsureDestruction()
{
  if (cublasGlobalHandle != NULL) {
    hipblasDestroy(cublasGlobalHandle);
    cublasGlobalHandle = NULL;
  }
}

void dftregistration_coderGPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void dftregistration_coderGPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  cublasEnsureDestruction();
}

/* End of code generation (dftregistration_coderGPU_terminate.cu) */
