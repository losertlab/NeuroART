/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * regseqRT_coderGPU_terminate.cu
 *
 * Code generation for function 'regseqRT_coderGPU_terminate'
 *
 */

/* Include files */
#include "regseqRT_coderGPU_terminate.h"
#include "_coder_regseqRT_coderGPU_mex.h"
#include "regseqRT_coderGPU.h"
#include "regseqRT_coderGPU_data.h"
#include "rt_nonfinite.h"

/* Function Definitions */
void regseqRT_coderGPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void regseqRT_coderGPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

/* End of code generation (regseqRT_coderGPU_terminate.cu) */
