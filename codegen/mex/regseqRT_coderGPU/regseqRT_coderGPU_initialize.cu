/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * regseqRT_coderGPU_initialize.cu
 *
 * Code generation for function 'regseqRT_coderGPU_initialize'
 *
 */

/* Include files */
#include "regseqRT_coderGPU_initialize.h"
#include "_coder_regseqRT_coderGPU_mex.h"
#include "regseqRT_coderGPU.h"
#include "regseqRT_coderGPU_data.h"
#include "rt_nonfinite.h"

/* Function Definitions */
void regseqRT_coderGPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Image_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

/* End of code generation (regseqRT_coderGPU_initialize.cu) */
