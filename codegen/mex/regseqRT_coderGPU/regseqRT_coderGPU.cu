#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * regseqRT_coderGPU.cu
 *
 * Code generation for function 'regseqRT_coderGPU'
 *
 */

/* Include files */
#include "regseqRT_coderGPU.h"
#include "MWCudaDimUtility.h"
#include "MWLaunchParametersUtilities.h"
#include "hip/hip_math_constants.h"
#include "rt_defines.h"
#include "rt_nonfinite.h"
#include <math.h>

/* Type Definitions */
#include "hipfft/hipfft.h"

/* Variable Definitions */
static hipfftHandle cufftGlobalHandle;
static hipfftHandle *cufftGlobalHandlePtr = NULL;
static int32_T cufftGlobalSize = 0U;
static int32_T cufftGlobalBatch = 0U;
static hipfftType_t cufftGlobalType = HIPFFT_R2C;
static int32_T cufftGlobalDist = 0U;

/* Function Declarations */
static __device__ real_T b_rt_atan2d_snf(real_T u0, real_T u1);
static __device__ real_T b_rt_hypotd_snf(real_T u0, real_T u1);
static void cufftEnsureDestruction();
static void cufftEnsureInitialization(int32_T nelem, hipfftType_t type, int32_T
  batch, int32_T idist);
static __global__ void regseqRT_coderGPU_kernel1(const real_T currentFrame
  [262144], real_T dstYWorld[262144]);
static __global__ void regseqRT_coderGPU_kernel10(int16_T idx[512], creal_T X
  [262144], creal_T ex[512]);
static __global__ void regseqRT_coderGPU_kernel11(const int16_T idx[512],
  int32_T *absx11);
static __global__ void regseqRT_coderGPU_kernel12(const int32_T *absx11, const
  int32_T itmp, int16_T NonFillOutputLocY[2]);
static __global__ void regseqRT_coderGPU_kernel13(const int16_T
  NonFillOutputLocY[2], real_T z1[2]);
static __global__ void regseqRT_coderGPU_kernel14(const int8_T iv[3], const
  int8_T iv1[3], int16_T A[9]);
static __global__ void regseqRT_coderGPU_kernel15(const int16_T
  NonFillOutputLocY[2], int16_T A[9]);
static __global__ void regseqRT_coderGPU_kernel16(int16_T A[9]);
static __global__ void regseqRT_coderGPU_kernel17(real_T dstYWorld[262144],
  real_T dstXWorld[262144]);
static __global__ void regseqRT_coderGPU_kernel18(const int16_T A[9], real_T x[9]);
static __global__ void regseqRT_coderGPU_kernel19(const int16_T A[9], real_T x[9]);
static __global__ void regseqRT_coderGPU_kernel2(creal_T b_y1[262144]);
static __global__ void regseqRT_coderGPU_kernel20(const int16_T A[9], real_T x[9]);
static __global__ void regseqRT_coderGPU_kernel21(const real_T absbr, const
  real_T absar, const real_T x, const real_T br, const real_T absbi, const
  real_T absai, real_T dstYWorld[262144], real_T dstXWorld[262144], real_T
  srcXIntrinsic[262144]);
static __global__ void regseqRT_coderGPU_kernel22(real_T inputImagePadded[268324]);
static __global__ void regseqRT_coderGPU_kernel23(real_T inputImagePadded[268324]);
static __global__ void regseqRT_coderGPU_kernel24(const real_T currentFrame
  [262144], real_T inputImagePadded[268324]);
static __global__ void regseqRT_coderGPU_kernel25(const real_T inputImagePadded
  [268324], const real_T dstXWorld[262144], const real_T srcXIntrinsic[262144],
  real_T regFrame[262144]);
static __global__ void regseqRT_coderGPU_kernel26(real_T regFrame[262144]);
static __global__ void regseqRT_coderGPU_kernel27(const int16_T
  NonFillOutputLocY[2], int16_T *b_NonFillOutputLocY, int16_T
  *c_NonFillOutputLocY);
static __global__ void regseqRT_coderGPU_kernel28(const int16_T
  *NonFillOutputLocY, const int16_T *b_NonFillOutputLocY, int16_T VWorld[2],
  int16_T UWorld[2]);
static __global__ void regseqRT_coderGPU_kernel29(const int16_T
  NonFillOutputLocY[2], int16_T *b_NonFillOutputLocY, int16_T
  *c_NonFillOutputLocY);
static __global__ void regseqRT_coderGPU_kernel3(creal_T b_y1[262144], creal_T
  X[262144]);
static __global__ void regseqRT_coderGPU_kernel30(const int16_T
  *NonFillOutputLocY, const int16_T UWorld[2], const int16_T
  *b_NonFillOutputLocY, const int16_T UWorldClippedToBounds[2], real_T outputC[2],
  real_T z1[2], boolean_T containedSubscripts[2], int16_T c_NonFillOutputLocY[2],
  int16_T VWorld[2]);
static __global__ void regseqRT_coderGPU_kernel31(const int16_T VWorld[2], const
  int8_T tmp_data[2], const int32_T tmp_size[2], real_T rPix_data[2]);
static __global__ void regseqRT_coderGPU_kernel32(const int16_T
  NonFillOutputLocY[2], const int8_T tmp_data[2], const int32_T tmp_size[2],
  real_T rInit_data[2]);
static __global__ void regseqRT_coderGPU_kernel33(const int32_T *absx11, real_T
  rPix_data[2]);
static __global__ void regseqRT_coderGPU_kernel34(const real_T rPix_data[2],
  const int32_T *absx11, real_T cPix_data[2]);
static __global__ void regseqRT_coderGPU_kernel35(const int32_T *absx11, real_T
  rInit_data[2]);
static __global__ void regseqRT_coderGPU_kernel36(const real_T rInit_data[2],
  const int32_T *absx11, real_T rPix_data[2]);
static __global__ void regseqRT_coderGPU_kernel37(const real_T currentFrame
  [262144], const int32_T p2, const int32_T *absx11, const int32_T itmp, const
  int32_T absx31, const int32_T p1, int32_T p3, real_T regFrame[262144]);
static __global__ void regseqRT_coderGPU_kernel4(const creal_T b_template[262144],
  creal_T f[262144]);
static __global__ void regseqRT_coderGPU_kernel5(creal_T X[262144]);
static __global__ void regseqRT_coderGPU_kernel6(creal_T X[262144], creal_T f
  [262144]);
static __global__ void regseqRT_coderGPU_kernel7(creal_T b_y1[262144]);
static __global__ void regseqRT_coderGPU_kernel8(creal_T b_y1[262144], creal_T
  X[262144]);
static __global__ void regseqRT_coderGPU_kernel9(int16_T idx[512]);
static real_T rt_atan2d_snf(real_T u0, real_T u1);
static real_T rt_hypotd_snf(real_T u0, real_T u1);
static __device__ real_T rt_roundd_snf(real_T u);

/* Function Definitions */
static __device__ real_T b_rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  int32_T b_u0;
  int32_T b_u1;
  if ((static_cast<int32_T>(isnan(u0))) || (static_cast<int32_T>(isnan(u1)))) {
    y = HIP_NAN;
  } else if ((static_cast<int32_T>(isinf(u0))) && (static_cast<int32_T>(isinf(u1))))
  {
    if (u0 > 0.0) {
      b_u0 = 1;
    } else {
      b_u0 = -1;
    }

    if (u1 > 0.0) {
      b_u1 = 1;
    } else {
      b_u1 = -1;
    }

    y = atan2(static_cast<real_T>(b_u0), static_cast<real_T>(b_u1));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = atan2(u0, u1);
  }

  return y;
}

static __device__ real_T b_rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T y;
  real_T a;
  real_T b;
  a = fabs(u0);
  b = fabs(u1);
  if (a < b) {
    a /= b;
    y = b * sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * sqrt(b * b + 1.0);
  } else if (isnan(b)) {
    y = b;
  } else {
    y = a * 1.4142135623730951;
  }

  return y;
}

static void cufftEnsureDestruction()
{
  if (cufftGlobalHandlePtr != NULL) {
    hipfftDestroy(*cufftGlobalHandlePtr);
    cufftGlobalHandlePtr = NULL;
    cufftGlobalSize = 0;
    cufftGlobalBatch = 0;
    cufftGlobalType = HIPFFT_R2C;
    cufftGlobalDist = 0;
  }
}

static void cufftEnsureInitialization(int32_T nelem, hipfftType_t type, int32_T
  batch, int32_T idist)
{
  if ((cufftGlobalDist != idist) || (cufftGlobalType != type) ||
      (cufftGlobalSize != nelem) || (cufftGlobalBatch != batch)) {
    if (cufftGlobalHandlePtr != NULL) {
      cufftEnsureDestruction();
    }

    cufftGlobalHandlePtr = &cufftGlobalHandle;
    hipfftPlanMany(cufftGlobalHandlePtr, 1U, &nelem, &idist, 1, idist, &nelem, 1,
                  nelem, type, batch);
    cufftGlobalSize = nelem;
    cufftGlobalBatch = batch;
    cufftGlobalType = type;
    cufftGlobalDist = idist;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel1(const
  real_T currentFrame[262144], real_T dstYWorld[262144])
{
  uint32_T threadId;
  int32_T iy;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iy = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iy)) / 512U);
  if (i < 512) {
    dstYWorld[iy + (i << 9)] = currentFrame[i + (iy << 9)];
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel10
  (int16_T idx[512], creal_T X[262144], creal_T ex[512])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T iy;
  int32_T i;
  int32_T ix;
  boolean_T p;
  boolean_T SCALEA;
  real_T qx2;
  real_T qx1;
  real_T absar;
  real_T ry;
  real_T absbr;
  real_T Ma;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iy = static_cast<int32_T>(threadId);
  if (iy < 512) {
    iv0 = (iy * 511 + iy) + 1;
    ex[iy] = X[iv0 - 1];
    for (i = 0; i < 511; i++) {
      ix = (iv0 + i) + 1;
      if ((static_cast<int32_T>(isnan(X[ix - 1].re))) || (static_cast<int32_T>
           (isnan(X[ix - 1].im)))) {
        p = false;
      } else if ((static_cast<int32_T>(isnan(ex[iy].re))) ||
                 (static_cast<int32_T>(isnan(ex[iy].im)))) {
        p = true;
      } else {
        if ((static_cast<int32_T>(fabs(ex[iy].re) > 8.9884656743115785E+307)) ||
            (static_cast<int32_T>(fabs(ex[iy].im) > 8.9884656743115785E+307))) {
          SCALEA = true;
        } else {
          SCALEA = false;
        }

        if ((static_cast<int32_T>(fabs(X[ix - 1].re) > 8.9884656743115785E+307))
            || (static_cast<int32_T>(fabs(X[ix - 1].im) >
              8.9884656743115785E+307))) {
          p = true;
        } else {
          p = false;
        }

        if ((static_cast<int32_T>(SCALEA)) || (static_cast<int32_T>(p))) {
          qx2 = b_rt_hypotd_snf(ex[iy].re / 2.0, ex[iy].im / 2.0);
          qx1 = b_rt_hypotd_snf(X[ix - 1].re / 2.0, X[ix - 1].im / 2.0);
        } else {
          qx2 = b_rt_hypotd_snf(ex[iy].re, ex[iy].im);
          qx1 = b_rt_hypotd_snf(X[ix - 1].re, X[ix - 1].im);
        }

        if (qx2 == qx1) {
          absar = fabs(ex[iy].re);
          ry = fabs(ex[iy].im);
          absbr = fabs(X[ix - 1].re);
          qx2 = fabs(X[ix - 1].im);
          if (absar > ry) {
            Ma = absar;
            absar = ry;
          } else {
            Ma = ry;
          }

          if (absbr > qx2) {
            ry = absbr;
            absbr = qx2;
          } else {
            ry = qx2;
          }

          if (Ma > ry) {
            if (absar < absbr) {
              qx2 = Ma - ry;
              qx1 = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + ry / 2.0) * (absbr
                - absar);
            } else {
              qx2 = Ma;
              qx1 = ry;
            }
          } else if (Ma < ry) {
            if (absar > absbr) {
              qx1 = ry - Ma;
              qx2 = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + ry / 2.0) * (absar
                - absbr);
            } else {
              qx2 = Ma;
              qx1 = ry;
            }
          } else {
            qx2 = absar;
            qx1 = absbr;
          }

          if (qx2 == qx1) {
            qx2 = b_rt_atan2d_snf(ex[iy].im, ex[iy].re);
            qx1 = b_rt_atan2d_snf(X[ix - 1].im, X[ix - 1].re);
            if (qx2 == qx1) {
              qx1 = X[ix - 1].re;
              ry = X[ix - 1].im;
              if (qx2 > 0.78539816339744828) {
                if (qx2 > 2.3561944901923448) {
                  qx2 = -ex[iy].im;
                  qx1 = -ry;
                } else {
                  qx2 = -ex[iy].re;
                  qx1 = -qx1;
                }
              } else if (qx2 > -0.78539816339744828) {
                qx2 = ex[iy].im;
                qx1 = ry;
              } else if (qx2 > -2.3561944901923448) {
                qx2 = ex[iy].re;
              } else {
                qx2 = -ex[iy].im;
                qx1 = -ry;
              }

              if (qx2 == qx1) {
                qx2 = 0.0;
                qx1 = 0.0;
              }
            }
          }
        }

        p = (qx2 < qx1);
      }

      if (p) {
        ex[iy] = X[ix - 1];
        idx[iy] = static_cast<int16_T>(i + 2);
      }
    }
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel11(const
  int16_T idx[512], int32_T *absx11)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    if (static_cast<int32_T>(idx[*absx11]) > 256) {
      *absx11 = static_cast<int32_T>(idx[*absx11]) - 513;
    } else {
      *absx11 = static_cast<int32_T>(idx[*absx11]) - 1;
    }
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel12(const
  int32_T *absx11, const int32_T itmp, int16_T NonFillOutputLocY[2])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    NonFillOutputLocY[0] = static_cast<int16_T>(itmp);
    NonFillOutputLocY[1] = static_cast<int16_T>(*absx11);
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel13(const
  int16_T NonFillOutputLocY[2], real_T z1[2])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T iy;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 2) {
    if (static_cast<int32_T>(NonFillOutputLocY[iv0]) == 0) {
      iy = 0;
    } else {
      iy = static_cast<int32_T>(fmod(static_cast<real_T>(NonFillOutputLocY[iv0]),
        1.0));
      if ((static_cast<int32_T>(iy != 0)) && (static_cast<int32_T>
           (static_cast<int32_T>(NonFillOutputLocY[iv0]) < 0))) {
        iy++;
      }
    }

    z1[iv0] = static_cast<real_T>(iy);
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel14(const
  int8_T iv[3], const int8_T iv1[3], int16_T A[9])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 3) {
    A[3 * i] = static_cast<int16_T>(iv1[i]);
    A[3 * i + 1] = static_cast<int16_T>(iv[i]);
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel15(const
  int16_T NonFillOutputLocY[2], int16_T A[9])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 2) {
    A[3 * i + 2] = NonFillOutputLocY[i];
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel16
  (int16_T A[9])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    A[8] = static_cast<int16_T>(1);
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel17
  (real_T dstYWorld[262144], real_T dstXWorld[262144])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T iy;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId % 512U);
  iy = static_cast<int32_T>((threadId - static_cast<uint32_T>(iv0)) / 512U);
  if (iy < 512) {
    dstXWorld[iv0 + (iy << 9)] = ((static_cast<real_T>(iy) + 1.0) - 0.5) + 0.5;
    dstYWorld[iv0 + (iy << 9)] = ((static_cast<real_T>(iv0) + 1.0) - 0.5) + 0.5;
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel18(const
  int16_T A[9], real_T x[9])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 9) {
    x[i] = static_cast<real_T>(A[i]);
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel19(const
  int16_T A[9], real_T x[9])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    x[0] = static_cast<real_T>(A[2]);
    x[2] = static_cast<real_T>(A[0]);
    x[3] = static_cast<real_T>(A[5]);
    x[5] = static_cast<real_T>(A[3]);
    x[6] = 1.0;
    x[8] = static_cast<real_T>(A[6]);
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel2
  (creal_T b_y1[262144])
{
  uint32_T threadId;
  int32_T iv0;
  int32_T i;
  int32_T iy;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iy = static_cast<int32_T>(threadId % 255U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iy)) / 255U);
  if (i < 512) {
    iv0 = i << 9;
    b_y1[(iv0 + iy) + 257].re = b_y1[(iv0 - iy) + 255].re;
    b_y1[(iv0 + iy) + 257].im = -b_y1[(iv0 - iy) + 255].im;
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel20(const
  int16_T A[9], real_T x[9])
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    x[0] = 1.0;
    x[1] = static_cast<real_T>(A[0]);
    x[3] = static_cast<real_T>(A[4]);
    x[4] = static_cast<real_T>(A[3]);
    x[6] = static_cast<real_T>(A[7]);
    x[7] = static_cast<real_T>(A[6]);
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel21(
  const real_T absbr, const real_T absar, const real_T x, const real_T br, const
  real_T absbi, const real_T absai, real_T dstYWorld[262144], real_T dstXWorld
  [262144], real_T srcXIntrinsic[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    srcXIntrinsic[i] = ((((absai * dstXWorld[i] + absbi * dstYWorld[i]) + br) -
                         0.5) + 0.5) + 3.0;
    dstXWorld[i] = ((((x * dstXWorld[i] + absar * dstYWorld[i]) + absbr) - 0.5)
                    + 0.5) + 3.0;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel22
  (real_T inputImagePadded[268324])
{
  uint32_T threadId;
  int32_T i;
  int32_T iy;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId % 518U);
  iy = static_cast<int32_T>((threadId - static_cast<uint32_T>(i)) / 518U);
  if (iy < 3) {
    inputImagePadded[i + 518 * iy] = 0.0;
    inputImagePadded[i + 518 * (iy + 515)] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel23
  (real_T inputImagePadded[268324])
{
  uint32_T threadId;
  int32_T i;
  int32_T iy;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId % 3U);
  iy = static_cast<int32_T>((threadId - static_cast<uint32_T>(i)) / 3U);
  if (iy < 512) {
    inputImagePadded[i + 518 * (iy + 3)] = 0.0;
    inputImagePadded[(i + 518 * (iy + 3)) + 515] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel24(
  const real_T currentFrame[262144], real_T inputImagePadded[268324])
{
  uint32_T threadId;
  int32_T i;
  int32_T iy;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId % 512U);
  iy = static_cast<int32_T>((threadId - static_cast<uint32_T>(i)) / 512U);
  if (iy < 512) {
    inputImagePadded[(i + 518 * (iy + 3)) + 3] = currentFrame[i + (iy << 9)];
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel25(
  const real_T inputImagePadded[268324], const real_T dstXWorld[262144], const
  real_T srcXIntrinsic[262144], real_T regFrame[262144])
{
  uint32_T threadId;
  int32_T iv0;
  real_T qx2;
  int32_T ix;
  int32_T iy;
  real_T qx1;
  real_T ry;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iv0 = static_cast<int32_T>(threadId);
  if (iv0 < 262144) {
    if ((static_cast<int32_T>((static_cast<int32_T>((static_cast<int32_T>
             (srcXIntrinsic[iv0] >= 1.0)) && (static_cast<int32_T>
             (srcXIntrinsic[iv0] <= 518.0)))) && (static_cast<int32_T>
           (dstXWorld[iv0] >= 1.0)))) && (static_cast<int32_T>(dstXWorld[iv0] <=
          518.0))) {
      if (srcXIntrinsic[iv0] <= 1.0) {
        ix = 1;
      } else if (srcXIntrinsic[iv0] <= 517.0) {
        ix = static_cast<int32_T>(floor(srcXIntrinsic[iv0]));
      } else {
        ix = 517;
      }

      if (dstXWorld[iv0] <= 1.0) {
        iy = 1;
      } else if (dstXWorld[iv0] <= 517.0) {
        iy = static_cast<int32_T>(floor(dstXWorld[iv0]));
      } else {
        iy = 517;
      }

      if (srcXIntrinsic[iv0] == static_cast<real_T>(ix)) {
        qx1 = inputImagePadded[(iy + 518 * (ix - 1)) - 1];
        qx2 = inputImagePadded[iy + 518 * (ix - 1)];
      } else if (srcXIntrinsic[iv0] == static_cast<real_T>(ix) + 1.0) {
        qx1 = inputImagePadded[(iy + 518 * ix) - 1];
        qx2 = inputImagePadded[iy + 518 * ix];
      } else {
        qx2 = (srcXIntrinsic[iv0] - static_cast<real_T>(ix)) /
          ((static_cast<real_T>(ix) + 1.0) - static_cast<real_T>(ix));
        if (inputImagePadded[(iy + 518 * (ix - 1)) - 1] == inputImagePadded[(iy
             + 518 * ix) - 1]) {
          qx1 = inputImagePadded[(iy + 518 * (ix - 1)) - 1];
        } else {
          qx1 = (1.0 - qx2) * inputImagePadded[(iy + 518 * (ix - 1)) - 1] + qx2 *
            inputImagePadded[(iy + 518 * ix) - 1];
        }

        if (inputImagePadded[iy + 518 * (ix - 1)] == inputImagePadded[iy + 518 *
            ix]) {
          qx2 = inputImagePadded[iy + 518 * (ix - 1)];
        } else {
          qx2 = (1.0 - qx2) * inputImagePadded[iy + 518 * (ix - 1)] + qx2 *
            inputImagePadded[iy + 518 * ix];
        }
      }

      if ((static_cast<int32_T>(dstXWorld[iv0] == static_cast<real_T>(iy))) || (
           static_cast<int32_T>(qx1 == qx2))) {
        qx2 = qx1;
      } else {
        if (!static_cast<int32_T>(dstXWorld[iv0] == static_cast<real_T>(iy) +
             1.0)) {
          ry = (dstXWorld[iv0] - static_cast<real_T>(iy)) / ((static_cast<real_T>
            (iy) + 1.0) - static_cast<real_T>(iy));
          qx2 = (1.0 - ry) * qx1 + ry * qx2;
        }
      }
    } else {
      qx2 = 0.0;
    }

    regFrame[iv0] = qx2;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel26
  (real_T regFrame[262144])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    regFrame[i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel27(const
  int16_T NonFillOutputLocY[2], int16_T *b_NonFillOutputLocY, int16_T
  *c_NonFillOutputLocY)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *c_NonFillOutputLocY = NonFillOutputLocY[0];
    *b_NonFillOutputLocY = NonFillOutputLocY[1];
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel28(const
  int16_T *NonFillOutputLocY, const int16_T *b_NonFillOutputLocY, int16_T
  VWorld[2], int16_T UWorld[2])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 2) {
    UWorld[i] = static_cast<int16_T>((511 * i - static_cast<int32_T>
      (*b_NonFillOutputLocY)) + 1);
    VWorld[i] = static_cast<int16_T>((511 * i - static_cast<int32_T>
      (*NonFillOutputLocY)) + 1);
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel29(const
  int16_T NonFillOutputLocY[2], int16_T *b_NonFillOutputLocY, int16_T
  *c_NonFillOutputLocY)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *c_NonFillOutputLocY = NonFillOutputLocY[0];
    *b_NonFillOutputLocY = NonFillOutputLocY[1];
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel3
  (creal_T b_y1[262144], creal_T X[262144])
{
  uint32_T threadId;
  int32_T iy;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iy = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iy)) / 512U);
  if (i < 512) {
    X[iy + (i << 9)] = b_y1[i + (iy << 9)];
  }
}

static __global__ __launch_bounds__(32, 1) void regseqRT_coderGPU_kernel30(const
  int16_T *NonFillOutputLocY, const int16_T UWorld[2], const int16_T
  *b_NonFillOutputLocY, const int16_T UWorldClippedToBounds[2], real_T outputC[2],
  real_T z1[2], boolean_T containedSubscripts[2], int16_T c_NonFillOutputLocY[2],
  int16_T VWorld[2])
{
  uint32_T threadId;
  int16_T i;
  int32_T b_i;
  int16_T i1;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  b_i = static_cast<int32_T>(threadId);
  if (b_i < 2) {
    i = static_cast<int16_T>(static_cast<int32_T>(UWorldClippedToBounds[b_i]) +
      static_cast<int32_T>(*b_NonFillOutputLocY));
    VWorld[b_i] = i;
    i1 = static_cast<int16_T>(static_cast<int32_T>(UWorld[b_i]) +
      static_cast<int32_T>(*NonFillOutputLocY));
    c_NonFillOutputLocY[b_i] = i1;
    containedSubscripts[b_i] = ((static_cast<int32_T>((static_cast<int32_T>((
      static_cast<int32_T>(static_cast<real_T>(static_cast<int32_T>(i)) >= 0.5))
      && (static_cast<int32_T>(static_cast<real_T>(static_cast<int32_T>(i)) <=
      512.5)))) && (static_cast<int32_T>(static_cast<real_T>(static_cast<int32_T>
      (i1)) >= 0.5)))) && (static_cast<int32_T>(static_cast<real_T>
      (static_cast<int32_T>(i1)) <= 512.5)));
    z1[b_i] = HIP_NAN;
    outputC[b_i] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel31(
  const int16_T VWorld[2], const int8_T tmp_data[2], const int32_T tmp_size[2],
  real_T rPix_data[2])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(tmp_size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    rPix_data[i] = (static_cast<real_T>(VWorld[static_cast<int32_T>(tmp_data[i])
      - 1]) - 0.5) + 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel32(
  const int16_T NonFillOutputLocY[2], const int8_T tmp_data[2], const int32_T
  tmp_size[2], real_T rInit_data[2])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(tmp_size[1] - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    i = static_cast<int32_T>(idx);
    rInit_data[i] = (static_cast<real_T>(NonFillOutputLocY[static_cast<int32_T>
      (tmp_data[i]) - 1]) - 0.5) + 0.5;
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel33(
  const int32_T *absx11, real_T rPix_data[2])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(*absx11 - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx);
    rPix_data[iv0] = rt_roundd_snf(rPix_data[iv0]);
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel34(
  const real_T rPix_data[2], const int32_T *absx11, real_T cPix_data[2])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  real_T ry;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(*absx11 - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx);
    if (rPix_data[iv0] < 512.0) {
      ry = rPix_data[iv0];
    } else {
      ry = 512.0;
    }

    cPix_data[iv0] = ry;
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel35(
  const int32_T *absx11, real_T rInit_data[2])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(*absx11 - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx);
    rInit_data[iv0] = rt_roundd_snf(rInit_data[iv0]);
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel36(
  const real_T rInit_data[2], const int32_T *absx11, real_T rPix_data[2])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iv0;
  real_T ry;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = static_cast<int64_T>(*absx11 - 1);
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iv0 = static_cast<int32_T>(idx);
    if (rInit_data[iv0] < 512.0) {
      ry = rInit_data[iv0];
    } else {
      ry = 512.0;
    }

    rPix_data[iv0] = ry;
  }
}

static __global__ __launch_bounds__(1024, 1) void regseqRT_coderGPU_kernel37(
  const real_T currentFrame[262144], const int32_T p2, const int32_T *absx11,
  const int32_T itmp, const int32_T absx31, const int32_T p1, int32_T p3, real_T
  regFrame[262144])
{
  uint32_T threadId;
  uint32_T threadStride;
  uint32_T idx;
  int32_T iy;
  int32_T i;
  int64_T loopEnd;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  threadStride = static_cast<uint32_T>(mwGetTotalThreadsLaunched());
  loopEnd = (static_cast<int64_T>(p1 - absx31) + 1LL) * (static_cast<int64_T>(p3
    - p2) + 1LL) - 1LL;
  for (idx = threadId; idx <= static_cast<uint32_T>(loopEnd); idx +=
       threadStride) {
    iy = static_cast<int32_T>(idx % (static_cast<uint32_T>(p1 - absx31) + 1U));
    i = static_cast<int32_T>((idx - static_cast<uint32_T>(iy)) /
      (static_cast<uint32_T>(p1 - absx31) + 1U));
    regFrame[(itmp + iy) + ((*absx11 + i) << 9)] = currentFrame[(absx31 + iy) +
      ((p2 + i) << 9)];
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel4(const
  creal_T b_template[262144], creal_T f[262144])
{
  uint32_T threadId;
  real_T ry;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    ry = f[i].re;
    f[i].re = b_template[i].re * f[i].re - b_template[i].im * -f[i].im;
    f[i].im = b_template[i].re * -f[i].im + b_template[i].im * ry;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel5
  (creal_T X[262144])
{
  uint32_T threadId;
  int32_T i;
  real_T ry;
  real_T qx2;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    if (X[i].im == 0.0) {
      ry = X[i].re / 512.0;
      qx2 = 0.0;
    } else if (X[i].re == 0.0) {
      ry = 0.0;
      qx2 = X[i].im / 512.0;
    } else {
      ry = X[i].re / 512.0;
      qx2 = X[i].im / 512.0;
    }

    X[i].re = ry;
    X[i].im = qx2;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel6
  (creal_T X[262144], creal_T f[262144])
{
  uint32_T threadId;
  int32_T iy;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iy = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iy)) / 512U);
  if (i < 512) {
    f[iy + (i << 9)] = X[i + (iy << 9)];
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel7
  (creal_T b_y1[262144])
{
  uint32_T threadId;
  int32_T i;
  real_T ry;
  real_T qx2;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 262144) {
    if (b_y1[i].im == 0.0) {
      ry = b_y1[i].re / 512.0;
      qx2 = 0.0;
    } else if (b_y1[i].re == 0.0) {
      ry = 0.0;
      qx2 = b_y1[i].im / 512.0;
    } else {
      ry = b_y1[i].re / 512.0;
      qx2 = b_y1[i].im / 512.0;
    }

    b_y1[i].re = ry;
    b_y1[i].im = qx2;
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel8
  (creal_T b_y1[262144], creal_T X[262144])
{
  uint32_T threadId;
  int32_T iy;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  iy = static_cast<int32_T>(threadId % 512U);
  i = static_cast<int32_T>((threadId - static_cast<uint32_T>(iy)) / 512U);
  if (i < 512) {
    X[iy + (i << 9)] = b_y1[i + (iy << 9)];
  }
}

static __global__ __launch_bounds__(512, 1) void regseqRT_coderGPU_kernel9
  (int16_T idx[512])
{
  uint32_T threadId;
  int32_T i;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    idx[i] = static_cast<int16_T>(1);
  }
}

static real_T rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  int32_T b_u0;
  int32_T b_u1;
  if (rtIsNaN(u0) || rtIsNaN(u1)) {
    y = rtNaN;
  } else if (rtIsInf(u0) && rtIsInf(u1)) {
    if (u0 > 0.0) {
      b_u0 = 1;
    } else {
      b_u0 = -1;
    }

    if (u1 > 0.0) {
      b_u1 = 1;
    } else {
      b_u1 = -1;
    }

    y = atan2(static_cast<real_T>(b_u0), static_cast<real_T>(b_u1));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = atan2(u0, u1);
  }

  return y;
}

static real_T rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T y;
  real_T a;
  real_T b;
  a = fabs(u0);
  b = fabs(u1);
  if (a < b) {
    a /= b;
    y = b * sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * sqrt(b * b + 1.0);
  } else if (rtIsNaN(b)) {
    y = b;
  } else {
    y = a * 1.4142135623730951;
  }

  return y;
}

static __device__ real_T rt_roundd_snf(real_T u)
{
  real_T y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

void regseqRT_coderGPU(const creal_T b_template[262144], const real_T
  currentFrame[262144], real_T regFrame[262144], real_T *tx, real_T *ty)
{
  creal_T ex[512];
  int32_T absx11;
  real_T ex_re;
  real_T ex_im;
  int32_T absx31;
  int32_T itmp;
  boolean_T c_locationsInSourceMapToDestina;
  real_T z1[2];
  boolean_T SCALEB;
  real_T x;
  real_T br;
  real_T absar;
  int16_T A[9];
  real_T absai;
  real_T absbr;
  real_T absbi;
  static const int8_T iv[3] = { 0, 1, 0 };

  int16_T UWorld[2];
  real_T Ma;
  static const int8_T iv1[3] = { 1, 0, 0 };

  int16_T VWorld[2];
  real_T b_x[9];
  int16_T c_intrinsicSourceBoundingRectan[4];
  int32_T p1;
  int32_T p2;
  int32_T p3;
  int16_T d_intrinsicSourceBoundingRectan[4];
  int16_T UWorldClippedToBounds[2];
  real_T tinv[9];
  boolean_T containedSubscripts[2];
  real_T outputC[2];
  int32_T tmp_size[2];
  real_T rPix_data[2];
  int8_T tmp_data[2];
  int32_T cInit_size[2];
  int32_T rInit_size[2];
  real_T cPix_data[2];
  int8_T cPix_size[2];
  int8_T rPix_size[2];
  real_T (*gpu_dstYWorld)[262144];
  creal_T (*gpu_y1)[262144];
  creal_T (*gpu_f)[262144];
  creal_T (*gpu_X)[262144];
  int16_T (*gpu_idx)[512];
  creal_T (*gpu_ex)[512];
  int32_T *gpu_absx11;
  int16_T (*gpu_NonFillOutputLocY)[2];
  real_T (*gpu_z1)[2];
  int8_T (*gpu_iv)[3];
  int8_T (*gpu_iv1)[3];
  int16_T (*gpu_A)[9];
  real_T (*gpu_dstXWorld)[262144];
  real_T (*gpu_x)[9];
  real_T (*gpu_srcXIntrinsic)[262144];
  real_T (*gpu_inputImagePadded)[268324];
  int16_T *b_gpu_NonFillOutputLocY;
  int16_T *c_gpu_NonFillOutputLocY;
  int16_T (*gpu_VWorld)[2];
  int16_T (*gpu_UWorld)[2];
  int16_T (*gpu_UWorldClippedToBounds)[2];
  real_T (*gpu_outputC)[2];
  boolean_T (*gpu_containedSubscripts)[2];
  int8_T (*gpu_tmp_data)[2];
  int32_T (*gpu_tmp_size)[2];
  dim3 grid;
  dim3 block;
  boolean_T validLaunchParams;
  real_T (*gpu_rPix_data)[2];
  dim3 b_grid;
  dim3 b_block;
  boolean_T b_validLaunchParams;
  real_T (*gpu_rInit_data)[2];
  dim3 c_grid;
  dim3 c_block;
  boolean_T c_validLaunchParams;
  dim3 d_grid;
  dim3 d_block;
  boolean_T d_validLaunchParams;
  real_T (*gpu_cPix_data)[2];
  dim3 e_grid;
  dim3 e_block;
  boolean_T e_validLaunchParams;
  dim3 f_grid;
  dim3 f_block;
  boolean_T f_validLaunchParams;
  dim3 g_grid;
  dim3 g_block;
  boolean_T g_validLaunchParams;
  boolean_T z1_dirtyOnGpu;
  boolean_T outputC_dirtyOnGpu;
  boolean_T containedSubscripts_dirtyOnGpu;
  boolean_T rPix_data_dirtyOnGpu;
  boolean_T cPix_data_dirtyOnGpu;
  boolean_T tmp_data_dirtyOnCpu;
  boolean_T tmp_size_dirtyOnCpu;
  boolean_T exitg1;
  hipMalloc(&gpu_cPix_data, 2U * sizeof(real_T));
  hipMalloc(&gpu_outputC, 16ULL);
  hipMalloc(&gpu_rInit_data, 2U * sizeof(real_T));
  hipMalloc(&gpu_rPix_data, 2U * sizeof(real_T));
  hipMalloc(&gpu_tmp_data, 2U * sizeof(int8_T));
  hipMalloc(&gpu_tmp_size, 8ULL);
  hipMalloc(&gpu_containedSubscripts, 2ULL);
  hipMalloc(&gpu_UWorldClippedToBounds, 4ULL);
  hipMalloc(&gpu_VWorld, 4ULL);
  hipMalloc(&gpu_UWorld, 4ULL);
  hipMalloc(&c_gpu_NonFillOutputLocY, 2ULL);
  hipMalloc(&b_gpu_NonFillOutputLocY, 2ULL);
  hipMalloc(&gpu_srcXIntrinsic, 2097152ULL);
  hipMalloc(&gpu_inputImagePadded, 2146592ULL);
  hipMalloc(&gpu_dstXWorld, 2097152ULL);
  hipMalloc(&gpu_x, 72ULL);
  hipMalloc(&gpu_A, 18ULL);
  hipMalloc(&gpu_iv1, 3ULL);
  hipMalloc(&gpu_iv, 3ULL);
  hipMalloc(&gpu_z1, 16ULL);
  hipMalloc(&gpu_NonFillOutputLocY, 4ULL);
  hipMalloc(&gpu_absx11, 4ULL);
  hipMalloc(&gpu_ex, 8192ULL);
  hipMalloc(&gpu_idx, 1024ULL);
  hipMalloc(&gpu_f, 4194304ULL);
  hipMalloc(&gpu_X, 4194304ULL);
  hipMalloc(&gpu_y1, 4194304ULL);
  hipMalloc(&gpu_dstYWorld, 2097152ULL);
  tmp_data_dirtyOnCpu = false;
  cPix_data_dirtyOnGpu = false;
  rPix_data_dirtyOnGpu = false;

  /*  Add kernelfun pragma to trigger GPU kernel creation */
  regseqRT_coderGPU_kernel1<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (currentFrame, *gpu_dstYWorld);
  cufftEnsureInitialization(512, HIPFFT_D2Z, 512, 512);
  hipfftExecD2Z(*cufftGlobalHandlePtr, (hipfftDoubleReal *)&(*gpu_dstYWorld)[0],
               (hipfftDoubleComplex *)&(*gpu_y1)[0]);
  regseqRT_coderGPU_kernel2<<<dim3(255U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_y1);
  regseqRT_coderGPU_kernel3<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_y1,
    *gpu_X);
  cufftEnsureInitialization(512, HIPFFT_Z2Z, 512, 512);
  hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*gpu_X)[0],
               (hipfftDoubleComplex *)&(*gpu_f)[0], HIPFFT_FORWARD);
  regseqRT_coderGPU_kernel4<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (b_template, *gpu_f);
  cufftEnsureInitialization(512, HIPFFT_Z2Z, 512, 512);
  hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*gpu_f)[0],
               (hipfftDoubleComplex *)&(*gpu_X)[0], 1);
  regseqRT_coderGPU_kernel5<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_X);
  regseqRT_coderGPU_kernel6<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_X, *
    gpu_f);
  cufftEnsureInitialization(512, HIPFFT_Z2Z, 512, 512);
  hipfftExecZ2Z(*cufftGlobalHandlePtr, (hipfftDoubleComplex *)&(*gpu_f)[0],
               (hipfftDoubleComplex *)&(*gpu_y1)[0], 1);
  regseqRT_coderGPU_kernel7<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_y1);
  regseqRT_coderGPU_kernel8<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_y1,
    *gpu_X);
  regseqRT_coderGPU_kernel9<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_idx);
  regseqRT_coderGPU_kernel10<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_idx,
    *gpu_X, *gpu_ex);

  /*  find column maximums */
  absx11 = 0;
  hipMemcpy(&ex[0], gpu_ex, 8192ULL, hipMemcpyDeviceToHost);
  ex_re = ex[0].re;
  ex_im = ex[0].im;
  for (absx31 = 0; absx31 < 511; absx31++) {
    if (rtIsNaN(ex[absx31 + 1].re) || rtIsNaN(ex[absx31 + 1].im)) {
      c_locationsInSourceMapToDestina = false;
    } else if (rtIsNaN(ex_re) || rtIsNaN(ex_im)) {
      c_locationsInSourceMapToDestina = true;
    } else {
      if ((fabs(ex_re) > 8.9884656743115785E+307) || (fabs(ex_im) >
           8.9884656743115785E+307)) {
        c_locationsInSourceMapToDestina = true;
      } else {
        c_locationsInSourceMapToDestina = false;
      }

      if ((fabs(ex[absx31 + 1].re) > 8.9884656743115785E+307) || (fabs(ex[absx31
            + 1].im) > 8.9884656743115785E+307)) {
        SCALEB = true;
      } else {
        SCALEB = false;
      }

      if (c_locationsInSourceMapToDestina || SCALEB) {
        x = rt_hypotd_snf(ex_re / 2.0, ex_im / 2.0);
        br = rt_hypotd_snf(ex[absx31 + 1].re / 2.0, ex[absx31 + 1].im / 2.0);
      } else {
        x = rt_hypotd_snf(ex_re, ex_im);
        br = rt_hypotd_snf(ex[absx31 + 1].re, ex[absx31 + 1].im);
      }

      if (x == br) {
        absar = fabs(ex_re);
        absai = fabs(ex_im);
        absbr = fabs(ex[absx31 + 1].re);
        absbi = fabs(ex[absx31 + 1].im);
        if (absar > absai) {
          Ma = absar;
          absar = absai;
        } else {
          Ma = absai;
        }

        if (absbr > absbi) {
          absai = absbr;
          absbr = absbi;
        } else {
          absai = absbi;
        }

        if (Ma > absai) {
          if (absar < absbr) {
            x = Ma - absai;
            br = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + absai / 2.0) * (absbr
              - absar);
          } else {
            x = Ma;
            br = absai;
          }
        } else if (Ma < absai) {
          if (absar > absbr) {
            br = absai - Ma;
            x = (absar / 2.0 + absbr / 2.0) / (Ma / 2.0 + absai / 2.0) * (absar
              - absbr);
          } else {
            x = Ma;
            br = absai;
          }
        } else {
          x = absar;
          br = absbr;
        }

        if (x == br) {
          x = rt_atan2d_snf(ex_im, ex_re);
          br = rt_atan2d_snf(ex[absx31 + 1].im, ex[absx31 + 1].re);
          if (x == br) {
            br = ex[absx31 + 1].re;
            absbi = ex[absx31 + 1].im;
            if (x > 0.78539816339744828) {
              if (x > 2.3561944901923448) {
                x = -ex_im;
                br = -absbi;
              } else {
                x = -ex_re;
                br = -br;
              }
            } else if (x > -0.78539816339744828) {
              x = ex_im;
              br = absbi;
            } else if (x > -2.3561944901923448) {
              x = ex_re;
            } else {
              x = -ex_im;
              br = -absbi;
            }

            if (x == br) {
              x = 0.0;
              br = 0.0;
            }
          }
        }
      }

      c_locationsInSourceMapToDestina = (x < br);
    }

    if (c_locationsInSourceMapToDestina) {
      ex_re = ex[absx31 + 1].re;
      ex_im = ex[absx31 + 1].im;
      absx11 = absx31 + 1;
    }
  }

  if (absx11 + 1 > 256) {
    itmp = absx11 - 512;
  } else {
    itmp = absx11;
  }

  hipMemcpy(gpu_absx11, &absx11, 4ULL, hipMemcpyHostToDevice);
  regseqRT_coderGPU_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_idx,
    gpu_absx11);
  *tx = itmp;
  hipMemcpy(&absx11, gpu_absx11, 4ULL, hipMemcpyDeviceToHost);
  *ty = absx11;
  regseqRT_coderGPU_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_absx11,
    itmp, *gpu_NonFillOutputLocY);
  regseqRT_coderGPU_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_NonFillOutputLocY, *gpu_z1);
  z1_dirtyOnGpu = true;
  c_locationsInSourceMapToDestina = true;
  absx11 = 0;
  exitg1 = false;
  while ((!exitg1) && (absx11 < 2)) {
    if (z1_dirtyOnGpu) {
      hipMemcpy(&z1[0], gpu_z1, 16ULL, hipMemcpyDeviceToHost);
      z1_dirtyOnGpu = false;
    }

    if (!(z1[absx11] == 0.0)) {
      c_locationsInSourceMapToDestina = false;
      exitg1 = true;
    } else {
      absx11++;
    }
  }

  if (c_locationsInSourceMapToDestina) {
    regseqRT_coderGPU_kernel26<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (regFrame);
    regseqRT_coderGPU_kernel27<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_NonFillOutputLocY, b_gpu_NonFillOutputLocY, c_gpu_NonFillOutputLocY);
    regseqRT_coderGPU_kernel28<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (b_gpu_NonFillOutputLocY, c_gpu_NonFillOutputLocY, *gpu_VWorld,
       *gpu_UWorld);
    hipMemcpy(&UWorld[0], gpu_UWorld, 4ULL, hipMemcpyDeviceToHost);
    c_intrinsicSourceBoundingRectan[0] = UWorld[0];
    c_intrinsicSourceBoundingRectan[1] = UWorld[1];
    c_intrinsicSourceBoundingRectan[2] = UWorld[1];
    c_intrinsicSourceBoundingRectan[3] = UWorld[0];
    hipMemcpy(&VWorld[0], gpu_VWorld, 4ULL, hipMemcpyDeviceToHost);
    d_intrinsicSourceBoundingRectan[0] = VWorld[0];
    d_intrinsicSourceBoundingRectan[1] = VWorld[0];
    d_intrinsicSourceBoundingRectan[2] = VWorld[1];
    d_intrinsicSourceBoundingRectan[3] = VWorld[1];
    c_locationsInSourceMapToDestina = false;
    absx11 = 0;
    exitg1 = false;
    while ((!exitg1) && (absx11 < 4)) {
      if ((c_intrinsicSourceBoundingRectan[absx11] >= 0.5) &&
          (c_intrinsicSourceBoundingRectan[absx11] <= 512.5) &&
          (d_intrinsicSourceBoundingRectan[absx11] >= 0.5) &&
          (d_intrinsicSourceBoundingRectan[absx11] <= 512.5)) {
        c_locationsInSourceMapToDestina = true;
        exitg1 = true;
      } else {
        absx11++;
      }
    }

    if (c_locationsInSourceMapToDestina) {
      absx11 = UWorld[0];
      if (1 > UWorld[0]) {
        absx11 = 1;
      }

      UWorldClippedToBounds[0] = static_cast<int16_T>(absx11);
      absx11 = UWorld[1];
      if (512 < UWorld[1]) {
        absx11 = 512;
      }

      UWorldClippedToBounds[1] = static_cast<int16_T>(absx11);
      absx11 = VWorld[0];
      if (1 > VWorld[0]) {
        absx11 = 1;
      }

      UWorld[0] = static_cast<int16_T>(absx11);
      absx11 = VWorld[1];
      if (512 < VWorld[1]) {
        absx11 = 512;
      }

      UWorld[1] = static_cast<int16_T>(absx11);
      regseqRT_coderGPU_kernel29<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (*gpu_NonFillOutputLocY, b_gpu_NonFillOutputLocY,
         c_gpu_NonFillOutputLocY);
      hipMemcpy(gpu_UWorld, &UWorld[0], 4ULL, hipMemcpyHostToDevice);
      hipMemcpy(gpu_UWorldClippedToBounds, &UWorldClippedToBounds[0], 4ULL,
                 hipMemcpyHostToDevice);
      regseqRT_coderGPU_kernel30<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
        (b_gpu_NonFillOutputLocY, *gpu_UWorld, c_gpu_NonFillOutputLocY,
         *gpu_UWorldClippedToBounds, *gpu_outputC, *gpu_z1,
         *gpu_containedSubscripts, *gpu_NonFillOutputLocY, *gpu_VWorld);
      containedSubscripts_dirtyOnGpu = true;
      outputC_dirtyOnGpu = true;
      z1_dirtyOnGpu = true;
      absx11 = 0;
      for (absx31 = 0; absx31 < 2; absx31++) {
        if (containedSubscripts_dirtyOnGpu) {
          hipMemcpy(&containedSubscripts[0], gpu_containedSubscripts, 2ULL,
                     hipMemcpyDeviceToHost);
          containedSubscripts_dirtyOnGpu = false;
        }

        if (containedSubscripts[absx31]) {
          absx11++;
        }
      }

      tmp_size[0] = 1;
      tmp_size[1] = absx11;
      tmp_size_dirtyOnCpu = true;
      itmp = 0;
      for (absx31 = 0; absx31 < 2; absx31++) {
        if (containedSubscripts_dirtyOnGpu) {
          hipMemcpy(&containedSubscripts[0], gpu_containedSubscripts, 2ULL,
                     hipMemcpyDeviceToHost);
          containedSubscripts_dirtyOnGpu = false;
        }

        if (containedSubscripts[absx31]) {
          tmp_data[itmp] = static_cast<int8_T>(absx31 + 1);
          tmp_data_dirtyOnCpu = true;
          itmp++;
        }
      }

      cInit_size[1] = absx11;
      validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((tmp_size[1]
        - 1) + 1LL), &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        if (tmp_data_dirtyOnCpu) {
          hipMemcpy(gpu_tmp_data, &tmp_data[0], tmp_size[0] * tmp_size[1] *
                     sizeof(int8_T), hipMemcpyHostToDevice);
          tmp_data_dirtyOnCpu = false;
        }

        hipMemcpy(gpu_tmp_size, &tmp_size[0], 8ULL, hipMemcpyHostToDevice);
        tmp_size_dirtyOnCpu = false;
        regseqRT_coderGPU_kernel31<<<grid, block>>>(*gpu_VWorld, *gpu_tmp_data, *
          gpu_tmp_size, *gpu_rPix_data);
        rPix_data_dirtyOnGpu = true;
      }

      rInit_size[1] = tmp_size[1];
      b_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((tmp_size
        [1] - 1) + 1LL), &b_grid, &b_block, 1024U, 65535U);
      if (b_validLaunchParams) {
        if (tmp_data_dirtyOnCpu) {
          hipMemcpy(gpu_tmp_data, &tmp_data[0], tmp_size[0] * tmp_size[1] *
                     sizeof(int8_T), hipMemcpyHostToDevice);
        }

        if (tmp_size_dirtyOnCpu) {
          hipMemcpy(gpu_tmp_size, &tmp_size[0], 8ULL, hipMemcpyHostToDevice);
        }

        regseqRT_coderGPU_kernel32<<<b_grid, b_block>>>(*gpu_NonFillOutputLocY, *
          gpu_tmp_data, *gpu_tmp_size, *gpu_rInit_data);
      }

      absx11 = cInit_size[1];
      c_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((absx11 -
        1) + 1LL), &c_grid, &c_block, 1024U, 65535U);
      if (c_validLaunchParams) {
        hipMemcpy(gpu_absx11, &absx11, 4ULL, hipMemcpyHostToDevice);
        regseqRT_coderGPU_kernel33<<<c_grid, c_block>>>(gpu_absx11,
          *gpu_rPix_data);
        rPix_data_dirtyOnGpu = true;
      }

      cPix_size[1] = static_cast<int8_T>(cInit_size[1]);
      absx11 = static_cast<int8_T>(cInit_size[1]);
      d_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((absx11 -
        1) + 1LL), &d_grid, &d_block, 1024U, 65535U);
      if (d_validLaunchParams) {
        hipMemcpy(gpu_absx11, &absx11, 4ULL, hipMemcpyHostToDevice);
        regseqRT_coderGPU_kernel34<<<d_grid, d_block>>>(*gpu_rPix_data,
          gpu_absx11, *gpu_cPix_data);
        cPix_data_dirtyOnGpu = true;
      }

      absx11 = rInit_size[1];
      e_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((absx11 -
        1) + 1LL), &e_grid, &e_block, 1024U, 65535U);
      if (e_validLaunchParams) {
        hipMemcpy(gpu_absx11, &absx11, 4ULL, hipMemcpyHostToDevice);
        regseqRT_coderGPU_kernel35<<<e_grid, e_block>>>(gpu_absx11,
          *gpu_rInit_data);
      }

      rPix_size[1] = static_cast<int8_T>(rInit_size[1]);
      absx11 = static_cast<int8_T>(rInit_size[1]);
      f_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>((absx11 -
        1) + 1LL), &f_grid, &f_block, 1024U, 65535U);
      if (f_validLaunchParams) {
        hipMemcpy(gpu_absx11, &absx11, 4ULL, hipMemcpyHostToDevice);
        regseqRT_coderGPU_kernel36<<<f_grid, f_block>>>(*gpu_rInit_data,
          gpu_absx11, *gpu_rPix_data);
        rPix_data_dirtyOnGpu = true;
      }

      itmp = 0;
      for (absx31 = 0; absx31 < 2; absx31++) {
        if (containedSubscripts_dirtyOnGpu) {
          hipMemcpy(&containedSubscripts[0], gpu_containedSubscripts, 2ULL,
                     hipMemcpyDeviceToHost);
          containedSubscripts_dirtyOnGpu = false;
        }

        if (containedSubscripts[absx31]) {
          if (outputC_dirtyOnGpu) {
            hipMemcpy(&outputC[0], gpu_outputC, 16ULL, hipMemcpyDeviceToHost);
            outputC_dirtyOnGpu = false;
          }

          if (cPix_data_dirtyOnGpu) {
            hipMemcpy(&cPix_data[0], gpu_cPix_data, cPix_size[1] * sizeof
                       (real_T), hipMemcpyDeviceToHost);
            cPix_data_dirtyOnGpu = false;
          }

          outputC[absx31] = cPix_data[itmp];
          itmp++;
        }
      }

      itmp = 0;
      for (absx31 = 0; absx31 < 2; absx31++) {
        if (containedSubscripts_dirtyOnGpu) {
          hipMemcpy(&containedSubscripts[0], gpu_containedSubscripts, 2ULL,
                     hipMemcpyDeviceToHost);
          containedSubscripts_dirtyOnGpu = false;
        }

        if (containedSubscripts[absx31]) {
          if (z1_dirtyOnGpu) {
            hipMemcpy(&z1[0], gpu_z1, 16ULL, hipMemcpyDeviceToHost);
            z1_dirtyOnGpu = false;
          }

          if (rPix_data_dirtyOnGpu) {
            hipMemcpy(&rPix_data[0], gpu_rPix_data, rPix_size[1] * sizeof
                       (real_T), hipMemcpyDeviceToHost);
            rPix_data_dirtyOnGpu = false;
          }

          z1[absx31] = rPix_data[itmp];
          itmp++;
        }
      }

      if (z1_dirtyOnGpu) {
        hipMemcpy(&z1[0], gpu_z1, 16ULL, hipMemcpyDeviceToHost);
      }

      if (z1[0] > z1[1]) {
        itmp = 0;
      } else {
        itmp = static_cast<int32_T>(z1[0]) - 1;
      }

      if (outputC_dirtyOnGpu) {
        hipMemcpy(&outputC[0], gpu_outputC, 16ULL, hipMemcpyDeviceToHost);
      }

      if (outputC[0] > outputC[1]) {
        absx11 = 0;
      } else {
        absx11 = static_cast<int32_T>(outputC[0]) - 1;
      }

      absx31 = UWorld[0] - 1;
      p1 = UWorld[1] - 1;
      p2 = UWorldClippedToBounds[0] - 1;
      p3 = UWorldClippedToBounds[1] - 1;
      g_validLaunchParams = mwGetLaunchParameters(static_cast<real_T>(((p1 -
        absx31) + 1LL) * ((p3 - p2) + 1LL)), &g_grid, &g_block, 1024U, 65535U);
      if (g_validLaunchParams) {
        hipMemcpy(gpu_absx11, &absx11, 4ULL, hipMemcpyHostToDevice);
        regseqRT_coderGPU_kernel37<<<g_grid, g_block>>>(currentFrame, p2,
          gpu_absx11, itmp, absx31, p1, p3, regFrame);
      }
    }
  } else {
    hipMemcpy(gpu_iv, (void *)&iv[0], 3ULL, hipMemcpyHostToDevice);
    hipMemcpy(gpu_iv1, (void *)&iv1[0], 3ULL, hipMemcpyHostToDevice);
    regseqRT_coderGPU_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_iv,
      *gpu_iv1, *gpu_A);
    regseqRT_coderGPU_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_NonFillOutputLocY, *gpu_A);
    regseqRT_coderGPU_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_A);
    regseqRT_coderGPU_kernel17<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_dstYWorld, *gpu_dstXWorld);
    regseqRT_coderGPU_kernel18<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_A, *
      gpu_x);
    p1 = 0;
    p2 = 3;
    p3 = 6;
    hipMemcpy(&A[0], gpu_A, 18ULL, hipMemcpyDeviceToHost);
    absx11 = A[0];
    absx31 = static_cast<int32_T>(fabs(static_cast<real_T>(A[2])));
    if ((A[1] > absx11) && (1 > absx31)) {
      p1 = 3;
      p2 = 0;
      regseqRT_coderGPU_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_A,
        *gpu_x);
    } else {
      if (absx31 > absx11) {
        p1 = 6;
        p3 = 0;
        regseqRT_coderGPU_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
          (*gpu_A, *gpu_x);
      }
    }

    hipMemcpy(&b_x[0], gpu_x, 72ULL, hipMemcpyDeviceToHost);
    absbi = b_x[1] / b_x[0];
    b_x[1] /= b_x[0];
    absai = b_x[2] / b_x[0];
    b_x[2] /= b_x[0];
    b_x[4] -= absbi * b_x[3];
    b_x[5] -= absai * b_x[3];
    b_x[7] -= absbi * b_x[6];
    b_x[8] -= absai * b_x[6];
    if (fabs(b_x[5]) > fabs(b_x[4])) {
      itmp = p2;
      p2 = p3;
      p3 = itmp;
      b_x[1] = absai;
      b_x[2] = absbi;
      absbi = b_x[4];
      b_x[4] = b_x[5];
      b_x[5] = absbi;
      absbi = b_x[7];
      b_x[7] = b_x[8];
      b_x[8] = absbi;
    }

    absbi = b_x[5] / b_x[4];
    b_x[5] /= b_x[4];
    b_x[8] -= absbi * b_x[7];
    absbi = (b_x[5] * b_x[1] - b_x[2]) / b_x[8];
    absai = -(b_x[1] + b_x[7] * absbi) / b_x[4];
    tinv[p1] = ((1.0 - b_x[3] * absai) - b_x[6] * absbi) / b_x[0];
    tinv[p1 + 1] = absai;
    tinv[p1 + 2] = absbi;
    absbi = -b_x[5] / b_x[8];
    absai = (1.0 - b_x[7] * absbi) / b_x[4];
    tinv[p2] = -(b_x[3] * absai + b_x[6] * absbi) / b_x[0];
    tinv[p2 + 1] = absai;
    tinv[p2 + 2] = absbi;
    absbi = 1.0 / b_x[8];
    absai = -b_x[7] * absbi / b_x[4];
    tinv[p3] = -(b_x[3] * absai + b_x[6] * absbi) / b_x[0];
    tinv[p3 + 1] = absai;
    tinv[p3 + 2] = absbi;
    regseqRT_coderGPU_kernel21<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>(tinv
      [5], tinv[4], tinv[3], tinv[2], tinv[1], tinv[0], *gpu_dstYWorld,
      *gpu_dstXWorld, *gpu_srcXIntrinsic);
    regseqRT_coderGPU_kernel22<<<dim3(4U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_inputImagePadded);
    regseqRT_coderGPU_kernel23<<<dim3(3U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_inputImagePadded);
    regseqRT_coderGPU_kernel24<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (currentFrame, *gpu_inputImagePadded);
    regseqRT_coderGPU_kernel25<<<dim3(512U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_inputImagePadded, *gpu_dstXWorld, *gpu_srcXIntrinsic, regFrame);
  }

  cufftEnsureDestruction();
  hipFree(*gpu_dstYWorld);
  hipFree(*gpu_y1);
  hipFree(*gpu_X);
  hipFree(*gpu_f);
  hipFree(*gpu_idx);
  hipFree(*gpu_ex);
  hipFree(gpu_absx11);
  hipFree(*gpu_NonFillOutputLocY);
  hipFree(*gpu_z1);
  hipFree(*gpu_iv);
  hipFree(*gpu_iv1);
  hipFree(*gpu_A);
  hipFree(*gpu_x);
  hipFree(*gpu_dstXWorld);
  hipFree(*gpu_inputImagePadded);
  hipFree(*gpu_srcXIntrinsic);
  hipFree(b_gpu_NonFillOutputLocY);
  hipFree(c_gpu_NonFillOutputLocY);
  hipFree(*gpu_UWorld);
  hipFree(*gpu_VWorld);
  hipFree(*gpu_UWorldClippedToBounds);
  hipFree(*gpu_containedSubscripts);
  hipFree(*gpu_tmp_size);
  hipFree(*gpu_tmp_data);
  hipFree(*gpu_rPix_data);
  hipFree(*gpu_rInit_data);
  hipFree(*gpu_outputC);
  hipFree(*gpu_cPix_data);
}

/* End of code generation (regseqRT_coderGPU.cu) */
